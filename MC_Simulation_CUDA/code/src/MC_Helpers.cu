#include "hip/hip_runtime.h"
//
// Created by mustafa on 6/3/20.
//

#include "../headers/MC_Photon.cuh"
#include "../headers/MC_Helpers.cuh"

void MCHelpers::streamOut(MC_Photon *_cpuPhotons, int n) {
    FILE *output;
    output = fopen("Results.csv", "w");
    std::string state;
    //Header
    fprintf(output, "%s,%s,%s,%s,%s\n", "X", "Y", "Z", "Weight", "State");
    for (int i = 0; i < n; i++) {
        switch (_cpuPhotons[i].state()) {
            case (MC_Photon::TERMINATED):
                state = "TERMINATED";
                break;
            case (MC_Photon::ROAMING):
                state = "ROAMING";
                break;
            case (MC_Photon::DETECTED):
                state = "DETECTED";
                break;
            case (MC_Photon::ESCAPED):
                state = "ESCAPED";
                break;
        }
        // Streaming out my output in a log file
        fprintf(output, "%f,%f,%f,%f,%s\n", _cpuPhotons[i].position().x(), _cpuPhotons[i].position().y(),
                _cpuPhotons[i].position().z(), _cpuPhotons[i].weight(), state.c_str());
    }
}