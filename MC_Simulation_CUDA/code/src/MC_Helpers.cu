#include "hip/hip_runtime.h"
//
// Created by mustafa on 6/3/20.
//

#include "../headers/MC_Photon.cuh"
#include "../headers/MC_Helpers.cuh"

void MCHelpers::streamOut(MC_Photon *_cpuPhotons, int n) {
    FILE *output;
    output = fopen("Results.csv", "w");
    std::string state;
    //Header
    fprintf(output, "%s,%s,%s,%s,%s\n", "X", "Y", "Z", "Weight", "State");
    for (int i = 0; i < n; i++) {
        switch (_cpuPhotons[i].state()) {
            case (MC_Photon::TERMINATED):
                state = "TERMINATED";
                break;
            case (MC_Photon::ROAMING):
                state = "ROAMING";
                break;
            case (MC_Photon::DETECTED):
                state = "DETECTED";
                break;
            case (MC_Photon::ESCAPED):
                state = "ESCAPED";
                break;
        }
        // Streaming out my output in a log file
        fprintf(output, "%f,%f,%f,%f,%s\n", _cpuPhotons[i].position().x(), _cpuPhotons[i].position().y(),
                _cpuPhotons[i].position().z(), _cpuPhotons[i].weight(), state.c_str());
    }
}

void
MCHelpers::endMsg(int p_n, float d_r, MC_Point d_p, MC_Vector d_n, float t_r, float t_ac, float t_sc, MC_Point t_c1,
                  MC_Point t_c2) {
    printf("\nSimulation Parameters:\n\n\tGeneral Parameters:\n\t\tNumber of Photons: %i\n\n\tDetector Parameters:\n\t\tRadius: %4.1f\n\t\tPosition: ( %4.1f, %4.1f, %4.1f )\n\t\tNormal: ( %4.1f, %4.1f, %4.1f )\n\n\tTissue Parameters:\n\t\tRadius: %4.1f\n\t\tAbsorption Coefficient: %4.1f\n\t\tScattering Coefficient: %4.1f\n\t\tCenter #1: ( %4.1f, %4.1f, %4.1f )\n\t\tCenter #2: ( %4.1f, %4.1f, %4.1f )\n\nCode executed successfully!\n",
           p_n, d_r, d_p.x(), d_p.y(), d_p.z(), d_n.x(), d_n.y(), d_n.z(), t_r, t_ac, t_sc, t_c1.x(), t_c1.y(),
           t_c1.z(), t_c2.x(), t_c2.y(), t_c2.z());
}