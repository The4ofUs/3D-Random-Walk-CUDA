#include "hip/hip_runtime.h"
//
// Created by mustafa on 6/3/20.
//

#include <stdexcept>
#include "../headers/MC_Tissue.cuh"
#include "../headers/MC_Math.cuh"


__host__ MC_Tissue::MC_Tissue(float const radius, MC_Point const c0, MC_Point const c1, float const ac, float const sc) {
    if (radius > 0 && ac > 0 && sc > 0) {
        this->_radius = radius;
        this->_interface = c0;
        this->_remote = c1;
        this->_normal = MCMath::normalized(MC_Vector(c0, c1));
        this->Ma = ac;
        this->Ms = sc;
        this->Mt = ac + sc;
    } else { throw std::invalid_argument("MC_Tissue::MC_Tissue : Illegal Argument!"); }
}

__device__ bool MC_Tissue::escaped(MC_Point const position) {
    MC_Point A = position;
    MC_Point B = this->_interface;
    MC_Vector C = this->_normal;
    float t =
            MCMath::dot(C, (A - B)) / MCMath::norm(C) * MCMath::norm(C);
    MC_Point P = B + C * t;
    float d = MCMath::absDistance(A, P);
    if (d > this->_radius) { return true; }
    float D = MCMath::norm(this->_remote - this->_interface);
    float E = MCMath::norm(P - this->_interface);
    if (E > D) { return true; }
    E = MCMath::norm((P - this->_remote));
    return E > D;
}

__device__ void MC_Tissue::attenuate(MC_Photon &photon) const {
    float newWeight = this->Ms * photon.weight() / this->Mt;
    photon.setWeight(newWeight);
}

__host__ __device__ MC_Point MC_Tissue::interface() {
    return this->_interface;
}

__host__ __device__ MC_Point MC_Tissue::remote() {
    return this->_remote;
}

__device__  __host__ float MC_Tissue::thickness() {
    return MCMath::absDistance(this->_interface, this->_remote);
}

__host__ __device__ float MC_Tissue::radius() const {
    return this->_radius;
}

__device__ __host__ float MC_Tissue::attenuationCoefficient() const {
    return this->Mt;
}

__host__ __device__ float MC_Tissue::absorption() const {
    return this->Ma;
}

__host__ __device__ float MC_Tissue::scattering() const {
    return this->Ms;
}
