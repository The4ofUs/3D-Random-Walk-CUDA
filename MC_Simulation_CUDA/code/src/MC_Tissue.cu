#include "hip/hip_runtime.h"
//
// Created by mustafa on 6/3/20.
//

#include <stdexcept>
#include "../headers/MC_Tissue.cuh"
#include "../headers/MC_Math.cuh"


__host__ __device__ MC_Tissue::MC_Tissue(float const radius, MC_Point const c0, MC_Point const c1, float const ac, float const sc, float rn) {
    if (radius > 0 && ac > 0 && sc > 0 && rn > 0) {
        _radius = radius;
        _interface = c0;
        _remote = c1;
        _normal = MCMath::normalized(MC_Vector(c0, c1));
        _Ma = ac;
        _Ms = sc;
        _Mt = ac + sc;
        _n = rn;
    }
}

__device__ bool MC_Tissue::escaped(MC_Point const position) {
    MC_Point A = position;
    MC_Point B = _interface;
    MC_Vector C = _normal;
    float t = MCMath::dot(C, (A - B)) / MCMath::norm(C) * MCMath::norm(C);
    MC_Point P = B + C * t;
    float d = MCMath::absDistance(A, P);
    if (d > _radius) { return true; }
    float D = MCMath::norm(_remote - _interface);
    float E = MCMath::norm(P - _interface);
    if (E > D) { return true; }
    E = MCMath::norm((P - _remote));
    return E > D;
}

__device__ void MC_Tissue::attenuate(MC_Photon &photon) const {
    float newWeight = _Ms * photon.weight() / _Mt;
    photon.setWeight(newWeight);
}

__host__ __device__ MC_Point MC_Tissue::interface() {
    return _interface;
}

__host__ __device__ MC_Point MC_Tissue::remote() {
    return _remote;
}

__device__  __host__ float MC_Tissue::thickness() {
    return MCMath::absDistance(_interface, _remote);
}

__host__ __device__ float MC_Tissue::radius() const {
    return _radius;
}

__device__ __host__ float MC_Tissue::attenuationCoefficient() const {
    return _Mt;
}

__host__ __device__ float MC_Tissue::absorption() const {
    return _Ma;
}

__host__ __device__ float MC_Tissue::scattering() const {
    return _Ms;
}

__host__ __device__ float MC_Tissue::n() const {
    return _n;
}
