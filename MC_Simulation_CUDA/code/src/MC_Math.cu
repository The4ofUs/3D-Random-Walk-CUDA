#include "hip/hip_runtime.h"
//
// Created by mustafa on 6/3/20.
//

#include "../headers/MC_Math.cuh"

__host__ __device__ float MCMath::absDistance(MC_Point const p1, MC_Point const p2)
{
    return sqrtf(((p2.x() - p1.x()) * (p2.x() - p1.x()) + (p2.y() - p1.y()) * (p2.y() - p1.y()) + (p2.z() - p1.z()) * (p2.z() - p1.z())));
}

__host__ __device__ float MCMath::norm(MC_Point const p)
{
    return sqrtf(((p.x()) * (p.x()) + (p.y()) * (p.y()) + (p.z()) * (p.z())));
}

__device__ __host__ float MCMath::dot(MC_Vector const v1, MC_Vector const v2)
{
    return v1.x() * v2.x() + v1.y() * v2.y() + v1.z() * v2.z();
}

__device__ __host__ MC_Vector MCMath::cross(MC_Vector const v1, MC_Vector const v2)
{
    float X = v1.y() * v2.z() - v1.z() * v2.y();
    float Y = (-1) * v1.x() * v2.z() + v1.z() * v2.x();
    float Z = v1.x() * v2.y() - v1.y() * v2.x();
    return {X, Y, Z};
}

__device__ __host__ MC_Vector MCMath::normalized(MC_Vector const v)
{
    float norm = sqrtf((powf(v.x(), 2) + powf(v.y(), 2) + powf(v.z(), 2)));
    float x = v.x() / norm;
    float y = v.y() / norm;
    float z = v.z() / norm;
    return {x, y, z};
}

// Needs to be tested
__device__ __host__ MC_Point MCMath::rayTip(MC_Point const origin, MC_Vector const direction, float step)
{
    return (origin + (direction * step));
}

__device__ __host__ float MCMath::point2PlaneDist(MC_Point p, MC_Point coord, MC_Vector n) {
    MC_Vector v = MC_Vector(coord, p);
    return abs(MCMath::dot(v, n));
}