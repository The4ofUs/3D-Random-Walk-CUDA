#include "hip/hip_runtime.h"
//
// Created by mustafa on 6/3/20.
//

#include "../headers/MC_RNG.cuh"
#include "../headers/MC_Math.cuh"

__device__ float MC_RNG::generate(hiprandState *globalState, int const i) {
    hiprandState localState = globalState[i];
    float random = hiprand_uniform(&localState);
    globalState[i] = localState;
    return random;
}

__device__ float MC_RNG::getRandomNumber(hiprandState *states, int i) {
    float step;
    step = generate(states, i);
    return step;
}

__device__ MC_Vector MC_RNG::getRandomDirection(hiprandState *globalState, int const i) {
    float u = generate(globalState, i);
    float v = generate(globalState, i);

    float theta = 2 * (float) M_PI * u;
    float phi = acos(1 - 2 * v);

    // Transforming into the cartesian space
    float x = sin(phi) * cos(theta);
    float y = sin(phi) * sin(theta);
    float z = cos(phi);

    return MCMath::normalized(MC_Vector(x, y, z));
}

__device__ MC_Point MC_RNG::getRandomPoint(hiprandState *globalState, int const i) {
    float u = generate(globalState, i);
    float v = generate(globalState, i);

    float theta = 2 * (float) M_PI * u;
    float phi = acos(1 - 2 * v);

    // Transforming into the cartesian space
    float x = sin(phi) * cos(theta);
    float y = sin(phi) * sin(theta);
    float z = cos(phi);

    return {x, y, z};
}

__device__ void MC_RNG::roulette(MC_Photon &photon, float const chance, hiprandState *globalState, int const i) {
    if (generate(globalState, i) >= chance) {
        photon.terminate();
    } else {
        photon.boost(chance);
    }
}

__device__ float MC_RNG::getRandomStep(hiprandState *states, int i, float coefficient) {
    return ((-1 * log(MC_RNG::getRandomNumber(states, i))) / coefficient);
}

__device__ MC_Path MC_RNG::getRandomPath(hiprandState *states, int i, MC_Point origin, float coefficient) {
    return {origin, MC_RNG::getRandomDirection(states, i),
            MC_RNG::getRandomStep(states, i, coefficient)};
}
