#include "hip/hip_runtime.h"
//
// Created by mustafa on 6/4/20.
//

#include <stdexcept>
#include "../headers/MC_MLTissue.cuh"
#include "../headers/MC_Math.cuh"

__host__ MC_MLTissue::MC_MLTissue(float const radius, MC_Point const c0, MC_Point const c1,
                                  std::vector<float> const &absorpCoeffs,
                                  std::vector<float> const &scatterCoeffs) {
    if (radius > 0 && absorpCoeffs.size() == scatterCoeffs.size()) {
        this->_radius = radius;
        this->_interface = c0;
        this->_remote = c1;
        this->_normal = MCMath::normalized(MC_Vector(c0, c1));
        this->_size = absorpCoeffs.size();
        this->_thickness = MCMath::absDistance(this->_interface, this->_remote);
        for (int i = 0; i < this->_size; i++) {
            // This method of population the array with undefined Points is reckless and should be modified
            MC_Point interface =
                    this->_interface + this->_normal * ((float) i * this->_thickness / (float) this->_size);
            MC_Point remote =
                    this->_interface + this->_normal * ((float) (i + 1) * this->_thickness / (float) this->_size);
            this->_layers[i] = MC_Tissue(radius, interface, remote, absorpCoeffs[i],
                                         scatterCoeffs[i]);
        }
    } else { throw std::invalid_argument("MC_MLTissue::MC_MLTissue : Illegal Argument!"); }
}

__device__ void MC_MLTissue::attenuate(MC_Photon &photon) {
    MC_Tissue t = whichLayer(photon.position());
    t.attenuate(photon);
}

__device__ MC_Tissue MC_MLTissue::whichLayer(MC_Point const position) {
    /**
     * P : Current position of the photon
     * R : Projection of P on the tissue interface
     * Q : Point on the tissue interface ( Center of the interface for simplicity )
     * N : Normal to the interface
     * V : Vector from Q to P
     */
    MC_Point P = position;
    MC_Point Q = this->_interface;
    MC_Vector N = this->_normal;
    MC_Vector V = MC_Vector(Q, P);
    float d = abs(MCMath::dot(V, N));
    int index = (int) round((d / this->_thickness) * (float) this->_size);
    MC_Tissue currentTissue = this->_layers[index];
    return currentTissue;
}

__device__ bool MC_MLTissue::escaped(MC_Point const position) {
    MC_Point A = position;
    MC_Point B = this->_interface;
    MC_Vector C = this->_normal;
    float t = MCMath::dot(C, (A - B)) / MCMath::norm(C) * MCMath::norm(C);
    MC_Point P = B + C * t;
    float d = MCMath::absDistance(A, P);
    if (d > this->_radius) { return true; }
    float D = MCMath::norm(this->_remote - this->_interface);
    float E = MCMath::norm(P - this->_interface);
    if (E > D) { return true; }
    E = MCMath::norm((P - this->_remote));
    return E > D;
}

__device__ int MC_MLTissue::size() const {
    return this->_size;
}

__device__ float MC_MLTissue::coefficient(MC_Point position) { return whichLayer(position).attenuationCoefficient(); }

__device__ bool MC_MLTissue::isCrossing(MC_Path path) {
    /*
     * Vector from interface to path origin
     */
    MC_Vector v1 = MC_Vector(_interface, path.origin());
    /*
     * Vector from interface to path end
     */
    MC_Vector v2 = MC_Vector(_interface, path.tip());
    /*
     * Distance of ray origin to the interface plane
     */
    float d1 = abs(MCMath::dot(v1, _normal));
    /*
     * Distance of ray tip to the interface plane
     */
    float d2 = abs(MCMath::dot(v2, _normal));
    /*
     * Which portion the origin lies in
     */
    int q1 = (int) ((d1 / _thickness) * (float) _size);
    /*
     * Which portion the tip lies in
     */
    int q2 = (int) ((d2 / _thickness) * (float) _size);

    return q1 != q2;
}

__device__ MC_Point MC_MLTissue::crossingPoint(MC_Path path) {
    /*
     * TODO: Solve identifying the idx of the layer as duality of coordinates are mixing up negative and positive
     */
    /*
     * Calculate the point of intersection between the path and layer boundary
     */
    /*
     * Layer thickness
     */
    float portion = _thickness / (float) _size;
    /*
     * Vector from interface to path origin
     */
    MC_Vector v1 = MC_Vector(_interface, path.origin());
    /*
     * Perpendicular Distance of ray origin to the interface plane
     */
    float g = abs(MCMath::dot(v1, _normal));
    /*
     * idx of the current layer
     */
    int layerIdx = (int) (g / portion);
    /*
     * Distance of that boundary from the interface
     */
    float l = portion * ((float) layerIdx + 1);
    printf("Distance : %f\nidx : %d\n",l, layerIdx);
    /*
     * A Point on the boundary plane
     */
    MC_Point coord = _normal*l;
    float d = MCMath::dot(_normal,coord);
    /*
     * Parametrized step in the direction of the path to touch the boundary
     */
    float t = (d - MCMath::dot(_normal,path.origin()))/MCMath::dot(_normal,path.direction());
    /*
     * New tip that lies on the boundary
     */
    MC_Point newTip = path.origin() + path.direction() * t;
    return newTip;
}


MC_MLTissue::MC_MLTissue() = default;

