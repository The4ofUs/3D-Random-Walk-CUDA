#include "hip/hip_runtime.h"
//
// Created by mustafa on 6/4/20.
//

#include <stdexcept>
#include "../headers/MC_MLTissue.cuh"
#include "../headers/MC_Math.cuh"
#include <cassert>
#include <math.h>

__host__ MC_MLTissue::MC_MLTissue(float const radius, MC_Point const c0, MC_Point const c1,
                                  std::vector<float> const &absorpCoeffs,
                                  std::vector<float> const &scatterCoeffs, std::vector<float> const &refractIndices) {
    if (radius > 0 && absorpCoeffs.size() == scatterCoeffs.size()) {
        _radius = radius;
        _interface = c0;
        _remote = c1;
        _normal = MCMath::normalized(MC_Vector(c0, c1));
        _size = absorpCoeffs.size();
        _thickness = MCMath::absDistance(_interface, _remote);
        _portion = _thickness / (float) _size;
        for (int i = 0; i < _size; i++) {
            // This method of population the array with undefined Points is reckless and should be modified
            MC_Point interface =
                    _interface + _normal * ((float) i * _thickness / (float) _size);
            MC_Point remote =
                    _interface + _normal * ((float) (i + 1) * _thickness / (float) _size);
            _layers[i] = MC_Tissue(radius, interface, remote, absorpCoeffs[i],
                                   scatterCoeffs[i], refractIndices[i]);
        }
    } else { throw std::invalid_argument("MC_MLTissue::MC_MLTissue : Illegal Argument!"); }
}

__device__ void MC_MLTissue::attenuate(MC_Photon &photon) {
    MC_Tissue t = getLayer(whichLayer(photon.position()));
    t.attenuate(photon);
}

__device__ __host__ int MC_MLTissue::whichLayer(MC_Point const position) {
    float d = MCMath::point2PlaneDist(position, _interface, _normal);
    int index = (int) floor(d / _portion);
    return index;
}

__device__ MC_Tissue MC_MLTissue::getLayer(int const idx) {
    assert(idx < _size);
    return _layers[idx];
}

__device__ bool MC_MLTissue::escaped(MC_Path const path) {
    MC_Point A = path.tip();
    MC_Point B = _interface;
    MC_Vector C = _normal;
    float t = MCMath::dot(C, (A - B)) / MCMath::norm(C) * MCMath::norm(C);
    MC_Point P = B + C * t;
    float d = MCMath::absDistance(A, P);
    if (d > _radius) { return true; }
    float D = MCMath::norm(_remote - _interface);
    float E = MCMath::norm(P - _interface);
    if (E > D) { return true; }
    E = MCMath::norm((P - _remote));
    return E > D;
}

__device__ float MC_MLTissue::coefficient(MC_Point position) {
    return getLayer(whichLayer(position)).attenuationCoefficient();
}

__device__ __host__ bool MC_MLTissue::isCrossing(MC_Path path) {
    /*
     * Distance of ray origin to the interface plane
     */
    float d1 = MCMath::point2PlaneDist(path.origin(), _interface, _normal);
    /*
     * Ignore the photons that already lay exactly on the boundary, it's highly likely that they were moved there by the
     * algorithm itself
     */
    if (fmod(d1, _portion) == 0.f) {
        return false;
    }
    /*
     * Distance of ray tip to the interface plane
     */
    float d2 = MCMath::point2PlaneDist(path.tip(), _interface, _normal);
    /*
     * Which portion the origin lies in
     */
    int q1 = (int) ((d1 / _thickness) * (float) _size);
    /*
     * Which portion the tip lies in
     */
    int q2 = (int) ((d2 / _thickness) * (float) _size);
    return q1 != q2;
}

/*
 * Calculate the point of intersection between the path and layer boundary
 */
__device__ void MC_MLTissue::updatePath(MC_Path &path) {
    /*
     * First we need to get a point on the boundary, however we need to determine some parameters first:
     * 1) Which direction did the crossing happen
     * 2) at which boundary
     */
    MC_Point coord = _interface * (1 - ((float) whichBoundary(path) * _portion));
    float d = MCMath::dot(_normal, coord);
    float t = (d - MCMath::dot(_normal, path.origin())) / MCMath::dot(_normal, path.direction());
    path.setTip(path.origin() + path.direction() * t);
}

__device__ int MC_MLTissue::whichBoundary(MC_Path path) {
    Direction direction = whichDirection(path);
    float d1 = MCMath::point2PlaneDist(path.origin(), _interface, _normal);
    int layerIdx = (int) floor(d1 / _portion);
    if (direction == DOWN) return layerIdx + 1;
    else return layerIdx;
}

__device__ MC_MLTissue::Direction MC_MLTissue::whichDirection(const MC_Path path) {
    float d1 = MCMath::point2PlaneDist(path.origin(), _interface, _normal);
    float d2 = MCMath::point2PlaneDist(path.tip(), _interface, _normal);
    if ((d2 - d1) > 0) return DOWN;
    else return UP;
}

__device__ __host__ bool MC_MLTissue::onBoundary(const MC_Path path) {
    return fmod(MCMath::point2PlaneDist(path.tip(), _interface, _normal), _portion) == 0;
}

__device__ int MC_MLTissue::nextLayer(int idx, Direction dir) const {
    if (dir == UP) {
        if (idx > 0) return idx - 1;
        else return idx;
    } else {
        if (idx < _size - 1) return idx + 1;
        else return idx;
    }
}

__device__ bool MC_MLTissue::isReflected(MC_Path path, float random) {
    float incidentAngle = acos(abs(path.direction().z()));
    int idx1 = whichLayer(path.origin()) - 1;
    int idx2 = nextLayer(idx1, whichDirection(path));
    float ni = getLayer(idx1).n();
    float nt = getLayer(idx2).n();
    float R;
    if (ni > nt) {
        return true;
    } else {
        float transmitAngle = asin(((ni * sin(incidentAngle)) / nt));
        float term_1 = sin(incidentAngle - transmitAngle);
        float term_2 = sin(incidentAngle + transmitAngle);
        float term_3 = tan(incidentAngle - transmitAngle);
        float term_4 = tan(incidentAngle + transmitAngle);
        R = 0.5f * ((term_1 / term_2) + (term_3 / term_4));
        printf("------------- isReflected() -------------\nmue = %f\nincident angle = %f\ntransmitting angle = %f\nni = %f\nnt = %f\nterm_1 = %f\nterm_2 = %f\nterm_3 = %f\nterm_4 = %f\n",
               path.direction().z(),
               incidentAngle, transmitAngle, ni, nt, term_1, term_2, term_3, term_4);
        if (R >= random) return true;
        return false;
    }
}

__device__ void MC_MLTissue::reflect(MC_Path &path, float const step) {
/*    MC_Point origin_orig = path.origin();
    MC_Point tip_orig = path.tip();*/
    path = MC_Path(path.tip(), MC_Vector(path.direction().x(), path.direction().y(), -1 * path.direction().z()), step);
/*    printf("reflect() :\n\tReflection Event:\n\t\tIncident :\n\t\t\torigin = (%f, %f, %f)\n\t\t\ttip = (%f, %f, %f)\n\t\tReflected :\n\t\t\torigin = (%f, %f, %f)\n\t\t\ttip = (%f, %f, %f)\n",
           origin_orig.x(), origin_orig.y(), origin_orig.z(), tip_orig.x(), tip_orig.y(), tip_orig.z(),
           path.origin().x(), path.origin().y(), path.origin().z(), path.tip().x(), path.tip().y(), path.tip().z());*/
}

__device__ void MC_MLTissue::refract(MC_Path &path, float const step) {
/*    MC_Point origin_orig = path.origin();
    MC_Point tip_orig = path.tip();*/
    float ni = getLayer(whichLayer(path.origin())).n();
    float nt = getLayer(whichLayer(path.tip())).n();
    float n = ni / nt;
    float cosI = -1 * MCMath::dot(_normal, path.direction());
    float sinT2 = n * n * (1.f - cosI * cosI);
    if (sinT2 > 1.f) printf("Error in refraction: sinT2 > 1.f\n");
    float cosT = sqrt(1.f - sinT2);
    MC_Vector newDirection = path.direction() * n + _normal * (n * cosT - cosT);
    path = MC_Path(path.tip(), newDirection, step);
/*    printf("refract() :\n\tRefraction Event:\n\t\tIncident :\n\t\t\torigin = (%f, %f, %f)\n\t\t\ttip = (%f, %f, %f)\n\t\tRefracted :\n\t\t\torigin = (%f, %f, %f)\n\t\t\ttip = (%f, %f, %f)\n",
           origin_orig.x(), origin_orig.y(), origin_orig.z(), tip_orig.x(), tip_orig.y(), tip_orig.z(),
           path.origin().x(), path.origin().y(), path.origin().z(), path.tip().x(), path.tip().y(), path.tip().z());*/
}

MC_MLTissue::MC_MLTissue() = default;

