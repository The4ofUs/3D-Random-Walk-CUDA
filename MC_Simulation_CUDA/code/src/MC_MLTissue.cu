#include "hip/hip_runtime.h"
//
// Created by mustafa on 6/4/20.
//

#include <stdexcept>
#include "../headers/MC_MLTissue.cuh"
#include "../headers/MC_Math.cuh"

__host__ MC_MLTissue::MC_MLTissue(float const radius, MC_Point const c0, MC_Point const c1,
                                  std::vector<float> const &absorpCoeffs,
                                  std::vector<float> const &scatterCoeffs) {
    if (radius > 0 && absorpCoeffs.size() == scatterCoeffs.size()) {
        _radius = radius;
        _interface = c0;
        _remote = c1;
        _normal = MCMath::normalized(MC_Vector(c0, c1));
        _size = absorpCoeffs.size();
        _thickness = MCMath::absDistance(_interface, _remote);
        _portion = _thickness / (float) _size;
        for (int i = 0; i < _size; i++) {
            // This method of population the array with undefined Points is reckless and should be modified
            MC_Point interface =
                    _interface + _normal * ((float) i * _thickness / (float) _size);
            MC_Point remote =
                    _interface + _normal * ((float) (i + 1) * _thickness / (float) _size);
            _layers[i] = MC_Tissue(radius, interface, remote, absorpCoeffs[i],
                                   scatterCoeffs[i]);
        }
    } else { throw std::invalid_argument("MC_MLTissue::MC_MLTissue : Illegal Argument!"); }
}

__device__ void MC_MLTissue::attenuate(MC_Photon &photon) {
    MC_Tissue t = whichLayer(photon.position());
    t.attenuate(photon);
}

__device__ MC_Tissue MC_MLTissue::whichLayer(MC_Point const position) {
    /**
     * P : Current position of the photon
     * R : Projection of P on the tissue interface
     * Q : Point on the tissue interface ( Center of the interface for simplicity )
     * N : Normal to the interface
     * V : Vector from Q to P
     */
    MC_Point P = position;
    MC_Point Q = _interface;
    MC_Vector N = _normal;
    MC_Vector V = MC_Vector(Q, P);
    float d = abs(MCMath::dot(V, N));
    int index = (int) round((d / _thickness) * (float) _size);
    MC_Tissue currentTissue = _layers[index];
    return currentTissue;
}

__device__ bool MC_MLTissue::escaped(MC_Point const position) {
    MC_Point A = position;
    MC_Point B = _interface;
    MC_Vector C = _normal;
    float t = MCMath::dot(C, (A - B)) / MCMath::norm(C) * MCMath::norm(C);
    MC_Point P = B + C * t;
    float d = MCMath::absDistance(A, P);
    if (d > _radius) { return true; }
    float D = MCMath::norm(_remote - _interface);
    float E = MCMath::norm(P - _interface);
    if (E > D) { return true; }
    E = MCMath::norm((P - _remote));
    return E > D;
}

__device__ int MC_MLTissue::size() const {
    return _size;
}

__device__ float MC_MLTissue::coefficient(MC_Point position) { return whichLayer(position).attenuationCoefficient(); }

__device__ bool MC_MLTissue::isCrossing(MC_Path path) {
    /*
     * Distance of ray origin to the interface plane
     */
    float d1 = MCMath::point2PlaneDist(path.origin(), _interface, _normal);
    /*
     * Ignore the photons that already lay exactly on the boundary, it's highly likely that they were moved there by the
     * algorithm itself
     */
    if (fmod(d1, _portion) == 0.f) {
        return false;
    }
    /*
     * Distance of ray tip to the interface plane
     */
    float d2 = MCMath::point2PlaneDist(path.tip(), _interface, _normal);
    /*
     * Which portion the origin lies in
     */
    int q1 = (int) ((d1 / _thickness) * (float) _size);
    /*
     * Which portion the tip lies in
     */
    int q2 = (int) ((d2 / _thickness) * (float) _size);
    return q1 != q2;
}

/*
 * Calculate the point of intersection between the path and layer boundary
 */
__device__ void MC_MLTissue::updatePath(MC_Path& path) {
    /*
     * First we need to get a point on the boundary, however we need to determine some parameters first:
     * 1) Which direction did the crossing happen
     * 2) at which boundary
     */
    MC_Point coord = _interface * (1 - ((float) whichBoundary(path) * _portion));
    float d = MCMath::dot(_normal, coord);
    float t = (d - MCMath::dot(_normal, path.origin())) / MCMath::dot(_normal, path.direction());
    path.setTip(path.origin() + path.direction() * t);
}

__device__ int MC_MLTissue::whichBoundary(MC_Path path) {
    Direction direction;
    float d1 = MCMath::point2PlaneDist(path.origin(), _interface, _normal);
    float d2 = MCMath::point2PlaneDist(path.tip(), _interface, _normal);
    if ((d2 - d1) > 0) direction = DOWN;
    else direction = UP;
    int layerIdx = (int) floor(d1 / _portion);
    if (direction == DOWN) return layerIdx + 1;
    else return layerIdx;
}


MC_MLTissue::MC_MLTissue() = default;

