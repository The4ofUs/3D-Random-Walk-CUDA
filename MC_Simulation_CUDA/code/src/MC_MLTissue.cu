#include "hip/hip_runtime.h"
//
// Created by mustafa on 6/4/20.
//

#include <stdexcept>
#include "../headers/MC_MLTissue.cuh"
#include "../headers/MC_Math.cuh"

__host__ MC_MLTissue::MC_MLTissue(float const radius, MC_Point const c0, MC_Point const c1,
                                  std::vector<float> const &absorptionCoefficients,
                                  std::vector<float> const &scatteringCoefficients) {
    if (radius > 0 && absorptionCoefficients.size() == scatteringCoefficients.size()) {
        this->_radius = radius;
        this->_interface = c0;
        this->_remote = c1;
        this->_normal = MCMath::normalized(MC_Vector(c0, c1));
        this->_size = absorptionCoefficients.size();
        this->_thickness = MCMath::absDistance(this->_interface, this->_remote);
        for (int i = 0; i < this->_size; i++) {
            // This method of population the array with undefined Points is reckless and should be modified
            MC_Point interface =
                    this->_interface + this->_normal * ((float) i * this->_thickness / (float) this->_size);
            MC_Point remote =
                    this->_interface + this->_normal * ((float) (i + 1) * this->_thickness / (float) this->_size);
            this->_layers[i] = MC_Tissue(radius, interface, remote, absorptionCoefficients[i], scatteringCoefficients[i]);
        }
    } else { throw std::invalid_argument("MC_MLTissue::MC_MLTissue : Illegal Argument!"); }
}

__host__ void MC_MLTissue::verbose() {
    printf("Tissue Radius : %f\nTissue Interface side position : (%f,%f,%f)\nTissue Remote side position : (%f,%f,%f)\nThickness : %f\nTissue Normal Vector : (%f,%f,%f)\nNumber of layers : %d\n\n--- Layers Properties ---",
           this->_radius, this->_interface.x(), this->_interface.y(), this->_interface.z(), this->_remote.x(),
           this->_remote.y(),
           this->_remote.z(), this->_thickness, this->_normal.x(), this->_normal.y(), this->_normal.z(), this->_size);
    for (int i = 0; i < this->_size; i++) {
        MC_Tissue current = this->_layers[i];
        printf("\nLayer #%d\nRadius : %f\nInterface : (%f,%f,%f)\nRemote : (%f,%f,%f)\nThickness : %f\nAbsorption Coefficient : %f\nScattering Coefficient : %f\nAttenuation Coefficient : %f\n",
               i, current.radius(), current.interface().x(), current.interface().y(), current.interface().z(),
               current.remote().x(),
               current.remote().y(), current.remote().z(), current.thickness(),
               current.absorption(), current.scattering(), current.attenuationCoefficient());
    }
}

__device__ void MC_MLTissue::attenuate(MC_Photon &photon) {
    MC_Tissue t = whichLayer(photon.position());
    t.attenuate(photon);
}

__device__ MC_Tissue MC_MLTissue::whichLayer(MC_Point const position) {
    /**
     * P : Current position of the photon
     * R : Projection of P on the tissue interface
     * Q : Point on the tissue interface ( Center of the interface for simplicity )
     * N : Normal to the interface
     * V : Vector from Q to P
     */
    MC_Point P = position;
    MC_Point Q = this->_interface;
    MC_Vector N = this->_normal;
    MC_Vector V = MC_Vector(Q, P);
    float d = abs(MCMath::dot(V, N));
    int index = (int) round((d / this->_thickness) * (float) this->_size);
    MC_Tissue currentTissue = this->_layers[index];
    return currentTissue;
}

__device__ bool MC_MLTissue::escaped(MC_Point const position) {
    MC_Point A = position;
    MC_Point B = this->_interface;
    MC_Vector C = this->_normal;
    float t =
            MCMath::dot(C, (A - B)) / MCMath::norm(C) * MCMath::norm(C);
    MC_Point P = B + C * t;
    float d = MCMath::absDistance(A, P);
    if (d > this->_radius) { return true; }
    float D = MCMath::norm(this->_remote - this->_interface);
    float E = MCMath::norm(P - this->_interface);
    if (E > D) { return true; }
    E = MCMath::norm((P - this->_remote));
    return E > D;
}

__device__ int MC_MLTissue::size() const {
    return _size;
}

__device__ float MC_MLTissue::attenuationCoefficient(MC_Point const position) {
    MC_Tissue t = whichLayer(position);
    return t.attenuationCoefficient();
}

MC_MLTissue::MC_MLTissue() = default;

