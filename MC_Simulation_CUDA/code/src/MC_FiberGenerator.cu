#include "hip/hip_runtime.h"
//
// Created by mustafa on 6/3/20.
//

#include <stdexcept>

#include "../headers/MC_FiberGenerator.cuh"
#include "../headers/MC_Math.cuh"


__host__ MC_FiberGenerator::MC_FiberGenerator(float const radius, MC_Point const center, MC_Vector const lookAt) {
    if (radius > 0) {
        _radius = radius;
        _center = center;
        _lookAt = MCMath::normalized(lookAt);
    } else {
        throw std::invalid_argument("MC_FiberGenerator::MC_FiberGenerator : Illegal Argument!");
    }
}

__device__ MC_Point MC_FiberGenerator::center() { return _center; }

__device__ MC_Vector MC_FiberGenerator::lookAt() { return _lookAt; }

__device__ bool MC_FiberGenerator::isHit(MC_Path &path) {
    float relative_distance = MCMath::absDistance(path.tip(), _center);
    float rayTipAbsDistance = MCMath::norm(path.tip());
    float rayOriginAbsDistance = MCMath::norm(path.origin());
    float detectorAbsDistance = MCMath::norm(_center);
    if (rayTipAbsDistance >= detectorAbsDistance && rayOriginAbsDistance <= detectorAbsDistance &&
        relative_distance < detectorAbsDistance) {
        MC_Point point = calculateIntersectionPoint(path);
        float d_c = MCMath::absDistance(point, _center);
        if (d_c <= _radius) {
            path.setTip(point);
            return true;
        }
    }
    return false;
}


__device__ MC_Point MC_FiberGenerator::calculateIntersectionPoint(MC_Path const path) {
    MC_Point A = path.origin();
    MC_Vector B = path.direction();
    MC_Vector V = MC_Vector(A, _center);
    float t = MCMath::dot(V, _lookAt) / MCMath::dot(B, _lookAt);
    return A + B * t;
}

MC_FiberGenerator::MC_FiberGenerator() =
default;
