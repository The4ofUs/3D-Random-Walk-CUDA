#include "hip/hip_runtime.h"
//
// Created by mustafa on 6/3/20.
//

#include <stdexcept>

#include "../headers/MC_Detector.cuh"
#include "../headers/MC_Math.cuh"


__host__ MC_Detector::MC_Detector(float const radius, MC_Point const center, MC_Vector const lookAt) {
    if (radius > 0) {
        this->_radius = radius;
        this->_center = center;
        this->_lookAt = MCMath::normalized(lookAt);
    } else {
        throw std::invalid_argument("MC_Detector::MC_Detector : Illegal Argument!");
    }
}

__device__ MC_Point MC_Detector::center() { return this->_center; }

__device__ MC_Vector MC_Detector::lookAt() { return this->_lookAt; }

__device__ bool MC_Detector::isHit(MC_Photon &photon, MC_Path const path) {
    float relative_distance = MCMath::absDistance(path.tip(), this->_center);
    float rayTipAbsDistance = MCMath::norm(path.tip());
    float rayOriginAbsDistance = MCMath::norm(path.origin());
    float detectorAbsDistance = MCMath::norm(this->_center);
    if (rayTipAbsDistance >= detectorAbsDistance && rayOriginAbsDistance <= detectorAbsDistance &&
        relative_distance < detectorAbsDistance) {
        MC_Point point = calculateIntersectionPoint(path);
        float d_c = MCMath::absDistance(point, this->_center);
        if (d_c <= this->_radius) {
            photon.setPosition(point);
            return true;
        } else
            return false;
    } else
        return false;
}

__device__ MC_Point MC_Detector::calculateIntersectionPoint(MC_Path const path) {
    MC_Point A = path.origin();
    MC_Vector B = path.direction();
    MC_Vector V = MC_Vector(A, this->_center);
    float t = MCMath::dot(V, this->_lookAt) / MCMath::dot(B, this->_lookAt);
    return A + B * t;
}

MC_Detector::MC_Detector() = default;
