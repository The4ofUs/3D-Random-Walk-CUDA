//
// Created by mostafa on ٢‏/٨‏/٢٠٢٠.
//

#include "../headers/MC_Simulation.cuh"
#include "../headers/MC_Kernels.cuh"
#include "../headers/MC_Helpers.cuh"


MC_Simulation::MC_Simulation(){
    detector = MC_Detector(DETECTOR_RADIUS, DETECTOR_POSITION, DETECTOR_LOOK_AT);
    mlTissue = MC_MLTissue(TISSUE_RADIUS, TISSUE_CENTER_1, TISSUE_CENTER_2, COEFFICIENTS1, COEFFICIENTS2);
}

void MC_Simulation::start() {
    int blocksCount = NUMBER_OF_PHOTONS + THREADS_PER_BLOCK - 1 / THREADS_PER_BLOCK;
    hiprandState_t *states;
    hipMalloc((void **) &states, NUMBER_OF_PHOTONS * sizeof(hiprandState_t));
    auto *_cpuPhotons = (MC_Photon *) malloc(sizeof(MC_Photon) * NUMBER_OF_PHOTONS);
    MC_Photon *_gpuPhotons = nullptr;
    hipMalloc((void **) &_gpuPhotons, NUMBER_OF_PHOTONS * sizeof(MC_Photon));
    MCKernels::simulate<<<blocksCount, THREADS_PER_BLOCK>>>(time(nullptr), states, _gpuPhotons, detector, rng, mlTissue, NUMBER_OF_PHOTONS);
    hipMemcpy(_cpuPhotons, _gpuPhotons, NUMBER_OF_PHOTONS * sizeof(MC_Photon), hipMemcpyDeviceToHost);
    MCHelpers::streamOut(&_cpuPhotons[0], NUMBER_OF_PHOTONS);
    free(_cpuPhotons);
    hipFree(_gpuPhotons);
    hipFree(states);
}