//
// Created by mostafa on ٢‏/٨‏/٢٠٢٠.
//

#include "../headers/MC_Simulation.cuh"
#include "../headers/MC_Kernels.cuh"
#include "../headers/MC_Helpers.cuh"


MC_Simulation::MC_Simulation(){
    detector = MC_Detector(DETECTOR_RADIUS, DETECTOR_POSITION, DETECTOR_LOOK_AT);
    mlTissue = MC_MLTissue(TISSUE_RADIUS, TISSUE_CENTER_1, TISSUE_CENTER_2, COEFFICIENTS1, COEFFICIENTS2);
}

void MC_Simulation::start() {
    int blocksCount = NUMBER_OF_PHOTONS + THREADS_PER_BLOCK - 1 / THREADS_PER_BLOCK;
    hiprandState_t *states;
    hipMalloc((void **) &states, NUMBER_OF_PHOTONS * sizeof(hiprandState_t));
    auto *hostMemory = (MC_Photon *) malloc(sizeof(MC_Photon) * NUMBER_OF_PHOTONS);
    MC_Photon *deviceMemory = nullptr;
    hipMalloc((void **) &deviceMemory, NUMBER_OF_PHOTONS * sizeof(MC_Photon));
    MCKernels::simulate<<<blocksCount, THREADS_PER_BLOCK>>>(time(nullptr), states, deviceMemory, detector, mlTissue, NUMBER_OF_PHOTONS);
    hipMemcpy(hostMemory, deviceMemory, NUMBER_OF_PHOTONS * sizeof(MC_Photon), hipMemcpyDeviceToHost);
    MCHelpers::streamOut(&hostMemory[0], NUMBER_OF_PHOTONS);
    free(hostMemory);
    hipFree(deviceMemory);
    hipFree(states);
}