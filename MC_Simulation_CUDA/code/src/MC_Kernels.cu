#include "hip/hip_runtime.h"
//
// Created by mustafa on 6/3/20.
//

#include "../headers/MC_Kernels.cuh"
#include "../headers/MC_RandomWalk.cuh"

__global__ void MCKernels::simulate(unsigned int seed, hiprandState_t *states, MC_Photon *_gpuPhotons, MC_Detector const detector,
                    MC_RNG const rng,
                    MC_MLTissue tissue, int const n) {
    int idx = (int) (blockIdx.x * blockDim.x + threadIdx.x);
    if (idx < n) {
        hiprand_init(seed, idx, 0, &states[idx]);
        MC_Photon finalState = RandomWalk(states, idx, detector, rng, tissue);
        _gpuPhotons[idx] = finalState;
    }
}

