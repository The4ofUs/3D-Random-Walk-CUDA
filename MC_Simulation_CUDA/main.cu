#include <hiprand/hiprand_kernel.h>
#include <vector>
#include <QVector>
#include "Network/Client/Headers/ClientSocket.h"
#include "code/headers/MC_Simulation.cuh"
#include "code/headers/MC_Helpers.cuh"



int  Number_of_photons;
float fiber_generator_radius;
float tissue_radius;

std::vector<float> a_coeffiecints;
std::vector<float> s_coefficients;

MC_Point fiber_generator_poistion;
MC_Point tissue_center_1;
MC_Point tissue_center_2;

void dumpReceivedParameters(QVector<float> parameters);

QVector<MC_Photon> total_photons_sent;


int main() {

    ClientSocket* socket = new ClientSocket();
    QVector<float> parameters = socket->requestParameters();
    dumpReceivedParameters(parameters);
    socket->setBatchPhotons(Number_of_photons);
    MC_Simulation simulation = MC_Simulation(fiber_generator_radius, fiber_generator_poistion, tissue_radius, tissue_center_1, tissue_center_2, a_coeffiecints, s_coefficients);
    while(simulation._batchAvailability){
         simulation.start( Number_of_photons);
         total_photons_sent.append(simulation._totalPhotonsPerPatch);
    }
    MCHelpers::streamOut(total_photons_sent);
    return 0;
}


void dumpReceivedParameters(QVector<float> parameters){
    Number_of_photons = (int) parameters[0];
    fiber_generator_radius = parameters[1];
    fiber_generator_poistion = MC_Point(parameters[2],parameters[3],parameters[4]);
    tissue_radius = parameters[5];
    tissue_center_1 =  MC_Point(parameters[6], parameters[7], parameters[8]);
    tissue_center_2 = MC_Point(parameters[9],parameters[10],parameters[11]);
    int reminder = (parameters.size()-12)/2;
    for(int i = 12; i<=(11+reminder);i++){
        a_coeffiecints.push_back(parameters[i]);
    }
   for(int i = parameters.size()-1;i>=parameters.size()-reminder;i--){
       s_coefficients.push_back(parameters[i]);
   }
   std::reverse(s_coefficients.begin(),s_coefficients.end());
}

