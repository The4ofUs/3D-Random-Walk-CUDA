#include <hiprand/hiprand_kernel.h>
#include <array>
#include <vector>
#include <QVector>
#include "code/headers/MC_Photon.cuh"
#include "code/headers/MC_Detector.cuh"
#include "code/headers/MC_RNG.cuh"
#include "code/headers/MC_Tissue.cuh"
#include "code/headers/MC_Kernels.cuh"
#include "code/headers/MC_Helpers.cuh"
#include "code/headers/MC_MLTissue.cuh"
#include "Network/Client/Headers/ClientSocket.h"
#define THREADS_PER_BLOCK 1024
#define DETECTOR_LOOK_AT MC_Vector(0.f, 0.f, -1.f)
int  Number_of_photons;
float Detector_Radius;
float Tissue_Radius;
float Tissue_Absorption_Coefficient;
float Tissue_Scattering_Coefficient;
std::vector<float> coefficients1;
std::vector<float> coefficients2;
MC_Point Detector_Position;
MC_Point Tissue_Center_1;
MC_Point Tissue_Center_2;
void dumpReceivedParameters(QVector<float> parameters);


int main() {
    ClientSocket* socket = new ClientSocket();
    QVector<float> parameters = socket->requestParameters();
    dumpReceivedParameters(parameters);
    socket->setBatchPhotons(Number_of_photons);
    MC_RNG rng;
    MC_Detector detector = MC_Detector(Detector_Radius, Detector_Position, DETECTOR_LOOK_AT);
    //std::vector<float> coefficients1 = {1.f, 6.f, 4.f, 15};
    //std::vector<float> coefficients2 = {100.f, 30.f, 12.f, 44.f};
    MC_MLTissue mlTissue = MC_MLTissue(Tissue_Radius, Tissue_Center_1, Tissue_Center_2, coefficients1, coefficients2);
    //MC_Tissue tissue = MC_Tissue(Tissue_Radius,Tissue_Center_1,Tissue_Center_2,Tissue_Absorption_Coefficient,Tissue_Scattering_Coefficient);
    QVector<MC_Photon> totalPhotonsSent;

    while(socket->getBatchAvailability()){
    	int nBlocks = Number_of_photons + THREADS_PER_BLOCK - 1 / THREADS_PER_BLOCK;
	hiprandState_t *states;
    	hipMalloc((void **) &states, Number_of_photons * sizeof(hiprandState_t));
	auto *_cpuPhotons = (MC_Photon *) malloc(sizeof(MC_Photon) * Number_of_photons);
        MC_Photon *_gpuPhotons = nullptr;
        hipMalloc((void **) &_gpuPhotons, Number_of_photons * sizeof(MC_Photon));
        MCKernels::simulate<<<nBlocks, THREADS_PER_BLOCK >>>
                                                           (time(nullptr), states, _gpuPhotons, detector, rng, mlTissue, Number_of_photons);
        hipMemcpy(_cpuPhotons, _gpuPhotons, Number_of_photons * sizeof(MC_Photon), hipMemcpyDeviceToHost);
        socket->sendResults(&_cpuPhotons[0]);
        //MCHelpers::streamOut(&_cpuPhotons[0], Number_of_photons);
        totalPhotonsSent.append(socket->getSentPhotons());
        Number_of_photons = socket->requestNewBatch();
        free(_cpuPhotons);
        hipFree(_gpuPhotons);
     	hipFree(states);
    }
 
    MCHelpers::streamOut(totalPhotonsSent);
    return 0;
}


void dumpReceivedParameters(QVector<float> parameters){
    Number_of_photons = (int) parameters[0];
    Detector_Radius  = parameters[1];
    Detector_Position = MC_Point(parameters[2],parameters[3],parameters[4]);
    Tissue_Radius = parameters[5];
    Tissue_Center_1 =  MC_Point(parameters[6], parameters[7], parameters[8]);
    Tissue_Center_2 = MC_Point(parameters[9],parameters[10],parameters[11]);
    int reminder = (parameters.size()-12)/2;
    for(int i = 12; i<=(11+reminder);i++){
        coefficients1.push_back(parameters[i]);
    }
   for(int i = parameters.size()-1;i>=parameters.size()-reminder;i--){
       coefficients2.push_back(parameters[i]);
   }
   std::reverse(coefficients2.begin(),coefficients2.end());
}
