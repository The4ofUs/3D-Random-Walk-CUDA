#include <hiprand/hiprand_kernel.h>
#include "code/headers/MC_Photon.cuh"
#include "code/headers/MC_Detector.cuh"
#include "code/headers/MC_RNG.cuh"
#include "code/headers/MC_Tissue.cuh"
#include "code/headers/MC_Kernels.cuh"
#include "code/headers/MC_Helpers.cuh"
#include "code/headers/MC_MLTissue.cuh"

#define NUMBER_OF_PHOTONS 1000
#define THREADS_PER_BLOCK 1024
#define DETECTOR_RADIUS 10.f
#define DETECTOR_POSITION MC_Point(0.f, 0.f, 10.f)
#define DETECTOR_LOOK_AT MC_Vector(0.f, 0.f, -1.f)
#define TISSUE_RADIUS 100.f
#define TISSUE_ABSORPTION_COEFFICIENT 1.f
#define TISSUE_SCATTERING_COEFFICIENT 100.f
#define TISSUE_CENTER_1 MC_Point(0.f, 0.f, 10.f)
#define TISSUE_CENTER_2 MC_Point(0.f, 0.f, -10.f)


int main() {
    printf("main(): Starting.");
    int nBlocks = NUMBER_OF_PHOTONS + THREADS_PER_BLOCK - 1 / THREADS_PER_BLOCK;
    hiprandState_t *states;
    hipMalloc((void **) &states, NUMBER_OF_PHOTONS * sizeof(hiprandState_t));
    auto *_cpuPhotons = (MC_Photon *) malloc(sizeof(MC_Photon) * NUMBER_OF_PHOTONS);
    MC_Photon *_gpuPhotons = nullptr;
    hipMalloc((void **) &_gpuPhotons, NUMBER_OF_PHOTONS * sizeof(MC_Photon));
    MC_RNG rng;
    MC_Detector detector = MC_Detector(DETECTOR_RADIUS, DETECTOR_POSITION, DETECTOR_LOOK_AT);
    std::vector<float> coefficients1 = {1.f, 6.f, 4.f, 15};
    std::vector<float> coefficients2 = {100.f, 30.f, 12.f, 44.f};
    MC_MLTissue mlTissue = MC_MLTissue(TISSUE_RADIUS, TISSUE_CENTER_1, TISSUE_CENTER_2, coefficients1, coefficients2);
    MC_Tissue tissue = MC_Tissue(TISSUE_RADIUS,TISSUE_CENTER_1,TISSUE_CENTER_2,TISSUE_ABSORPTION_COEFFICIENT,TISSUE_SCATTERING_COEFFICIENT);
    MCKernels::simulate <<<nBlocks, THREADS_PER_BLOCK>>> (time(nullptr), states, _gpuPhotons, detector, rng, tissue, NUMBER_OF_PHOTONS);
    hipMemcpy(_cpuPhotons, _gpuPhotons, NUMBER_OF_PHOTONS * sizeof(MC_Photon), hipMemcpyDeviceToHost);
    MCHelpers::streamOut(&_cpuPhotons[0], NUMBER_OF_PHOTONS);
    MCHelpers::endMsg(NUMBER_OF_PHOTONS,DETECTOR_RADIUS,DETECTOR_POSITION,DETECTOR_LOOK_AT,TISSUE_RADIUS,TISSUE_ABSORPTION_COEFFICIENT,TISSUE_SCATTERING_COEFFICIENT,TISSUE_CENTER_1,TISSUE_CENTER_2);
    free(_cpuPhotons);
    hipFree(_gpuPhotons);
    hipFree(states);
    return 0;
}


