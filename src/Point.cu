#include "hip/hip_runtime.h"
#include "Point.h"


__device__ __host__ 
Point::Point(float x, float y, float z){
    setCoordinates(x, y, z);
}

__device__ __host__ 
Point::Point(){
    setCoordinates(0.f, 0.f, 0.f);
}

__device__
 void Point::setCoordinates(float x, float y, float z)
{
    this->_x = x;
    this->_y = y;
    this->_z = z;
}
__device__ float Point::x() const { return this->_x; }

__device__  float Point::y() const { return this->_y; }

__device__  float Point::z() const { return this->_z; }

__host__ __device__ float Point::getAbsDistance(){
    float absDistance = sqrtf(this->_x*this->_x + this->_y*this->_y + this->_z*this->_z);
    return absDistance;
}

__host__ __device__ float Point::getAbsDistance(Point relative){
    Point relativeV = Point(relative.x() - this->_x, relative.y() - this->_y, relative.z() - this->_z);
    float absDistance = sqrtf(relativeV.x()*relativeV.x()+ relativeV.y()*relativeV.y() + relativeV.z()*relativeV.z());
    return absDistance;
}

__device__ __host__
    Point Point::operator - (Point const &other) { 
        float result_x = this->_x - other.x();
        float result_y = this->_y - other.y();
        float result_z = this->_z - other.z();
        return Point(result_x, result_y, result_z); 
    }

__device__ __host__
    Point Point::operator + (Point const &other) { 
        float result_x = this->_x + other.x();
        float result_y = this->_y + other.y();
        float result_z = this->_z + other.z();
        return Point(result_x, result_y, result_z); 
    }

    __device__ __host__
    Point Point::operator * (float const &other) { 
        float result_x = this->_x * other;
        float result_y = this->_y * other;
        float result_z = this->_z * other;
        return Point(result_x, result_y, result_z); 
    }


