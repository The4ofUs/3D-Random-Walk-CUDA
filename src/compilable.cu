
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>

#define NUMBER_OF_PHOTONS 1000
#define THREADS_PER_BLOCK 1024  
#define BOUNDARY_RADIUS 10.0  


/* ----------- Point ----------- */
class Point{

public:
    __device__ __host__ Point(float x, float y, float z){
        setCoordinates(x, y, z);
    }
    __device__ __host__ Point(){
        setCoordinates(0.f, 0.f, 0.f);
    }
    __device__ __host__ void setCoordinates(float x, float y, float z)
    {
        this->_x = x;
        this->_y = y;
        this->_z = z;
    }
    __device__ __host__ float x() const { return this->_x; }
    __device__ __host__ float y() const { return this->_y; }
    __device__ __host__ float z() const { return this->_z; }
    __device__ __host__ Point operator - (Point const &other) { 
        float result_x = this->_x - other.x();
        float result_y = this->_y - other.y();
        float result_z = this->_z - other.z();
        return Point(result_x, result_y, result_z); 
    }
    __device__ __host__ Point operator + (Point const &other) { 
        float result_x = this->_x + other.x();
        float result_y = this->_y + other.y();
        float result_z = this->_z + other.z();
        return Point(result_x, result_y, result_z); 
    }
    __device__ __host__ Point operator * (float const &other) { 
        float result_x = this->_x * other;
        float result_y = this->_y * other;
        float result_z = this->_z * other;
        return Point(result_x, result_y, result_z); 
    }
private:
    float _x;
    float _y;
    float _z;
};

/* ----------- RNG ----------- */
class RNG{
private:
    __device__ float generate( hiprandState* globalState, int i) {
        hiprandState localState = globalState[i];
        float random = hiprand_uniform( &localState );
        globalState[i] = localState;
        return random;
    }
public:
    __device__ float getRandomStep( hiprandState* globalState , int i) { 
        float step = 0.f;       // Intialize for step value
        step = generate (globalState, i);
        return step;
    } 
    __device__ Point getRandomPoint( hiprandState* globalState , int i){
        float u = generate (globalState , i);
        float v = generate (globalState, i);
        float theta = 2 * M_PI * u;
        float phi = acos(1 - 2 * v);
        float x = sin(phi) * cos(theta);
        float y = sin(phi) * sin(theta);
        float z = cos(phi);
        return Point(x,y,z);
    }
};

/* ----------- Ray ----------- */
class Ray{
private:
    Point _prevPos;
    Point _currentPos;
    Point _direction;
    float _step;
    __device__ __host__ void updateRayState(Point direction, float step){
        this->_prevPos = this->_currentPos;
        this->_direction = direction;
        this->_step = step;
    }
public:
    __device__ __host__ Ray(Point startingPoint, Point direction){
        this->_currentPos.setCoordinates(startingPoint.x(), startingPoint.y(), startingPoint.z());
        this->_direction.setCoordinates(direction.x(), direction.y(), direction.z());
    }
    __device__ __host__ void setDirection(Point direction) { this->_direction.setCoordinates(direction.x(), direction.y(), direction.z()); }
    __device__ __host__ void setStep(float step) { this->_step = step; }
    __device__ __host__ Point getCurrentPos() const { return this->_currentPos; }
    __device__ __host__ Point getDirection() const { return this->_direction; }
    __device__ __host__ Point getPrevPos() const { return this->_prevPos; }
    __device__ __host__ float getStep() const { return this->_step; }
    __device__ __host__ void move(Point direction, float step)
    {
        updateRayState(direction, step);
        this->_currentPos = this->_currentPos + (direction * step);
    }
};

/* ----------- Boundary ----------- */
class Boundary{
private:
    float _radius;
    Point _center;
    __device__ __host__ float dot(Point point1, Point point2){return point1.x()*point2.x() + point1.y()*point2.y() + point1.z()*point2.z();}
public:
    __device__ __host__ Boundary(float r, Point c){
        _radius = r;
        _center = c;
    }
    __device__ __host__ void setRadius(float r){_radius = r;}
    __device__ __host__ float getRadius() const {return _radius;}
    __device__ __host__ void setCenter(Point c){_center = c;}
    __device__ __host__ Point getCenter() const {return _center;}
    __device__ __host__ bool isHit(Ray ray){
        float absDistance = (float) sqrtf((float) powf(ray.getCurrentPos().x(),2) + (float) powf(ray.getCurrentPos().y(),2) + (float) powf(ray.getCurrentPos().z(),2));
        if(absDistance >= _radius){ return true;} else { return false;}
    }
    __device__ __host__ Point getIntersectionPoint(Ray ray){
            Point A = ray.getPrevPos();
            Point B = ray.getDirection();
            Point S = A + _center;
            Point A_C = A - _center;
            float a = dot(B, B);
            float b = 2.0 * dot(B, A_C);
            float c = dot(A_C, A_C) - _radius*_radius;
            float discriminant = b*b - 4*a*c;
            float t1 = (-b + sqrtf(discriminant)) / (2.0*a);
            float t2 = (-b - sqrtf(discriminant)) / (2.0*a);
            float t;
            if(t1 < 0){ t = t2;} else { t = t1;}
            return (A+B*t);
    }
};

/* ----------- RandomWalk ----------- */
__device__ Point randomWalk(hiprandState_t *states, int idx, Boundary boundary, RNG rng)
{
    Ray ray = Ray(Point(), Point());
    while (!boundary.isHit(ray)){ 
        ray.move(rng.getRandomPoint(states, idx), rng.getRandomStep(states, idx));
    }
    return boundary.getIntersectionPoint(ray);
}

/* ----------- streamOut ----------- */
void streamOut(Point* _cpuPoints)  
{
    FILE *output;
    output = fopen("output.csv", "w");
    for (int i = 0; i < NUMBER_OF_PHOTONS; i++)
    {
        fprintf(output, "%f,%f,%f\n", _cpuPoints[i].x(), _cpuPoints[i].y(), _cpuPoints[i].z());
    }
}

/* ----------- FinalPosition Kernel ----------- */
__global__ void finalPosition(unsigned int seed, hiprandState_t* states, Point* _gpuPoints, Boundary boundary, RNG rng, int n) {
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx < n){
    hiprand_init(seed, idx, 0, &states[idx]);
    Point finalPos = Point();
    finalPos = randomWalk(states, idx, boundary, rng);
    _gpuPoints[idx] = finalPos;
    }
}

/* ----------- Main ----------- */
int main() {
    int nBlocks = NUMBER_OF_PHOTONS/THREADS_PER_BLOCK + 1;
    hiprandState_t* states;
    hipMalloc((void**) &states, NUMBER_OF_PHOTONS * sizeof(hiprandState_t));
    // Allocate host memory for final positions
    Point * _cpuPoints= (Point*)malloc(sizeof(Point) * NUMBER_OF_PHOTONS);
    // Allocate device  memory for final positions
    Point* _gpuPoints = nullptr;
    hipMalloc((void**) &_gpuPoints, NUMBER_OF_PHOTONS * sizeof(Point));
    // Initialize the Boundary and the RandomNumberGenerator
    RNG rng;
    Boundary boundary = Boundary(BOUNDARY_RADIUS, Point());
    // Kernel Call
    finalPosition<<<nBlocks,THREADS_PER_BLOCK>>>(time(0), states , _gpuPoints, boundary, rng, NUMBER_OF_PHOTONS);
    // Copy device data to host memory to stream them out
    hipMemcpy(_cpuPoints, _gpuPoints, NUMBER_OF_PHOTONS* sizeof(Point), hipMemcpyDeviceToHost);
    streamOut (&_cpuPoints[0]);
    free(_cpuPoints);
    hipFree(_gpuPoints);
    return 0;
}

