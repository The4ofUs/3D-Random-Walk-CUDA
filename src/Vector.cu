#include "hip/hip_runtime.h"
#include "Vector.h"

    Vector::Vector(){
        this->setCoordinates(0.f,0.f,0.f);
    }

    Vector::Vector(Point point)
    {
        this->_x = point.x();
        this->_y = point.y();
        this->_z = point.z();
    }

    Vector::Vector(float x, float y, float z){
        this->_x = x;
        this->_y = y;
        this->_z = z;
    }

    Vector::Vector(Point point1, Point point2)
    {
        this->_x = point2.x() - point1.x();
        this->_y = point2.y() - point1.y();
        this->_z = point2.z() - point1.z();
    }

    __device__ __host__ float Vector::dot(Vector otherVector) { return this->x() * otherVector.x() + this->y() * otherVector.y() + this->z() * otherVector.z(); }

    __device__ __host__ Vector Vector::cross(Vector otherVector)
    {
        float X = this->y() * otherVector.z() - this->z() * otherVector.y();
        float Y = (-1) * this->x() * otherVector.z() + this->z() * otherVector.x();
        float Z = this->x() * otherVector.y() - this->y() * otherVector.x();
        return Point(X, Y, Z);
    }

    __device__ __host__ Vector Vector::normalize()
    {
        float norm = sqrtf((powf(this->x(), 2) + powf(this->y(), 2) + powf(this->z(), 2)));
        float Xhat = this->x() / norm;
        float Yhat = this->y() / norm;
        float Zhat = this->z() / norm;
        return Vector(Xhat,Yhat,Zhat);
    }
