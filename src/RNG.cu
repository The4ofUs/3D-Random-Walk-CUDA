#include "hip/hip_runtime.h"
#include "RNG.h"

__device__  float RNG::generate( hiprandState* globalState, int i) 
{
    hiprandState localState = globalState[i];
    float random = hiprand_uniform( &localState );
    globalState[i] = localState;
    return random;
}

__device__   float RNG::getRandomStep( hiprandState* globalState , int i) { 
    float step = 0.f;       // Intialize for step value
    step = generate (globalState, i);
    return step;
 } 

__device__  Point RNG::getRandomPoint( hiprandState* globalState , int i)
{

    float u = generate (globalState , i);
    float v = generate (globalState, i);
    
    float theta = 2 * M_PI * u;
    float phi = acos(1 - 2 * v);

    // Transforming into the cartesian space
    float x = sin(phi) * cos(theta);
    float y = sin(phi) * sin(theta);
    float z = cos(phi);

    return Point(x,y,z);
}