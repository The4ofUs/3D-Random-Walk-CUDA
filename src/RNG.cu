#include "hip/hip_runtime.h"
#include "RNG.h"

__device__
float RNG::generate( hiprandState* globalState, int i)
{
    hiprandState localState = globalState[i];
    float random = hiprand_uniform( &localState );
    globalState[i] = localState;
    return random;
}

__device__
float RNG::getRandomStep( hiprandState* globalState , int i)
{
    // Intialize for step value
    float step = 0.f;
    step = generate (globalState, i);
    return step;
 } 


__device__ Point RNG::getRandomPoint(hiprandState* globalState , int i)
{
    // Instance of the Point struct to return with the random coordinates.
    Point point;

    float u = generate (globalState , i);
    float v = generate (globalState, i);
    
    float theta = 2 * M_PI * u;
    float phi = acos(1 - 2 * v);

    // Transforming into the cartesian space
    float x = sin(phi) * cos(theta);
    float y = sin(phi) * sin(theta);
    float z = cos(phi);

    point.setCoordinates(x, y, z);
  
    return point;
}
