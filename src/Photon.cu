#include "Photon.h"
#include "RNG.h"
#define WEIGHT_THRESHOLD 0.0001f
#define ROULETTE_CHANCE 0.1f

__device__ Photon::Photon()
{
    this->_weight = 1.f;
}

__device__ Photon::Photon(Point rayOrigin, Vector rayDirection)
{
    this->_origin = rayOrigin;
    this->_position = rayOrigin;
    this->_direction = rayDirection;
    this->_weight = 1.f;
}

__device__ void Photon::roulette(RNG rng, hiprandState *globalState, int i)
{
    if (rng.generate(globalState, i) < ROULETTE_CHANCE)
    {
        this->terminate();
    }
    else
    {
        this->boost();
    }
}

__device__ float Photon::getWeight()
{
    return this->_weight;
}

__device__ void Photon::terminate()
{
    this->_weight = 0.f;
}
__device__ void Photon::boost()
{
    this->_weight = this->_weight / ROULETTE_CHANCE;
}
