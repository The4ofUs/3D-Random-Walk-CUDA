#include "Photon.h"
#include "RNG.h"
#define WEIGHT_THRESHOLD 0.0001f
#define ROULETTE_CHANCE 0.1f

__device__ Photon::Photon()
{
    this->_weight = 1.f;
}

__device__ Photon::Photon(Point rayOrigin, Vector rayDirection)
{
    this->_origin = rayOrigin;
    this->_position = rayOrigin;
    this->_direction = rayDirection;
    this->_weight = 1.f;
}

__device__ void Photon::roulette(RNG rng, hiprandState *globalState, int i)
{
    if (rng.generate(globalState, i) < ROULETTE_CHANCE)
    {
        this->terminate();
    }
    else
    {
        this->boost();
    }
}

__device__ float Photon::weight()
{
    return this->_weight;
}

__device__ __host__ Point Photon::position()
{
    return this->_position;
}

__device__ void Photon::setPosition(Point point)
{
    this->_position = point;
}

__device__ void Photon::terminate()
{
    this->_weight = 0.f;
}
__device__ void Photon::boost()
{
    this->_weight = this->_weight / ROULETTE_CHANCE;
}

__device__ void Photon::updateState(Vector direction)
{
    this->_origin = this->_tip;
    this->_direction = direction;
}

__device__ void Photon::move(Vector direction, float step) // The photon moves in the specified direction with the given step
{
    updateState(direction);
    this->_tip = this->_tip + (direction * step);
    this->_position = this->_tip;
}
