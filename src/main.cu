#include "hip/hip_runtime.h"
#include "RandomWalk.h"
#include "Detector.h"
#include "Vector.h"
#include "common.h"
#include "Photon.h"
#define NUMBER_OF_PHOTONS 100
#define THREADS_PER_BLOCK 1024
#define BOUNDARY_RADIUS 5.0

void streamOut(Photon *_cpuPhotons);

__global__ void finalState(unsigned int seed, hiprandState_t *states, Photon *_gpuPhotons, Detector detector, RNG rng, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        hiprand_init(seed, idx, 0, &states[idx]);
        Photon finalState = Photon();
        finalState = randomWalk(states, idx, detector, rng);
        _gpuPhotons[idx] = finalState;
    }
}

int main()
{
    int nBlocks = NUMBER_OF_PHOTONS / THREADS_PER_BLOCK + 1;
    hiprandState_t *states;
    hipMalloc((void **)&states, NUMBER_OF_PHOTONS * sizeof(hiprandState_t));
    // Allocate host memory for final positions
    Photon *_cpuPhotons = (Photon *)malloc(sizeof(Photon) * NUMBER_OF_PHOTONS);
    // Allocate device  memory for final positions
    Photon *_gpuPhotons = nullptr;
    hipMalloc((void **)&_gpuPhotons, NUMBER_OF_PHOTONS * sizeof(Photon));
    // Initialize the Boundary and the RandomNumberGenerator
    RNG rng;
    //Boundary boundary = Boundary(BOUNDARY_RADIUS, Point());
    Detector detector = Detector(5.f, Point(0.f, 1.f, 0.f), Vector(0.f, -1.f, 0.f));
    // Kernel Call
    //finalPosition<<<nBlocks,THREADS_PER_BLOCK>>>(time(0), states , _gpuPoints, boundary, rng, NUMBER_OF_PHOTONS);
    finalState<<<nBlocks, THREADS_PER_BLOCK>>>(time(0), states, _gpuPhotons, detector, rng, NUMBER_OF_PHOTONS);
    // Copy device data to host memory to stream them out
    hipMemcpy(_cpuPhotons, _gpuPhotons, NUMBER_OF_PHOTONS * sizeof(Photon), hipMemcpyDeviceToHost);
    streamOut(&_cpuPhotons[0]);
    free(_cpuPhotons);
    hipFree(_gpuPhotons);
    return 0;
}

void streamOut(Photon *_cpuPhotons)
{
    FILE *output;
    output = fopen("output.csv", "w");
    for (int i = 0; i < NUMBER_OF_PHOTONS; i++)
    {
        // Streaming out my output in a log file
        fprintf(output, "%f,%f,%f\n", _cpuPhotons[i].position().x(), _cpuPhotons[i].position().y(), _cpuPhotons[i].position().z());
    }
}
