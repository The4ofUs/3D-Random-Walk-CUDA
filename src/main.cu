#include "hip/hip_runtime.h"
#include "RandomWalk.h"
#include "Detector.h"
#include "Vector.h"
#define NUMBER_OF_PHOTONS 100
#define THREADS_PER_BLOCK 1024
#define BOUNDARY_RADIUS 10.0


void streamOut(Point* _cpuPoints);

__global__ void finalPosition(unsigned int seed, hiprandState_t* states, Point* _gpuPoints, Detector detector, RNG rng, int n) {
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx < n){
    hiprand_init(seed, idx, 0, &states[idx]);
    Point finalPos = Point();
    finalPos = randomWalk(states, idx, detector, rng);
    _gpuPoints[idx] = finalPos;
    }
}

  int main() {
    int nBlocks = NUMBER_OF_PHOTONS/THREADS_PER_BLOCK + 1;
    hiprandState_t* states;
    hipMalloc((void**) &states, NUMBER_OF_PHOTONS * sizeof(hiprandState_t));
    // Allocate host memory for final positions
    Point * _cpuPoints= (Point*)malloc(sizeof(Point) * NUMBER_OF_PHOTONS);
    // Allocate device  memory for final positions
    Point* _gpuPoints = nullptr;
    hipMalloc((void**) &_gpuPoints, NUMBER_OF_PHOTONS * sizeof(Point));
    // Initialize the Boundary and the RandomNumberGenerator
    RNG rng;
    //Boundary boundary = Boundary(BOUNDARY_RADIUS, Point());
    Detector detector = Detector(10.f,Point(0.f,0.f,5.f), Vector(0.f,0.f,-1.f));
    // Kernel Call
    //finalPosition<<<nBlocks,THREADS_PER_BLOCK>>>(time(0), states , _gpuPoints, boundary, rng, NUMBER_OF_PHOTONS);
    finalPosition<<<nBlocks,THREADS_PER_BLOCK>>>(time(0), states , _gpuPoints, detector, rng, NUMBER_OF_PHOTONS);
    // Copy device data to host memory to stream them out
    hipMemcpy(_cpuPoints, _gpuPoints, NUMBER_OF_PHOTONS* sizeof(Point), hipMemcpyDeviceToHost);
    streamOut (&_cpuPoints[0]);
    free(_cpuPoints);
    hipFree(_gpuPoints);
    return 0;
}

void streamOut(Point* _cpuPoints)  
{
    FILE *output;
    output = fopen("output.csv", "w");
    for (int i = 0; i < NUMBER_OF_PHOTONS; i++)
    {
        // Streaming out my output in a log file
        fprintf(output, "%f,%f,%f\n", _cpuPoints[i].x(), _cpuPoints[i].y(), _cpuPoints[i].z());
    }
}
