#include "hip/hip_runtime.h"
#include "RandomWalk.h"
#define NUMBER_OF_PHOTONS 1000
#define THREADS_PER_BLOCK 1024  
#define BOUNDARY_RADIUS 10.0  


void streamOut(Point* _cpuPoints);

__global__ void finalPosition(unsigned int seed, hiprandState_t* states, Point* _gpuPoints, Boundary boundary, RNG rng, int n) {
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx < n){
    hiprand_init(seed, idx, 0, &states[idx]);
    Point finalPos = Point();
    finalPos = randomWalk(states, idx, boundary, rng);
    _gpuPoints[idx] = finalPos;
    }
}

  int main() {
    int nBlocks = NUMBER_OF_PHOTONS/THREADS_PER_BLOCK + 1;
    hiprandState_t* states;
    hipMalloc((void**) &states, NUMBER_OF_PHOTONS * sizeof(hiprandState_t));
    // Allocate host memory for final positions
    Point * _cpuPoints= (Point*)malloc(sizeof(Point) * NUMBER_OF_PHOTONS);
    // Allocate device  memory for final positions
    Point* _gpuPoints = nullptr;
    hipMalloc((void**) &_gpuPoints, NUMBER_OF_PHOTONS * sizeof(Point));
    // Initialize the Boundary and the RandomNumberGenerator
    RNG rng;
    Boundary boundary = Boundary(BOUNDARY_RADIUS, Point());
    // Kernel Call
    finalPosition<<<nBlocks,THREADS_PER_BLOCK>>>(time(0), states , _gpuPoints, boundary, rng, NUMBER_OF_PHOTONS);
    // Copy device data to host memory to stream them out
    hipMemcpy(_cpuPoints, _gpuPoints, NUMBER_OF_PHOTONS* sizeof(Point), hipMemcpyDeviceToHost);
    streamOut (&_cpuPoints[0]);
    free(_cpuPoints);
    hipFree(_gpuPoints);
    return 0;
}

void streamOut(Point* _cpuPoints)  
{
    FILE *output;
    output = fopen("output.csv", "w");
    for (int i = 0; i < NUMBER_OF_PHOTONS; i++)
    {
        // Streaming out my output in a log file
        fprintf(output, "%f,%f,%f\n", _cpuPoints[i].x(), _cpuPoints[i].y(), _cpuPoints[i].z());
    }
}
