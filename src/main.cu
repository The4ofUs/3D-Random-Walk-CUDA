#include "hip/hip_runtime.h"
#include "RandomWalk.h"
#define N 5000 //number of photons


void streamOut(Point* _cpuPoints);

__global__ void finalPosition(unsigned int seed, hiprandState_t* states, Point* _gpuPoints) {
    hiprand_init(seed, blockIdx.x, 0, &states[blockIdx.x]);
    Point finalPos = Point();
    finalPos = randomWalk(states);
    _gpuPoints[blockIdx.x] = finalPos;
}

  int main() {
    hiprandState_t* states;
    hipMalloc((void**) &states, N * sizeof(hiprandState_t));

// Allocate host memory for final positions
    Point * _cpuPoints= (Point*)malloc(sizeof(Point) * N);

// Allocate device  memory for final positions
    Point* _gpuPoints = nullptr;
    hipMalloc((void**) &_gpuPoints, N * sizeof(Point));
  
// Call Kernel
    finalPosition<<<N,1>>>(time(0), states , _gpuPoints);

// Copy device data to host memory to stream them out
    hipMemcpy(_cpuPoints, _gpuPoints, N* sizeof( Point), hipMemcpyDeviceToHost);


    streamOut (&_cpuPoints[0]);

    free(_cpuPoints);
    hipFree(_gpuPoints);

    return 0;

}

void streamOut(Point* _cpuPoints)  
{
    FILE *output;
    output = fopen("output.csv", "w");

    for (int i = 0; i < N; i++)
    {
        // Streaming out my output in a log file
        fprintf(output, "%f,%f,%f\n", _cpuPoints[i].getX(), _cpuPoints[i].getY(), _cpuPoints[i].getZ());
    }
}
