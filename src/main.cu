#include "hip/hip_runtime.h"
#include "RandomWalk.h"
#define N 1000 //number of photons


void streamOut(Point* _cpuPoints);

__global__ void finalPosition(unsigned int seed, hiprandState_t* states, Point* _gpuPoints, int n) {
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx < n){
    hiprand_init(seed, idx, 0, &states[idx]);
    Point finalPos = Point();
    finalPos = randomWalk(states, idx);
    _gpuPoints[idx] = finalPos;
    }
}

  int main() {

    int threadsPerBlock = 1024;
    int nBlocks = N/threadsPerBlock + 1;
 
    hiprandState_t* states;
    hipMalloc((void**) &states, N * sizeof(hiprandState_t));

// Allocate host memory for final positions
    Point * _cpuPoints= (Point*)malloc(sizeof(Point) * N);

// Allocate device  memory for final positions
    Point* _gpuPoints = nullptr;
    hipMalloc((void**) &_gpuPoints, N * sizeof(Point));
  
// Call Kernel
    finalPosition<<<nBlocks,threadsPerBlock>>>(time(0), states , _gpuPoints, N);

// Copy device data to host memory to stream them out
    hipMemcpy(_cpuPoints, _gpuPoints, N* sizeof( Point), hipMemcpyDeviceToHost);


    streamOut (&_cpuPoints[0]);

    free(_cpuPoints);
    hipFree(_gpuPoints);

    return 0;

}

void streamOut(Point* _cpuPoints)  
{
    FILE *output;
    output = fopen("output.csv", "w");

    for (int i = 0; i < N; i++)
    {
        // Streaming out my output in a log file
        fprintf(output, "%f,%f,%f\n", _cpuPoints[i].getX(), _cpuPoints[i].getY(), _cpuPoints[i].getZ());
    }
}
