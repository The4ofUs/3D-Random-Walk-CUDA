#include "hip/hip_runtime.h"
#include "RandomWalk.h"
#define N 1000          // Number of photons
#define THREADS_PER_BLOCK 256         // Threads per Block
#define BOUNDARY_RADIUS 10.0


void streamOut(Point* _cpuPoints);

__global__ void finalPosition(unsigned int seed, hiprandState_t* states, Point* _gpuPoints,Boundary boundary,RNG rng, int n) {
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx < n){
    hiprand_init(seed, idx, 0, &states[idx]);
    Point finalPos;
    finalPos = randomWalk(states, idx, boundary, rng);
    _gpuPoints[idx] = finalPos;
    }
}

  int main() {

    int nBlocks = N/THREADS_PER_BLOCK + 1;
 
    hiprandState_t* states;
    hipMalloc((void**) &states, N * sizeof(hiprandState_t));

// Allocate host memory for final positions
    Point * _cpuPoints= (Point*)malloc(sizeof(Point) * N);

// Allocate device  memory for final positions
    Point* _gpuPoints = nullptr;
    hipMalloc((void**) &_gpuPoints, N * sizeof(Point));

// Initializing the Boundary and the Random Number Generator
    Boundary boundary = Boundary(BOUNDARY_RADIUS, Point(0.f, 0.f, 0.f));
    RNG rng;
  
// Call Kernel
    finalPosition<<<nBlocks,THREADS_PER_BLOCK>>>(time(0), states , _gpuPoints, boundary, rng, N);

// Copy device data to host memory to stream them out
    hipMemcpy(_cpuPoints, _gpuPoints, N* sizeof( Point), hipMemcpyDeviceToHost);


    streamOut (&_cpuPoints[0]);

    free(_cpuPoints);
    hipFree(_gpuPoints);


    return 0;

}

void streamOut(Point* _cpuPoints)  
{
    FILE *output;
    output = fopen("output.csv", "w");

    for (int i = 0; i < N; i++)
    {
        // Streaming out my output in a log file
        float absDistance = (float) sqrtf((float) powf(_cpuPoints[i].getX(), 2) 
                            + (float) powf(_cpuPoints[i].getY(), 2) 
                            + (float) powf(_cpuPoints[i].getZ(), 2));
        fprintf(output, "%f,%f,%f,%f\n", _cpuPoints[i].getX(), _cpuPoints[i].getY(), _cpuPoints[i].getZ(), absDistance);
    }
}
