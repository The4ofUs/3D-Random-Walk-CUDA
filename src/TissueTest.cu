#include "hip/hip_runtime.h"
#include "Vector.h"
#include "Point.h"
#include "Ray.h"
#include "Vector.cu"
#include "Point.cu"
#include "Ray.cu"


/**
    TODO: 
        1- Check if the photon is out of radial boundary
        2- Check if Photon is out of axial boundary

        (1) 
        AxisDirection = (center1 - center0)     // Won't matter because we are dealing with abs distances
        P = Center + AxisDirection*t      //Point on the cylinder axis
        V1 = P1 - P     // P1 is the position of the photon
        V1 = P1 - Center - AxisDirection*t
        V1.AxisDirection = 0
        (P1 - Center - AxisDirection*t).AxisDirection = 0
        (P1x - Centerx - AxisDirectionx*t)*AxisDirectionx + 
        (P1y - Centery - AxisDirectiony*t)*AxisDirectiony + 
        (P1z - Centerz - AxisDirectionz*t)*AxisDirectionz = 0 
        
        t = C . (A - B) / ||C||^2

        where
        A : Photon Position
        B : Center of one of the two circular faces of the cylinder
        C : Axis Direction



        (2)
        D = (Center1 - Center0).getAbsDistance()
*/



class Tissue
{
public:
    __host__ Tissue(float radius, Point c0, Point c1)
    {
        this->_radius = radius;
        this->_center0 = c0;
        this->_center1 = c1;
        this->_axis = Vector(c0,c1).normalize();
    }
    
    __host__ bool escaped(Point point){
        Point A = point;
        Point B = this->_center0;
        Vector C = this->_axis;
        printf("A = (%f,%f,%f)\nB = (%f,%f,%f)\nC = (%f,%f,%f)\n", A.x(),A.y(),A.z(),B.x(),B.y(),B.z(),C.x(),C.y(),C.z());
        float t = C.dot(A - B)/ C.getAbsDistance()*C.getAbsDistance();
        Point P = B + C*t;
        printf("t = %f\nP = (%f,%f,%f)\n", t,P.x(),P.y(),P.z());
        float d = A.getAbsDistance(P);
        printf("d = %f\n", d);
        if(d > this->_radius){
            return true;
        }
        float D = (this->_center1 - this->_center0).getAbsDistance();
        float E = (P - this->_center0).getAbsDistance();
        printf("D = %f\nE = %f\n", D, E);
        if(E > D){
            return true;
        }
        E = (P - this->_center1).getAbsDistance();
        if(E > D){
            return true;
        }
        return false;
    }
    Vector getAxis(){
        return this->_axis;
    }

private:
    float _radius;
    Point _center0;
    Point _center1;
    Vector _axis;
};


int main(){
    Tissue tissue = Tissue(5.f,Point(-3.f,0.f,0.f),Point(3.f,0.f,0.f));
    Vector Axis = tissue.getAxis();
    Point P = Point(-4.f,3.f,0.f);
    bool didEscape = tissue.escaped(P);
    printf("Tissue Axis = (%f,%f,%f)\n",Axis.x(), Axis.y(), Axis.z());
    printf("Point current position = (%f,%f,%f)\n",P.x(), P.y(), P.z());
    printf("Did the ray escape? %d\n", didEscape);
    return 0;
}
