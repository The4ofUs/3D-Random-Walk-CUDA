
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>

#define N 256                           // Number of photons
#define THREADS_PER_BLOCK 256           // Threads per Block
#define BOUNDARY_RADIUS 5.0

class Point{
private:
    float _x;
    float _y;
    float _z;
public:
    __device__ __host__ Point(float x, float y, float z){
        setCoordinates(x, y, z);
    }
    
    __device__ __host__  Point(){
        setCoordinates(0.f, 0.f, 0.f);
    }

    __device__ __host__
 void setCoordinates(float x, float y, float z)
{
    this->_x = x;
    this->_y = y;
    this->_z = z;
}
__device__ __host__  float getX() const { return this->_x; }

__device__ __host__   float getY() const { return this->_y; }

__device__ __host__   float getZ() const { return this->_z; }

__device__ __host__  
    Point add(Point point){
        float result_x = this->_x + point.getX();
        float result_y = this->_y + point.getY();
        float result_z = this->_z + point.getZ();
        return Point( result_x, result_y, result_z );
    }

    __device__ __host__  
    Point subtract(Point point){
        float result_x = this->_x - point.getX();
        float result_y = this->_y - point.getY();
        float result_z = this->_z - point.getZ();
        return Point( result_x, result_y, result_z );
    }
};

class RNG{
private:
__device__  float generate( hiprandState* globalState, int i) 
{
    hiprandState localState = globalState[i];
    float random = hiprand_uniform( &localState );
    globalState[i] = localState;
    return random;
}
public:
__device__   float getRandomStep( hiprandState* globalState , int i) { 
    float step = 0.f;       // Intialize for step value
    step = generate (globalState, i);
    return step;
 } 

__device__  Point getRandomPoint( hiprandState* globalState , int i)
{

    float u = generate (globalState , i);
    float v = generate (globalState, i);
    
    float theta = 2 * M_PI * u;
    float phi = acos(1 - 2 * v);

    // Transforming into the cartesian space
    float x = sin(phi) * cos(theta);
    float y = sin(phi) * sin(theta);
    float z = cos(phi);

    return Point(x,y,z);
}
};

class Ray{
private:
    Point _prevPos;
    Point _currentPos;
    Point _direction;
    float _step;
public:
    __device__ Ray(Point startingPoint, Point direction){
        this->_currentPos.setCoordinates(startingPoint.getX(), startingPoint.getY(), startingPoint.getZ());
        this->_direction.setCoordinates(direction.getX(), direction.getY(), direction.getZ());
    }
    
    __device__ void setDirection(Point direction) { this->_direction.setCoordinates(direction.getX(), direction.getY(), direction.getZ()); }
    
    __device__ void setStep(float step) { this->_step = step; }
    
    __device__ Point getCurrentPos() const { return this->_currentPos; }
    
    __device__ Point getDirection() const { return this->_direction; }
    
    __device__ Point getPrevPos() const { return this->_prevPos; }
    
    __device__ float getStep() const { return this->_step; }
    
    __device__ void move(Point direction, float step) // The point moves in the specified direction with the given step
    {
        this->_prevPos = this->_currentPos;
        this->_direction = direction;
        this->_step = step;
        float newX = this->_currentPos.getX() + (direction.getX() * step);
        float newY = this->_currentPos.getY() + (direction.getY() * step);
        float newZ = this->_currentPos.getZ() + (direction.getZ() * step);
        this->_currentPos.setCoordinates(newX, newY, newZ);
    }
};

class Boundary{
private:
    float _radius;
    Point _center;

    __device__
    float dot(Point point1, Point point2){return point1.getX()*point2.getX() + point1.getY()*point2.getY() + point1.getZ()*point2.getZ();}
    
public:
    __device__ __host__ Boundary(float r, Point c){
        _radius = r;
        _center = c;
    }
    
    __device__ bool isCrossed(Ray ray){
        float absDistance = (float) sqrtf((float) powf(ray.getCurrentPos().getX(),2)
                            + (float) powf(ray.getCurrentPos().getY(),2) 
                            + (float) powf(ray.getCurrentPos().getZ(),2));
        if(absDistance >= _radius){
            return true;
        } else {
            return false;
        }
    };
    
    
    __device__ Point getIntersectionPoint(Ray ray){
            Point A = ray.getPrevPos();
            Point B = ray.getDirection();
            Point S = A.add(_center);
            Point A_C = A.subtract(_center);
            float a = dot(B, B);
            float b = 2.0 * dot(B, A_C);
            float c = dot(A_C, A_C) - _radius*_radius;
            float discriminant = b*b - 4*a*c;
            float t1 = (-b + sqrtf(discriminant)) / (2.0*a);
            float t2 = (-b - sqrtf(discriminant)) / (2.0*a);
            float t;
    
            if(t1 < 0){
                t = t2;
            } else {
                t = t1;
            }
    
            return Point((A.getX()+B.getX()*t),(A.getY()+B.getY()*t),(A.getZ()+B.getZ()*t));
    }
};



/**
 * @brief randomWalk
 * keeps wandering around with the photon in the 3D space
 * @return The Point where the Photon hits the Boundary
 */
 __device__ Point randomWalk(hiprandState_t *states, int idx, Boundary boundary, RNG rng)
 {
     Ray ray = Ray(Point(0.f, 0.f, 0.f), Point(0.f, 0.f, 0.f));
 
     while (!boundary.isCrossed(ray))
     {
         ray.move(rng.getRandomPoint(states, idx), rng.getRandomStep(states, idx));
     }
     return boundary.getIntersectionPoint(ray);
 }





void streamOut(Point* _cpuPoints);

__global__ void finalPosition(unsigned int seed, hiprandState_t* states, Point* _gpuPoints,Boundary boundary,RNG rng) {
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    hiprand_init(seed, idx, 0, &states[idx]);
    Point finalPos;
    finalPos = randomWalk(states, idx, boundary, rng);
    _gpuPoints[idx] = finalPos;
}

  int main() {

    int nBlocks = N/THREADS_PER_BLOCK + 1;
 
    hiprandState_t* states;
    hipMalloc((void**) &states, N * sizeof(hiprandState_t));

// Allocate host memory for final positions
    Point * _cpuPoints= (Point*)malloc(sizeof(Point) * N);

// Allocate device  memory for final positions
    Point* _gpuPoints = nullptr;
    hipMalloc((void**) &_gpuPoints, N * sizeof(Point));

// Initializing the Boundary and the Random Number Generator
    Boundary boundary = Boundary(BOUNDARY_RADIUS, Point(0.f, 0.f, 0.f));
    RNG rng;
  
// Call Kernel
    finalPosition<<<nBlocks,THREADS_PER_BLOCK>>>(time(0), states , _gpuPoints, boundary, rng);

// Copy device data to host memory to stream them out
    hipMemcpy(_cpuPoints, _gpuPoints, N* sizeof( Point), hipMemcpyDeviceToHost);


    streamOut (&_cpuPoints[0]);

    free(_cpuPoints);
    hipFree(_gpuPoints);


    return 0;

}

void streamOut(Point* _cpuPoints)  
{
    FILE *output;
    output = fopen("output.csv", "w");

    for (int i = 0; i < N; i++)
    {
        // Streaming out my output in a log file
        float absDistance = (float) sqrtf((float) powf(_cpuPoints[i].getX(), 2) 
                            + (float) powf(_cpuPoints[i].getY(), 2) 
                            + (float) powf(_cpuPoints[i].getZ(), 2));
        fprintf(output, "%f,%f,%f,%f\n", _cpuPoints[i].getX(), _cpuPoints[i].getY(), _cpuPoints[i].getZ(), absDistance);
    }
}
