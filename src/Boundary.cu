#include "hip/hip_runtime.h"
#include "Boundary.h"
using namespace std;

__device__
float Boundary::dot(Point point1, Point point2){return point1.x()*point2.x() + point1.y()*point2.y() + point1.z()*point2.z();}


__device__ __host__ Boundary::Boundary(float r, Point c){
    _radius = r;
    _center = c;
}

__device__ void Boundary::setRadius(float r){_radius = r;}

__device__ float Boundary::getRadius() const {return _radius;}

__device__ void Boundary::setCenter(Point c){_center = c;}

__device__ Point Boundary::getCenter() const {return _center;}

__device__ bool Boundary::isHit(Ray ray){
    float absDistance = (float) sqrtf((float) powf(ray.getCurrentPos().x(),2) + (float) powf(ray.getCurrentPos().y(),2) + (float) powf(ray.getCurrentPos().z(),2));
    if(absDistance >= _radius){
        return true;
    } else {
        return false;
    }
}


__device__ Point Boundary::getIntersectionPoint(Ray ray){
        /**
            P(t) = A + tB
            P(t) is a point on the ray 
            A is the ray origin
            B is the ray direction
            t is a parameter used to move away from ray origin
            S = P - Center
            ||S||^2 = r^2
            Sphere: dot(S,S) = r^2
            Ray: P(t) = A + tB
            Combined: dot((A + tB - Center),(A + tB - Center)) = r^2
            in Quadratic form: t^2.dot(B,B) + 2t.dot(B, A - C) + dot(A - C, A - C) - r^2 = 0
            let a = dot(B,B)
                b = 2.dot(B, A - C)
                c = dot(A - C, A - C) - r^2
            t1, t2 = (-b (+/-) sqrt(b^2 - 4ac) / 2a)
        */
        Point A = ray.getPrevPos();
        Point B = ray.getDirection();
        Point S = A + _center;
        Point A_C = A - _center;
        float a = dot(B, B);
        float b = 2.0 * dot(B, A_C);
        float c = dot(A_C, A_C) - _radius*_radius;
        float discriminant = b*b - 4*a*c;
        float t1 = (-b + sqrtf(discriminant)) / (2.0*a);
        float t2 = (-b - sqrtf(discriminant)) / (2.0*a);
        float t;

        if(t1 < 0){
            t = t2;
        } else {
            t = t1;
        }

        return (A+B*t);
}


