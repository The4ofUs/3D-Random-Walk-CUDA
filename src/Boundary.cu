#include "hip/hip_runtime.h"
#include "Boundary.h"
using namespace std;

__device__
float Boundary::dotProduct(Point point1, Point point2){return point1.getX()*point2.getX() + point1.getY()*point2.getY() + point1.getZ()*point2.getZ();}

__device__
void Boundary::swap(float &num1, float &num2){
    float temp = num1;
    num1 = num2;
    num2 = temp;
}

__device__ Boundary::Boundary(float r, Point c){
    _radius = r;
    _center = c;
}

__device__ void Boundary::setRadius(float r){_radius = r;}

__device__ float Boundary::getRadius() const {return _radius;}

__device__ void Boundary::setCenter(Point c){_center = c;}

__device__ Point Boundary::getCenter() const {return _center;}

__device__ bool Boundary::isCrossed(Ray ray){
    float absDistance = (float) sqrtf((float) powf(ray.getCurrentPos().getX(),2) + (float) powf(ray.getCurrentPos().getY(),2) + (float) powf(ray.getCurrentPos().getZ(),2));
    if(absDistance >= _radius){
        return true;
    } else {
        return false;
    }
}


__device__ Point Boundary::getIntersectionPoint(Ray ray){
    if(this->isCrossed(ray)){
        Point rayOrigin = ray.getPrevPos();
        Point rayDirection = ray.getDirection();
        Point p = Point((_center.getX() - rayOrigin.getX()),(_center.getY() - rayOrigin.getY()), (_center.getZ() - rayOrigin.getZ()));
        float tca = dotProduct(p,rayDirection);
        float d2 = dotProduct(p,p) - tca * tca; 
        float thc = (float) sqrtf((float) powf(_radius,2.0) - d2); 
        float t0 = tca - thc; 
        float t1 = tca + thc;
        float t;
        if (t0 > t1) swap(t0, t1);
 
        if (t0 < 0) { 
            t0 = t1; // if t0 is negative, let's use t1 instead 
        } 
        t = t0;        // this is the intersection distance from the ray origin to the hit point 

        return Point((rayOrigin.getX()+rayDirection.getX()*t),(rayOrigin.getY()+rayDirection.getY()*t),(rayOrigin.getZ()+rayDirection.getZ()*t));

    } else {
        return Point(0.f,0.f,0.f);
    }
}


