#include "hip/hip_runtime.h"
#include "Detector.h"
#include "Point.h"
#include "RNG.h"
#include <math.h>

__host__ Detector::Detector(float radius, Point center, Vector normal)
{
    if (radius > 0)
    {
        this->_radius = radius;
        this->_center = center;
        this->_normal = normal.normalize();
        this->_distance = center.getAbsDistance();
    }
    else
    {
        radius = fabs(radius);
        this->_radius = radius;
        this->_center = center;
        this->_normal = normal.normalize();
        this->_distance = center.getAbsDistance();
    }
};

__device__ float Detector::getAbsDistance()
{
    return this->_distance;
}

__device__ bool Detector::isHit(Ray ray)
{
    float rayAbsDistance = ray.getTip().getAbsDistance();
    float rayOriginAbsDistance = ray.getOrigin().getAbsDistance();
    if (rayAbsDistance >= this->_distance && rayOriginAbsDistance <= this->_distance)
    {
        Point point = this->getIntersectionPoint(ray);
        float dfromc = point.getAbsDistance(this->_center);
        if (dfromc <= this->_radius)
        {
            return true;
        }
        else
            return false;
    }
    else
        return false;
}

__device__ Point Detector::getIntersectionPoint(Ray ray)
{
    /**
        P is a point on the ray
        A is the ray's origin
        B is the direction of the ray
        let's assume that P is on the detector's plane
        We know that P = A + B*t    (1)
        where t is a parameter that determines how far the ray will move in direction B
        (P - Center) will yield a vector on the plane
        so (P - Center).normal = 0  (2)
        now substitute from (1) in (2)
        we will yield an equation with t as the unknown
        we calculate t and substitute in the ray's equation to get the intersection point
        and Voila
    */
    Point A = ray.getOrigin();
    Vector B = ray.getDirection();
    Vector V = (this->_center - A);
    float t = V.dot(this->_normal) / (B.dot(this->_normal));
    return A + B * t;
}
