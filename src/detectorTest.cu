#include "hip/hip_runtime.h"
#include "Point.h"
#include "Point.cu"
#include "RNG.h"
#include "RNG.cu"
#include "Vector.h"
#include "Vector.cu"
#include <array>

#define NUMBER_OF_SAMPLES 100


void streamOut(std::array<Vector, NUMBER_OF_SAMPLES> vectors)  
{
    FILE *output;
    output = fopen("detector.csv", "w+");
    for(int i = 0; i < vectors.size(); i++)
    fprintf(output, "%f,%f,%f\n", vectors[i].x(), vectors[i].y(), vectors[i].z());
}

class Detector
{

public:
    __host__ Detector(float radius, Point center, Vector normal){
        if(radius > 0){
            this->_radius = radius;
            this->_center = center;
            this->_normal = normal.normalize();
            construct();
        }
    }

    __host__ void construct(){
        RNG rng;
        float r = this->_radius;
        float dtheta = 2* M_PI / NUMBER_OF_SAMPLES;
        Point P = rng.getRandomPoint();
        printf("P = (%f,%f,%f) \n", P.x(),P.y(),P.z());
        Point P1 = this->_center;
        printf("P1 = (%f,%f,%f) \n", P1.x(),P1.y(),P1.z());
        Vector P_P1 = Vector(P1,P);
        printf("P - P1 = (%f,%f,%f) \n", P_P1.x(),P_P1.y(),P_P1.z());
        Vector P2_P1 = this->_normal;
        printf("P2 - P1 = (%f,%f,%f) \n", P2_P1.x(),P2_P1.y(),P2_P1.z());
        Vector R = P_P1.cross(P2_P1);
        printf("R = (%f,%f,%f) \n", R.x(),R.y(),R.z());
        Vector S = R.cross(P2_P1);
        printf("S = (%f,%f,%f) \n", S.x(),S.y(),S.z());
        Vector normalizedR = R.normalize();
        printf("Norm R = (%f,%f,%f) \n", normalizedR.x(),normalizedR.y(),normalizedR.z());
        Vector normalizedS = S.normalize();
        printf("Norm S = (%f,%f,%f) \n", normalizedS.x(),normalizedS.y(),normalizedS.z());
        Vector Q = Vector();
        std::array<Vector, NUMBER_OF_SAMPLES> disc;
        int i = 0;
        printf("------------------------------------------------------------------------------------------------------------------------------ \n");
        for(float theta = 0; theta < 2*M_PI; theta+=dtheta){
            Q = P1 + R*(r*cos(theta)) + S*(r*sin(theta));
            Q = Q.normalize();
            printf("%i) Q = (%f,%f,%f) \n", i+1, Q.x(), Q.y(), Q.z());
            disc[i++] = Q;
        }
        streamOut(disc);
    }

private:
    float _radius;
    Point _center;
    Vector _normal;
};


int main(){
    Detector detector = Detector(1.0, Point(0.f,0.f,0.f), Point(1.f,1.f,1.f));
    return 0;
}





