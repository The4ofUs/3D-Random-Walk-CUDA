#include "hip/hip_runtime.h"
#include "/home/gamila/Documents/GP/Task4-RandomWalkCUDA/3D-Random-Walk-CUDA/include/RandomnessGenerator.h"
using namespace std;


__global__ void randomPoint( unsigned int seed, float* randomParameters, hiprandState_t* states){
    // initialize the random states
 hiprand_init(seed, //must be different every run so the sequence of numbers change. 
    blockIdx.x, // the sequence number should be different for each core ???
    0, //step between random numbers
    &states[blockIdx.x]);
    randomParameters[blockIdx.x]=hiprand_uniform(&states[blockIdx.x]);
  }
  
  // Simple random number generator function, generates a float between 0.0 and 1.0
  float RandomnessGenerator::getRandomStep() const { 
    const int N=1;
    float step=0.f;
    float* randomParameters_step= nullptr;
    hipMalloc((void**) &randomParameters_step,  N* sizeof(float)); 
    hiprandState_t* states;
    hipMalloc((void**) &states, N * sizeof(hiprandState_t));

    randomPoint<<<N, 1>>>(time(0),randomParameters_step, states);
    hipMemcpy(&step, randomParameters_step, N*sizeof(float), hipMemcpyDeviceToHost);
    return step;
 } 

// Returns a Point object that has randomized x,y and z coordinates after converting from randomized spherical coordinates
float RandomnessGenerator::getRandomPoint()
{
    Point point; // Instance of the Point struct to return with the random coordinates

    int N=3;
    float* randomParameters= nullptr;
    hipMalloc((void**) &randomParameters,  N* sizeof(float)); 
    hiprandState_t* states;
    hipMalloc((void**) &states, N * sizeof(hiprandState_t)); 

    float* cpu_randoms= (float*)malloc(sizeof(float) * N);
    // Getting random values for spherical coordinates transformation parameters
    randomPoint<<<N, 1>>>(time(0),randomParameters, states);
    
    hipMemcpy(cpu_randoms, randomParameters, N*sizeof(float), hipMemcpyDeviceToHost);
    float u=cpu_randoms[0] ;
    float v= cpu_randoms[1];
    //float r= cpu_randoms[2];
    
    float theta = 2 * M_PI * u;
    float phi = acos(1 - 2 * v);

    // Transforming into the cartesian space
    float x = sin(phi) * cos(theta);
    float y = sin(phi) * sin(theta);
    float z = cos(phi);

    point.setCoordinates(x, y, z);
  


   exportSamplingPlot(point);

    return x;

}

// A helper function to generate a csv file to use in plotting
void RandomnessGenerator::exportSamplingPlot(Point point)
{
    // For streaming out my output in a log file
    FILE *sampling;
    sampling = fopen("sampling.csv", "a");
    // Streaming out my output in a log file
    fprintf(sampling, "%f,%f,%f\n", point.getX(), point.getY(), point.getZ());
} 

int main() {
RandomnessGenerator randomnessGenerator;
float x= randomnessGenerator.getRandomPoint();
float step =randomnessGenerator.getRandomStep();

  
std:: cout << x <<endl;
std:: cout <<step <<endl;


  /* free the memory we allocated */
 //hipFree(u_randoms); 
  return 0;
    
}