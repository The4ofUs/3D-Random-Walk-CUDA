#include "hip/hip_runtime.h"
#include "../headers/mathematics.h"

__host__ __device__ float Mathematics::calculateAbsDistance(Point p1, Point p2)
{
    return sqrtf(((p2.x() - p1.x()) * (p2.x() - p1.x()) + (p2.y() - p1.y()) * (p2.y() - p1.y()) + (p2.z() - p1.z()) * (p2.z() - p1.z())));
}

__host__ __device__ float Mathematics::calculateAbsDistance(Point p)
{
    return sqrtf(((p.x()) * (p.x()) + (p.y()) * (p.y()) + (p.z()) * (p.z())));
}

__device__ __host__ float Mathematics::calculateDotProduct(Vector v1, Vector v2)
{
    return v1.x() * v2.x() + v1.y() * v2.y() + v1.z() * v2.z();
}

__device__ __host__ Vector Mathematics::calculateCrossProduct(Vector v1, Vector v2)
{
    float X = v1.y() * v2.z() - v1.z() * v2.y();
    float Y = (-1) * v1.x() * v2.z() + v1.z() * v2.x();
    float Z = v1.x() * v2.y() - v1.y() * v2.x();
    return Vector(X, Y, Z);
}

__device__ __host__ Vector Mathematics::calculateNormalizedVector(Vector v)
{
    float norm = sqrtf((powf(v.x(), 2) + powf(v.y(), 2) + powf(v.z(), 2)));
    float xhat = v.x() / norm;
    float yhat = v.y() / norm;
    float zhat = v.z() / norm;
    return Vector(xhat, yhat, zhat);
}

// Needs to be tested
__device__ __host__ Point Mathematics::calculateRayTip(Point origin, Vector direction, float step)
{
    return (origin + direction * step);
}