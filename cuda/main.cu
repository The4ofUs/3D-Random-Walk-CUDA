#include "hip/hip_runtime.h"
#include "code/headers/randomwalk.h"
<<<<<<< HEAD:main.cu
#include "Network/Client/Headers/socket.h"
#include <QDebug>
#include <QVector>
=======
#include <sstream>

#define NUMBER_OF_PHOTONS 1000
>>>>>>> origin/master:cuda/main.cu
#define THREADS_PER_BLOCK 1024
#define DETECTOR_LOOK_DOWNWARDS Vector(0.f, 0.f, -1.f)

/*#define NUMBER_OF_PHOTONS 10
#define DETECTOR_RADIUS 10.f
#define DETECTOR_POSITION Point(0.f, 0.f, 50.f)
<<<<<<< HEAD:main.cu
=======
#define DETECTOR_LOOKAT Vector(0.f, 0.f, -1.f)
>>>>>>> origin/master:cuda/main.cu
#define TISSUE_RADIUS 100.f
#define TISSUE_ABSORBTION_COEFFICIENT 1.f
#define TISSUE_SCATTERING_COEFFICIENT 100.f
#define TISSUE_CENTER_1 Point(0.f, 0.f, 50.f)
<<<<<<< HEAD:main.cu
#define TISSUE_CENTER_2 Point(0.f, 0.f, -50.f)*/
int numberOfPhotons;
float detectorRadius;
float tissueRadius;
float tissueAbsCoeff;
float tissueScatCoeff;
Point detectorPosition;
Point tissueFirstCenter;
Point tissueSecondCenter;
QVector<Photon> photons;
bool newBatchAvailable;
char *stateToString(int state);
void streamOut(Photon *_cpuPhotons);
void sendResults(Photon *_cpuPhotons);
void requestParameters();
void populateParameters(QVector<float> parameters);
void applyMC();
void askForNewBatch();
=======
#define TISSUE_CENTER_2 Point(0.f, 0.f, -50.f)



void streamOut(Photon *_cpuPhotons);

>>>>>>> origin/master:cuda/main.cu
__global__ void finalState(unsigned int seed, hiprandState_t *states, Photon *_gpuPhotons, Detector detector, RNG rng, Tissue tissue, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        hiprand_init(seed, idx, 0, &states[idx]);
        Photon finalState = randomWalk(states, idx, detector, rng, tissue);
        _gpuPhotons[idx] = finalState;
    }
}


int main()
{
<<<<<<< HEAD:main.cu
    requestParameters();
    while(newBatchAvailable){
        applyMC();
    }
   // applyMC();
    return 0;
}

void applyMC(){
    int nBlocks = numberOfPhotons / THREADS_PER_BLOCK + 1;
    hiprandState_t *states;
    hipMalloc((void **)&states, numberOfPhotons * sizeof(hiprandState_t));
    // Allocate host memory for final positions
    Photon *_cpuPhotons = (Photon *)malloc(sizeof(Photon) * numberOfPhotons);
    // Allocate device  memory for final positions
    Photon *_gpuPhotons = nullptr;
    hipMalloc((void **)&_gpuPhotons, numberOfPhotons * sizeof(Photon));
    // Initialize the Boundary and the RandomNumberGenerator
    RNG rng;
    //Boundary boundary = Boundary(BOUNDARY_RADIUS, Point());
    Detector detector = Detector(detectorRadius, detectorPosition, DETECTOR_LOOK_DOWNWARDS);
    Tissue tissue = Tissue(tissueRadius, tissueFirstCenter, tissueSecondCenter, tissueAbsCoeff, tissueScatCoeff);
    // Kernel Call
    //finalPosition<<<nBlocks,THREADS_PER_BLOCK>>>(time(0), states , _gpuPoints, boundary, rng, NUMBER_OF_PHOTONS);
    finalState<<<nBlocks, THREADS_PER_BLOCK>>>(time(0), states, _gpuPhotons, detector, rng, tissue, numberOfPhotons);
    // Copy device data to host memory to stream them out
    hipMemcpy(_cpuPhotons, _gpuPhotons, numberOfPhotons * sizeof(Photon), hipMemcpyDeviceToHost);
=======
    int nBlocks = NUMBER_OF_PHOTONS + THREADS_PER_BLOCK - 1 / THREADS_PER_BLOCK;
    hiprandState_t *states;
    hipMalloc((void **)&states, NUMBER_OF_PHOTONS * sizeof(hiprandState_t));
    Photon *_cpuPhotons = (Photon *)malloc(sizeof(Photon) * NUMBER_OF_PHOTONS);
    Photon *_gpuPhotons = nullptr;
    hipMalloc((void **)&_gpuPhotons, NUMBER_OF_PHOTONS * sizeof(Photon));
    RNG rng;
    Detector detector = Detector(DETECTOR_RADIUS, DETECTOR_POSITION, DETECTOR_LOOKAT);
    Tissue tissue = Tissue(TISSUE_RADIUS, TISSUE_CENTER_1, TISSUE_CENTER_2, TISSUE_ABSORBTION_COEFFICIENT, TISSUE_SCATTERING_COEFFICIENT);
    finalState<<<nBlocks, THREADS_PER_BLOCK>>>(time(0), states, _gpuPhotons, detector, rng, tissue, NUMBER_OF_PHOTONS);
    hipMemcpy(_cpuPhotons, _gpuPhotons, NUMBER_OF_PHOTONS * sizeof(Photon), hipMemcpyDeviceToHost);
>>>>>>> origin/master:cuda/main.cu
    streamOut(&_cpuPhotons[0]);
    sendResults(&_cpuPhotons[0]);
    askForNewBatch();
    free(_cpuPhotons);
    hipFree(_gpuPhotons);
<<<<<<< HEAD:main.cu
}





void sendResults(Photon *_cpuPhotons){

    QVector<Photon> vectorOfPhotons;
    for (int i = 0; i < numberOfPhotons; i++)
    {
        vectorOfPhotons.push_back(_cpuPhotons[i]);
    }
    socket *newSocket =new socket();
    newSocket->queryType="prepareForReceiving";
    newSocket->socket::getVectorOfPhotons(vectorOfPhotons);
    newSocket->createSocket();

}


void requestParameters(){
    socket *newSocket =new socket();
    newSocket->queryType="requestParameters";
    newSocket->createSocket();
    QVector<float> parameters = newSocket->getParameters();
    //qDebug()<<parameters<<parameters.size();
    if(parameters.size()>0){
        populateParameters(parameters);
        newBatchAvailable = true;
    }
}

void askForNewBatch(){
    socket *newSocket =new socket();
    newSocket->queryType="requestBatch";
    newSocket->createSocket();
    numberOfPhotons = newSocket->numberOfPhotons;
    if (numberOfPhotons==0){
        newBatchAvailable = false;
    }
}

void populateParameters(QVector<float> parameters){
    numberOfPhotons = (int) parameters[0];
    detectorRadius  = parameters[1];
    detectorPosition = Point(parameters[2],parameters[3],parameters[4]);
    tissueRadius = parameters[5];
    tissueAbsCoeff = parameters[6];
    tissueScatCoeff = parameters[7];
    tissueFirstCenter =  Point(parameters[8], parameters[9], parameters[10]);
    tissueSecondCenter = Point(parameters[11],parameters[12],parameters[13]);
    qDebug()<<"Parameters are received";
=======
    hipFree(states);
    return 0;
>>>>>>> origin/master:cuda/main.cu
}

void streamOut(Photon *_cpuPhotons)
{
    FILE *output;
    output = fopen("Results.csv", "w");
    std::string state;
<<<<<<< HEAD:main.cu
    fprintf(output, "X,Y,Z,WEIGHT,STATE\n");
    for (int i = 0; i < numberOfPhotons; i++)
=======
    //Header
    fprintf(output, "%s,%s,%s,%s,%s\n", "X", "Y", "Z", "Weight", "State");
    for (int i = 0; i < NUMBER_OF_PHOTONS; i++)
>>>>>>> origin/master:cuda/main.cu
    {
        switch (_cpuPhotons[i].getState())
        {
        case (-1):
            state = "TERMINATED";
            break;
        case (0):
            state = "ROAMING";
            break;
        case (1):
            state = "DETECTED";
            break;
        case (2):
            state = "ESCAPED";
            break;
        }
        // Streaming out my output in a log file
        fprintf(output, "%f,%f,%f,%f,%s\n", _cpuPhotons[i].getPosition().x(), _cpuPhotons[i].getPosition().y(), _cpuPhotons[i].getPosition().z(), _cpuPhotons[i].getWeight(), state.c_str());
        //qDebug()<< _cpuPhotons[i].getPosition().x()<< _cpuPhotons[i].getPosition().y()<< _cpuPhotons[i].getPosition().z()<< _cpuPhotons[i].getWeight()<< state.c_str();


    }
}



