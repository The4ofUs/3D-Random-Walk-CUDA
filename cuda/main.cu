#include "hip/hip_runtime.h"
#include "code/headers/randomwalk.h"
#include <sstream>

#define NUMBER_OF_PHOTONS 1000
#define THREADS_PER_BLOCK 1024
#define DETECTOR_RADIUS 10.f
#define DETECTOR_POSITION Point(0.f, 0.f, 50.f)
#define DETECTOR_LOOKAT Vector(0.f, 0.f, -1.f)
#define TISSUE_RADIUS 100.f
#define TISSUE_ABSORBTION_COEFFICIENT 1.f
#define TISSUE_SCATTERING_COEFFICIENT 100.f
#define TISSUE_CENTER_1 Point(0.f, 0.f, 50.f)
#define TISSUE_CENTER_2 Point(0.f, 0.f, -50.f)



void streamOut(Photon *_cpuPhotons);

__global__ void finalState(unsigned int seed, hiprandState_t *states, Photon *_gpuPhotons, Detector detector, RNG rng, Tissue tissue, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        hiprand_init(seed, idx, 0, &states[idx]);
        Photon finalState = randomWalk(states, idx, detector, rng, tissue);
        _gpuPhotons[idx] = finalState;
    }
}


int main()
{
    int nBlocks = NUMBER_OF_PHOTONS + THREADS_PER_BLOCK - 1 / THREADS_PER_BLOCK;
    hiprandState_t *states;
    hipMalloc((void **)&states, NUMBER_OF_PHOTONS * sizeof(hiprandState_t));
    Photon *_cpuPhotons = (Photon *)malloc(sizeof(Photon) * NUMBER_OF_PHOTONS);
    Photon *_gpuPhotons = nullptr;
    hipMalloc((void **)&_gpuPhotons, NUMBER_OF_PHOTONS * sizeof(Photon));
    RNG rng;
    Detector detector = Detector(DETECTOR_RADIUS, DETECTOR_POSITION, DETECTOR_LOOKAT);
    Tissue tissue = Tissue(TISSUE_RADIUS, TISSUE_CENTER_1, TISSUE_CENTER_2, TISSUE_ABSORBTION_COEFFICIENT, TISSUE_SCATTERING_COEFFICIENT);
    finalState<<<nBlocks, THREADS_PER_BLOCK>>>(time(0), states, _gpuPhotons, detector, rng, tissue, NUMBER_OF_PHOTONS);
    hipMemcpy(_cpuPhotons, _gpuPhotons, NUMBER_OF_PHOTONS * sizeof(Photon), hipMemcpyDeviceToHost);
    streamOut(&_cpuPhotons[0]);
    free(_cpuPhotons);
    hipFree(_gpuPhotons);
    hipFree(states);
    return 0;
}

void streamOut(Photon *_cpuPhotons)
{
    FILE *output;
    output = fopen("Results.csv", "w");
    std::string state;
    //Header
    fprintf(output, "%s,%s,%s,%s,%s\n", "X", "Y", "Z", "Weight", "State");
    for (int i = 0; i < NUMBER_OF_PHOTONS; i++)
    {
        switch (_cpuPhotons[i].getState())
        {
        case (-1):
            state = "TERMINATED";
            break;
        case (0):
            state = "ROAMING";
            break;
        case (1):
            state = "DETECTED";
            break;
        case (2):
            state = "ESCAPED";
            break;
        }
        // Streaming out my output in a log file
        fprintf(output, "%f,%f,%f,%f,%s\n", _cpuPhotons[i].getPosition().x(), _cpuPhotons[i].getPosition().y(), _cpuPhotons[i].getPosition().z(), _cpuPhotons[i].getWeight(), state.c_str());
    }
}



