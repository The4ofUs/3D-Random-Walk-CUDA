#include "hip/hip_runtime.h"
#include "code/headers/randomwalk.h"
#include <sstream>

#define NUMBER_OF_PHOTONS 1000
#define THREADS_PER_BLOCK 1024
#define DETECTOR_RADIUS 100.f
#define DETECTOR_POSITION Point(0.f, 0.f, 50.f)
#define DETECTOR_LOOKAT Vector(0.f, 0.f, -1.f)
#define TISSUE_RADIUS 1000.f
#define LAYER_ABSORBTION_COEFFICIENT1 1.f
#define LAYER_SCATTERING_COEFFICIENT1 100.f
#define LAYER_ABSORBTION_COEFFICIENT2 1.f
#define LAYER_SCATTERING_COEFFICIENT2 50.f
#define LAYER_ABSORBTION_COEFFICIENT3 10.f
#define LAYER_SCATTERING_COEFFICIENT3 100.f
#define LAYER_CENTER_00 Point(0.f, 0.f, 50.f)
#define LAYER_CENTER_01 Point(0.f, 0.f, -50.f)
#define LAYER_CENTER_11 Point(0.f, 0.f, -150.f)
#define LAYER_CENTER_22 Point(0.f, 0.f, -250.f)


void streamOut(Photon *_cpuPhotons);

__global__ void finalState(unsigned int seed, hiprandState_t *states, Photon *_gpuPhotons, Detector detector, RNG rng, MultiLayer layer1, MultiLayer layer2, MultiLayer layer3 ,int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        hiprand_init(seed, idx, 0, &states[idx]);
        Photon finalState = randomWalk(states, idx, detector, rng, layer1, layer2, layer3);
        _gpuPhotons[idx] = finalState;
    }
}


int main()
{
    int nBlocks = NUMBER_OF_PHOTONS + THREADS_PER_BLOCK - 1 / THREADS_PER_BLOCK;
    hiprandState_t *states;
    hipMalloc((void **)&states, NUMBER_OF_PHOTONS * sizeof(hiprandState_t));
    Photon *_cpuPhotons = (Photon *)malloc(sizeof(Photon) * NUMBER_OF_PHOTONS);
    Photon *_gpuPhotons = nullptr;
    hipMalloc((void **)&_gpuPhotons, NUMBER_OF_PHOTONS * sizeof(Photon));
    RNG rng;
    Detector detector = Detector(DETECTOR_RADIUS, DETECTOR_POSITION, DETECTOR_LOOKAT);
    MultiLayer layer1 = MultiLayer(TISSUE_RADIUS, LAYER_CENTER_00, LAYER_CENTER_01, LAYER_ABSORBTION_COEFFICIENT1, LAYER_SCATTERING_COEFFICIENT1);
    MultiLayer layer2 = MultiLayer(TISSUE_RADIUS, LAYER_CENTER_01, LAYER_CENTER_11, LAYER_ABSORBTION_COEFFICIENT2, LAYER_SCATTERING_COEFFICIENT2);
    MultiLayer layer3 = MultiLayer(TISSUE_RADIUS, LAYER_CENTER_11, LAYER_CENTER_22, LAYER_ABSORBTION_COEFFICIENT3, LAYER_SCATTERING_COEFFICIENT3);

    finalState<<<nBlocks, THREADS_PER_BLOCK>>>(time(0), states, _gpuPhotons, detector, rng, layer1, layer2, layer3, NUMBER_OF_PHOTONS);
    hipMemcpy(_cpuPhotons, _gpuPhotons, NUMBER_OF_PHOTONS * sizeof(Photon), hipMemcpyDeviceToHost);
    streamOut(&_cpuPhotons[0]);
    free(_cpuPhotons);
    hipFree(_gpuPhotons);
    hipFree(states);
    return 0;
}

void streamOut(Photon *_cpuPhotons)
{
    FILE *output;
    output = fopen("results.csv", "w");
    std::string state;
    //Header
    fprintf(output, "%s,%s,%s,%s,%s\n", "X", "Y", "Z", "Weight", "State");
    for (int i = 0; i < NUMBER_OF_PHOTONS; i++)
    {
        switch (_cpuPhotons[i].getState())
        {
        case (-1):
            state = "TERMINATED";
            break;
        case (0):
            state = "ROAMING";
            break;
        case (1):
            state = "DETECTED";
            break;
        case (2):
            state = "ESCAPED";
            break;
        }
        // Streaming out my output in a log file
        fprintf(output, "%f,%f,%f,%f,%s\n", _cpuPhotons[i].getPosition().x(), _cpuPhotons[i].getPosition().y(), _cpuPhotons[i].getPosition().z(), _cpuPhotons[i].getWeight(), state.c_str());
    }
}



