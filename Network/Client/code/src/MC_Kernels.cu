#include "hip/hip_runtime.h"
//
// Created by mustafa on 6/3/20.
//

#include <hiprand/hiprand_kernel.h>
#include "../headers/MC_Kernels.cuh"
#include "../headers/MC_RandomWalk.cuh"

__global__ void
MCKernels::simulate(unsigned int seed, hiprandState_t *states, MC_Photon *_gpuPhotons, MC_FiberGenerator const mcFiberGenerator,
                    MC_MLTissue tissue, int const n) {
    int idx = (int) (blockIdx.x * blockDim.x + threadIdx.x);
    if (idx < n) {
        hiprand_init(seed, idx, 0, &states[idx]);
        MC_Photon finalState = RandomWalk(states, idx, mcFiberGenerator, tissue);
        _gpuPhotons[idx] = finalState;
    }
}

