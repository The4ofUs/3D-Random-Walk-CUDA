//
// Created by mostafa on ٢‏/٨‏/٢٠٢٠.
//

#include "../headers/MC_Simulation.cuh"
#include "../headers/MC_Kernels.cuh"
#include "../headers/MC_Helpers.cuh"
#include "../../clientSide/headers/ClientSocket.h"
#include <QVector>

MC_Simulation::MC_Simulation(float MC_FIBER_GENERATOR_RADIUS, MC_Point MC_FIBER_GENERATOR_POSITION, float TISSUE_RADIUS, MC_Point TISSUE_CENTER_1, MC_Point TISSUE_CENTER_2
                             , std::vector<float>A_COEFFICIENTS, std::vector<float>S_COEFFICIENTS, std::vector<float> R_INDICES) {
    _mcFiberGenerator = MC_FiberGenerator(MC_FIBER_GENERATOR_RADIUS, MC_FIBER_GENERATOR_POSITION, MC_FIBER_GENERATOR_NORMAL);
    _mcMLTissue = MC_MLTissue(TISSUE_RADIUS, TISSUE_CENTER_1, TISSUE_CENTER_2, A_COEFFICIENTS, S_COEFFICIENTS,
                              R_INDICES);
}


void MC_Simulation::start(int NUMBER_OF_PHOTONS) {
    _totalPhotonsPerPatch.clear();
    ClientSocket* socket = new ClientSocket();
    socket->setBatchPhotons(NUMBER_OF_PHOTONS);
    int blocksCount = NUMBER_OF_PHOTONS + THREADS_PER_BLOCK - 1 / THREADS_PER_BLOCK;
    hiprandState_t *states;
    hipMalloc((void **) &states, NUMBER_OF_PHOTONS * sizeof(hiprandState_t));
    auto *hostMemory = (MC_Photon *) malloc(sizeof(MC_Photon) * NUMBER_OF_PHOTONS);
    MC_Photon *deviceMemory = nullptr;
    hipMalloc((void **) &deviceMemory, NUMBER_OF_PHOTONS * sizeof(MC_Photon));
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    MCKernels::simulate<<<blocksCount, THREADS_PER_BLOCK>>>(time(nullptr), states, deviceMemory, _mcFiberGenerator, _mcMLTissue, NUMBER_OF_PHOTONS);
    hipEventRecord(stop);
    hipMemcpy(hostMemory, deviceMemory, NUMBER_OF_PHOTONS * sizeof(MC_Photon), hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    _totalTime += milliseconds;
    hipEventDestroy( start );
    hipEventDestroy( stop );
    socket->sendResults(&hostMemory[0]);
    _totalPhotonsPerPatch.append(socket->getSentPhotons());
    // MCHelpers::streamOut(&hostMemory[0], NUMBER_OF_PHOTONS);
    NUMBER_OF_PHOTONS = socket->requestNewBatch();
   _batchAvailability = socket->getBatchAvailability();
    qDebug()<<"Batch availability flag is "<<_batchAvailability;
    free(hostMemory);
    hipFree(deviceMemory);
    hipFree(states);
}
