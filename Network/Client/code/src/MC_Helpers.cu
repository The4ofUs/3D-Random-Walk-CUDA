#include "hip/hip_runtime.h"
//
// Created by mustafa on 6/3/20.
//

#include "../headers/MC_Photon.cuh"
#include "../headers/MC_Helpers.cuh"

void MCHelpers::streamOut(MC_Photon *_cpuPhotons, int n) {
    FILE *output;
    output = fopen("Results.csv", "w");
    std::string state;
    //Header
    fprintf(output, "%s,%s,%s,%s,%s\n", "X", "Y", "Z", "Weight", "State");
    for (int i = 0; i < n; i++) {
        switch (_cpuPhotons[i].state()) {
            case (MC_Photon::TERMINATED):
                state = "TERMINATED";
                break;
            case (MC_Photon::ROAMING):
                state = "ROAMING";
                break;
            case (MC_Photon::DETECTED):
                state = "DETECTED";
                break;
            case (MC_Photon::ESCAPED):
                state = "ESCAPED";
                break;
        }
        // Streaming out my output in a log file
        fprintf(output, "%f,%f,%f,%f,%s\n", _cpuPhotons[i].position().x(), _cpuPhotons[i].position().y(),
                _cpuPhotons[i].position().z(), _cpuPhotons[i].weight(), state.c_str());
    }
}

void MCHelpers::streamOut(QVector<MC_Photon> results){
    FILE *output;
    output = fopen("clientSentPhotons.csv", "w");
    std::string state;
    fprintf(output, "X,Y,Z,WEIGHT,STATE\n");
   // qDebug()<<"Total sent photons"<<results.size();
    for (int i = 0; i < results.size(); i++)
    {
        switch (results[i].state())
        {
        case (MC_Photon::TERMINATED):
                state = "TERMINATED";
                break;
            case (MC_Photon::ROAMING):
                state = "ROAMING";
                break;
            case (MC_Photon::DETECTED):
                state = "DETECTED";
                break;
            case (MC_Photon::ESCAPED):
                state = "ESCAPED";
                break;
        }
        fprintf(output, "%f,%f,%f,%f,%s\n", results[i].position().x(), results[i].position().y(), results[i].position().z(), results[i].weight(), state.c_str());

    }
    fclose(output);
}



