#include "hip/hip_runtime.h"
#include "RandomWalk.h"
#define N 1000 // Number of photons 


void streamOut(float* _cpuX, float* _cpuY , float* _cpuZ);
/*
__global__ void finalPosition(unsigned int seed, hiprandState_t* states, float* _gpuX, float* _gpuY, float* _gpuZ) {
    hiprand_init(seed, blockIdx.x, 0, &states[blockIdx.x]);

    Point finalPos; //should pass states here as an arg
    finalPos = randomWalk(states);
    _gpuX[blockIdx.x] = finalPos.getX();
    _gpuY[blockIdx.x] = finalPos.getY();
    _gpuZ[blockIdx.x] = finalPos.getZ();

}
*/

__global__ void finalPosition(unsigned int seed, hiprandState_t* states, Point* _gpuPoints) {
    hiprand_init(seed, blockIdx.x, 0, &states[blockIdx.x]);
    Point finalPos; //should pass states here as an arg
    finalPos = randomWalk(states);
    _gpuPoints[blockIdx.x] = finalPos;
}

  int main() {
    hiprandState_t* states;
    hipMalloc((void**) &states, N * sizeof(hiprandState_t));

// Allocate host memory for final positions
/*
    float * _cpuX= (float*)malloc(sizeof(float) * N);
    float * _cpuY= (float*)malloc(sizeof(float) * N);
    float * _cpuZ= (float*)malloc(sizeof(float) * N);
*/
    Point * _cpuPoints= (Point*)malloc(sizeof(Point) * N);

// Allocate device  memory for final positions
/*    
    float* _gpuX = nullptr;
    hipMalloc((void**) &_gpuX, N * sizeof(float));
    float* _gpuY = nullptr;
    hipMalloc((void**) &_gpuY, N * sizeof(float));
    float* _gpuZ = nullptr;
    hipMalloc((void**) &_gpuZ, N * sizeof(float));
*/
    Point* _gpuPoints = nullptr;
    hipMalloc((void**) &_gpuPoints, N * sizeof(Point));
  
// Call Kernel
/*
    finalPosition<<<N , 1>>>(time(0), states , _gpuX, _gpuY, _gpuZ);
*/
    finalPosition<<<N , 1>>>(time(0), states , _gpuPoints);

// Copy device data to host memory to stream them out
/*
    hipMemcpy(_cpuX, _gpuX, N * sizeof( float), hipMemcpyDeviceToHost);
    hipMemcpy(_cpuY, _gpuY, N * sizeof( float), hipMemcpyDeviceToHost);
    hipMemcpy(_cpuZ, _gpuZ, N * sizeof( float), hipMemcpyDeviceToHost);
*/
    hipMemcpy(_cpuPoints, _gpuPoints, N* sizeof( Point), hipMemcpyDeviceToHost);

    // Stream out final position of each photon to file
/*
    streamOut (&_cpuX[0], &_cpuY[0], &_cpuZ[0]);
*/
    streamOut (&_cpuPoints);

// Free Memory
/*
    free(_cpuX);
    free(_cpuY);
    free(_cpuZ);
    hipFree(_gpuX);
    hipFree(_gpuY);
    hipFree(_gpuZ);
*/
    free(_cpuPoints);
    hipFree(_gpuPoints);

    return 0;

}

/*
void streamOut(float* _cpuX, float* _cpuY , float* _cpuZ)  
{
    FILE *output;
    output = fopen("output.csv", "a");

    for (int i = 0; i < N; i++)
    {
        // Streaming out my output in a log file
        fprintf(output, "%f,%f,%f\n", _cpuX[i], _cpuY[i], _cpuZ[i]);
    }
}
*/
void streamOut(Point* _cpuPoints)  
{
    FILE *output;
    output = fopen("output.csv", "a");

    for (int i = 0; i < N; i++)
    {
        // Streaming out my output in a log file
        fprintf(output, "%f,%f,%f\n", _cpuPoints[i].getX(), _cpuPoints[i].getY(), _cpuPoints[i].getZ());
    }
}
