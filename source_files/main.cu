#include "hip/hip_runtime.h"
#include "RandomWalk.h"
#define N 1000 // Number of photons 


void streamOut(float* _cpuX, float* _cpuY , float* _cpuZ);

__global__ void finalPosition(unsigned int seed, hiprandState_t* states, float* _gpuX, float* _gpuY, float* _gpuZ) {
    hiprand_init(seed, blockIdx.x, 0, &states[blockIdx.x]);

    Point finalPos; //should pass states here as an arg
    finalPos = randomWalk(states);
    _gpuX[blockIdx.x] = finalPos.getX();
    _gpuY[blockIdx.x] = finalPos.getY();
    _gpuZ[blockIdx.x] = finalPos.getZ();

}


  int main() {
    hiprandState_t* states;
    hipMalloc((void**) &states, N * sizeof(hiprandState_t));

// Allocate host memory for final positions
    float * _cpuX= (float*)malloc(sizeof(float) * N);
    float * _cpuY= (float*)malloc(sizeof(float) * N);
    float * _cpuZ= (float*)malloc(sizeof(float) * N);

// Allocate device  memory for final positions
    float* _gpuX = nullptr;
    hipMalloc((void**) &_gpuX, N * sizeof(float));
    float* _gpuY = nullptr;
    hipMalloc((void**) &_gpuY, N * sizeof(float));
    float* _gpuZ = nullptr;
    hipMalloc((void**) &_gpuZ, N * sizeof(float));
  
// Call Kernel
    finalPosition<<<N , 1>>>(time(0), states , _gpuX, _gpuY, _gpuZ);

// Copy device data to host memory to stream them out
    hipMemcpy(_cpuX, _gpuX, N * sizeof( float), hipMemcpyDeviceToHost);
    hipMemcpy(_cpuY, _gpuY, N * sizeof( float), hipMemcpyDeviceToHost);
    hipMemcpy(_cpuZ, _gpuZ, N * sizeof( float), hipMemcpyDeviceToHost);

    // Stream out final position of each photon to file
    streamOut (&_cpuX[0], &_cpuY[0], &_cpuZ[0]);

// Free Memory
    free(_cpuX);
    free(_cpuY);
    free(_cpuZ);
    hipFree(_gpuX);
    hipFree(_gpuY);
    hipFree(_gpuZ);

    return 0;

}
void streamOut(float* _cpuX, float* _cpuY , float* _cpuZ)  
{
    FILE *output;
    output = fopen("output.csv", "a");

    for (int i = 0; i < N; i++)
    {
        // Streaming out my output in a log file
        fprintf(output, "%f,%f,%f\n", _cpuX[i], _cpuY[i], _cpuZ[i]);
    }
}
