#include "hip/hip_runtime.h"
#include "header.h"
#define N 1000 // Number of photons 

__global__ void setupKernel(unsigned int seed, hiprandState_t* states){

    hiprand_init(seed, //must be different every run so the sequence of numbers change. 
        blockIdx.x, // the sequence number should be different for each core ???
        0, //step between random numbers
        &states[blockIdx.x]);
      
}

__global__ void finalPosition(hiprandState_t* states, float* _gpuX, float* _gpuY, float* _gpuZ) {
    
    uniform_random_numbers[blockIdx.x] = hiprand_uniform(&states[blockIdx.x]);
  }


  int main() {
    hiprandState_t* states;
    hipMalloc((void**) &states, N * sizeof(hiprandState_t));

// Allocate host memory for final positions
    float * _cpuX= (float*)malloc(sizeof(float) * N);
    float * _cpuY= (float*)malloc(sizeof(float) * N);
    float * _cpuZ= (float*)malloc(sizeof(float) * N);

// Allocate device  memory for final positions
    float* _gpuX = nullptr;
    hipMalloc((void**) &_gpuX, N * sizeof(float));
    float* _gpuY = nullptr;
    hipMalloc((void**) &_gpuY, N * sizeof(float));
    float* _gpuZ = nullptr;
    hipMalloc((void**) &_gpuZ, N * sizeof(float));
  
// Call Kernel
    setupKernel<<<N , 1>>> (time(0), states;)
    finalPosition<<<N , 1>>>(states , _gpuX, _gpuY, _gpuZ);

// Copy device data to host memory to stream them out
    hipMemcpy(_cpuX, _gpuX, N * sizeof( float), hipMemcpyDeviceToHost);
    hipMemcpy(_cpuY, _gpuY, N * sizeof( float), hipMemcpyDeviceToHost);
    hipMemcpy(_cpuZ, _gpuZ, N * sizeof( float), hipMemcpyDeviceToHost);

    // Stream out final position of each photon to file
    streamOut (&_cpuX[0], &_cpuY[0], &_cpuZ[0]);

// Free Memory
    free(_cpuX);
    free(_cpuY);
    free(_cpuZ);
    hipFree(_gpuX);
    hipFree(_gpuY);
    hipFree(_gpuZ);

    return 0;

}
void streamOut(float* _cpuX, float* _cpuY , float* _cpuZ)
{
    FILE *output;
    output = fopen("output.csv", "a");

    for (int i = 0; i < NUMBER_OF_ITERATIONS; i++)
    {
        //Checking output
        std::cout << "Movement #" << i << ":\n"
                  << "Current Position: "
                  << "( " << ray.getCurrentPos().getX() << ", "
                  << ray.getCurrentPos().getY() << ", " << ray.getCurrentPos().getZ() << " )\n"
                  << "Direction : "
                  << "( "
                  << ray.getDirection().getX() << ", " << ray.getDirection().getY() << ", " << ray.getDirection().getZ()
                  << " )\n"
                  << "Step: " << ray.getStep() << "\n"
                  << std::endl;

        // Streaming out my output in a log file
        fprintf(output, "%f,%f,%f\n", ray.getCurrentPos().getX(), ray.getCurrentPos().getY(), ray.getCurrentPos().getZ());
    }
}
