#include "hip/hip_runtime.h"
#include "Boundary.h"
using namespace std;

__device__
float Boundary::dotProduct(Point point1, Point point2){return point1.getX()*point2.getX() + point1.getY()*point2.getY() + point1.getZ()*point2.getZ();}

__device__
void Boundary::swap(float &num1, float &num2){
    float temp = num1;
    num1 = num2;
    num2 = temp;
}

__device__ Boundary::Boundary(float r, Point c){
    radius = r;
    center = c;
}

__device__ void Boundary::setRadius(float r){radius = r;}

__device__ float Boundary::getRadius() const {return radius;}

__device__ void Boundary::setCenter(Point c){center = c;}

__device__ Point Boundary::getCenter() const {return center;}

__device__ bool Boundary::isCrossed(Ray ray){
    float absDistance = (float) sqrtf((float) powf(ray.getCurrentPos().getX(),2) + (float) powf(ray.getCurrentPos().getY(),2) + (float) powf(ray.getCurrentPos().getZ(),2));
    if(absDistance >= radius){
        return true;
    } else {
        return false;
    }
}


__device__ Point Boundary::getIntersectionPoint(Ray ray){
    if(this->isCrossed(ray)){
        Point rayOrigin = ray.getPrevPos();
        Point rayDirection = ray.getDirection();
        Point p = Point((center.getX() - rayOrigin.getX()),(center.getY() - rayOrigin.getY()), (center.getZ() - rayOrigin.getZ()));
        float tca = dotProduct(p,rayDirection);
        float d2 = dotProduct(p,p) - tca * tca; 
        float thc = (float) sqrtf((float) powf(radius,2.0) - d2); 
        float t0 = tca - thc; 
        float t1 = tca + thc;
        float t;
        if (t0 > t1) swap(t0, t1);
 
        if (t0 < 0) { 
            t0 = t1; // if t0 is negative, let's use t1 instead 
        } 
        t = t0;        // this is the intersection distance from the ray origin to the hit point 

        intersectionPoint.setCoordinates((rayOrigin.getX()+rayDirection.getX()*t),(rayOrigin.getY()+rayDirection.getY()*t),(rayOrigin.getZ()+rayDirection.getZ()*t));

    }
    return intersectionPoint;
}


