#include "hip/hip_runtime.h"
#include "RandomnessGenerator.h"

__global__ void randomPoint( unsigned int seed, float* gpu_random_parameters, hiprandState_t* states){
    // initialize the random states
 hiprand_init(seed, //must be different every run so the sequence of numbers change
    blockIdx.x, // the sequence number should be different for each core 
    0, //step between random numbers
    &states[blockIdx.x]);
    gpu_random_parameters[blockIdx.x]=hiprand_uniform(&states[blockIdx.x]);
  }
  
  // Simple random number generator function, generates a float between 0.0 and 1.0
  float RandomnessGenerator::getRandomStep() const { 
//Define number of parameters to be randomly generated
    const int N = 1;

// Intialize for step value
    float step = 0.f;

// Generates array of different states
    hiprandState_t* states;
    hipMalloc((void**) &states, N * sizeof(hiprandState_t)); 

// Allocating gpu array for step generated value in kernel
    float* gpu_random_step = nullptr;
    hipMalloc((void**) &gpu_random_step,  N * sizeof(float)); 

// Calls kernel to generate random step
    randomPoint<<<N, 1>>>(time(nullptr),gpu_random_step, states);

// Copy GPU parameters into Host parameters to be able to pass it to host functions
    hipMemcpy(&step, gpu_random_step, N * sizeof(float), hipMemcpyDeviceToHost);

    return step;
 } 

// Returns a Point object that has randomized x,y and z coordinates after converting from randomized spherical coordinates
 Point RandomnessGenerator::getRandomPoint()
{
    Point point; // Instance of the Point struct to return with the random coordinates
// Define number of parameters to be randomly generated
    const int N = 2;

// Generates array of different states
    hiprandState_t* states;
    hipMalloc((void**) &states, N * sizeof(hiprandState_t)); 

// Allocating gpu array for u,v,r generated value in kernel
    float* gpu_random_parameters = nullptr;
    hipMalloc((void**) &gpu_random_parameters,  N* sizeof(float)); 

// Allocate CPU array to copy the elements of the GPU array in it to be able to stream thhem out.
    float* cpu_random_parameters = (float*)malloc(sizeof(float) * N);

// Getting random values for spherical coordinates transformation parameters
    randomPoint<<<N, 1>>>(time(nullptr),gpu_random_parameters, states);

// Copy GPU parameters into Host parameters to be able to pass it to host functions  
    hipMemcpy(cpu_random_parameters, gpu_random_parameters, N*sizeof(float), hipMemcpyDeviceToHost);

    float u = cpu_random_parameters[0] ;
    float v = cpu_random_parameters[1];
    
    float theta = 2 * M_PI * u;
    float phi = acos(1 - 2 * v);

    // Transforming into the cartesian space
    float x = sin(phi) * cos(theta);
    float y = sin(phi) * sin(theta);
    float z = cos(phi);

    point.setCoordinates(x, y, z);
  


    exportSamplingPlot(point);

    return point;

}

// A helper function to generate a csv file to use in plotting
void RandomnessGenerator::exportSamplingPlot(Point point)
{
    // For streaming out my output in a log file
    FILE *sampling;
    sampling = fopen("sampling.csv", "a");
    // Streaming out my output in a log file
    fprintf(sampling, "%f,%f,%f\n", point.getX(), point.getY(), point.getZ());
} 

