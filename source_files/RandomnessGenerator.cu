#include "hip/hip_runtime.h"
#include "RandomnessGenerator.h"
__device__ float generate( hiprandState* globalState) 
{
    hiprandState localState = globalState[blockIdx.x];
    float RANDOM = hiprand_uniform( &localState );
    globalState[blockIdx.x] = localState;
    return RANDOM;
}

  // Simple random number generator function, generates a float between 0.0 and 1.0
__device__  float RandomnessGenerator::getRandomStep( hiprandState* globalState ) { 
// Intialize for step value
    float step = 0.f;
    step = generate (globalState);
    return step;
 } 

// Returns a Point object that has randomized x,y and z coordinates after converting from randomized spherical coordinates
__device__ Point RandomnessGenerator::getRandomPoint( hiprandState* globalState )
{
    Point point; // Instance of the Point struct to return with the random coordinates

    float u = generate (globalState);
    float v = generate (globalState);
    
    float theta = 2 * M_PI * u;
    float phi = acos(1 - 2 * v);

    // Transforming into the cartesian space
    float x = sin(phi) * cos(theta);
    float y = sin(phi) * sin(theta);
    float z = cos(phi);

    point.setCoordinates(x, y, z);
  
    return point;

}