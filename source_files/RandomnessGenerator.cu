#include "hip/hip_runtime.h"
#include "RandomnessGenerator.h"
  // Simple random number generator function, generates a float between 0.0 and 1.0
__device__ float RandomnessGenerator::generate( hiprandState* globalState, int i) 
{
    hiprandState localState = globalState[i];
    float RANDOM = hiprand_uniform( &localState );
    globalState[i] = localState;
    return RANDOM;
}

__device__  float RandomnessGenerator::getRandomStep( hiprandState* globalState , int i) { 
// Intialize for step value
    float step = 0.f;
    step = generate (globalState, i);
    return step;
 } 

// Returns a Point object that has randomized x,y and z coordinates after converting from randomized spherical coordinates
__device__ Point RandomnessGenerator::getRandomPoint( hiprandState* globalState , int i)
{
    Point point; // Instance of the Point struct to return with the random coordinates

    float u = generate (globalState , i);
    float v = generate (globalState, i);
    
    float theta = 2 * M_PI * u;
    float phi = acos(1 - 2 * v);

    // Transforming into the cartesian space
    float x = sin(phi) * cos(theta);
    float y = sin(phi) * sin(theta);
    float z = cos(phi);

    point.setCoordinates(x, y, z);
  
    return point;

}