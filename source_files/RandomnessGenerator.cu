#include "hip/hip_runtime.h"
#include "RandomnessGenerator.h"

__global__ void getRandomParameters( unsigned int seed, float* gpu_random_parameters, hiprandState_t* states){
    // initialize the random states
 hiprand_init(seed, //must be different every run so the sequence of numbers change
    blockIdx.x, // the sequence number should be different for each core 
    0, //step between random numbers
    &states[blockIdx.x]);
    gpu_random_parameters[blockIdx.x]=hiprand_uniform(&states[blockIdx.x]);
  }
  
  // Simple random number generator function, generates a float between 0.0 and 1.0
__device__  float RandomnessGenerator::getRandomStep() const { 
//Define number of parameters to be randomly generated
    const int NUMBER_OF_THREADS = 1;

// Intialize for step value
    float step = 0.f;

// Generates array of different states
    hiprandState_t* states;
    hipMalloc((void**) &states, NUMBER_OF_THREADS * sizeof(hiprandState_t)); 

// Allocating gpu array for step generated value in kernel
    float* gpu_random_step = nullptr;
    hipMalloc((void**) &gpu_random_step,  NUMBER_OF_THREADS * sizeof(float)); 

// Calls kernel to generate random step
    getRandomParameters<<<NUMBER_OF_THREADS, 1>>>(time(0),gpu_random_step, states);

// Copy GPU parameters into Host parameters to be able to pass it to host functions
    hipMemcpy(&step, gpu_random_step, NUMBER_OF_THREADS * sizeof(float), hipMemcpyDeviceToHost);

    return step;
 } 

// Returns a Point object that has randomized x,y and z coordinates after converting from randomized spherical coordinates
__device__ Point RandomnessGenerator::getRandomPoint()
{
    Point point; // Instance of the Point struct to return with the random coordinates
// Define number of parameters to be randomly generated
    const int NUMBER_OF_THREADS = 2;

// Generates array of different states
    hiprandState_t* states;
    hipMalloc((void**) &states, NUMBER_OF_THREADS * sizeof(hiprandState_t)); 

// Allocating gpu array for u,v,r generated value in kernel
    float* gpu_random_parameters = nullptr;
    hipMalloc((void**) &gpu_random_parameters,  NUMBER_OF_THREADS* sizeof(float)); 

// Allocate CPU array to copy the elements of the GPU array in it to be able to stream thhem out.
    float* cpu_random_parameters = (float*)malloc(sizeof(float) * NUMBER_OF_THREADS);

// Getting random values for spherical coordinates transformation parameters
    getRandomParameters<<<NUMBER_OF_THREADS, 1>>>(time(0),gpu_random_parameters, states);

// Copy GPU parameters into Host parameters to be able to pass it to host functions  
    hipMemcpy(cpu_random_parameters, gpu_random_parameters, NUMBER_OF_THREADS*sizeof(float), hipMemcpyDeviceToHost);

    float u = cpu_random_parameters[0] ;
    float v = cpu_random_parameters[1];
    
    float theta = 2 * M_PI * u;
    float phi = acos(1 - 2 * v);

    // Transforming into the cartesian space
    float x = sin(phi) * cos(theta);
    float y = sin(phi) * sin(theta);
    float z = cos(phi);

    point.setCoordinates(x, y, z);
  


    exportSamplingPlot(point);

    return point;

}

// A helper function to generate a csv file to use in plotting
__device__ void RandomnessGenerator::exportSamplingPlot(Point point)
{
    // For streaming out my output in a log file
    FILE *sampling;
    sampling = fopen("sampling.csv", "a");
    // Streaming out my output in a log file
    fprintf(sampling, "%f,%f,%f\n", point.getX(), point.getY(), point.getZ());
} 

