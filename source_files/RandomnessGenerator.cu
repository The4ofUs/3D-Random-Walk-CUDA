#include "hip/hip_runtime.h"
#include "RandomnessGenerator.h"
  // Simple random number generator function, generates a float between 0.0 and 1.0
__device__ float RandomnessGenerator::generate( hiprandState* globalState, int i) 
{
    hiprandState localState = globalState[i];
    float random = hiprand_uniform( &localState );
    globalState[i] = localState;
    return random;
}

__device__  float RandomnessGenerator::getRandomStep( hiprandState* globalState , int i) { 
// Intialize for step value
    float step = 0.f;
    step = generate (globalState, i);
    return step;
 } 

// Returns a Point object that has randomized x,y and z coordinates after converting from randomized spherical coordinates
__device__ Point RandomnessGenerator::getRandomPoint( hiprandState* globalState , int i)
{
    float u = generate (globalState , i);
    float v = generate (globalState, i);
    
    float theta = 2 * M_PI * u;
    float phi = acos(1 - 2 * v);

    // Transforming into the cartesian space
    float x = sin(phi) * cos(theta);
    float y = sin(phi) * sin(theta);
    float z = cos(phi);

    return Point(x,y,z);
}