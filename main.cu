#include "hip/hip_runtime.h"
#include "code/headers/randomwalk.h"
#include <sstream>

float totalTime = 0.f;
int NUMBER_OF_PHOTONS = 0;
int THREADS_PER_BLOCK = 32;
float DETECTOR_RADIUS = 0.f;
Point DETECTOR_POSITION = Point();
Vector DETECTOR_LOOKAT = Vector();
float TISSUE_RADIUS = 0.f;
float TISSUE_ABSORBTION_COEFFICIENT = 0.f;
float TISSUE_SCATTERING_COEFFICIENT = 0.f;
Point TISSUE_CENTER_1 = Point();
Point TISSUE_CENTER_2 = Point();
Point SOURCE_POSITION = Point();
Vector SOURCE_LOOKAT = Vector();


void streamOut(Photon *_cpuPhotons);
char *stateToString(int state);
void printMetrics(hipEvent_t e1, hipEvent_t e2, int NUMBER_OF_BLOCKS, float &time);

__global__ void finalState(unsigned int seed, hiprandState_t *states, Photon *_gpuPhotons, Detector detector, RNG rng, Tissue tissue, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        hiprand_init(seed, idx, 0, &states[idx]);
        Photon finalState = randomWalk(states, idx, detector, rng, tissue);
        _gpuPhotons[idx] = finalState;
    }
}

bool parseUserInput(int argc, char *argv[], int &nPhotons, int &nThreads, float &dRadius, Point &dPosition, Vector &dLookAt, float &tRadius, float &tAbsorpCoeff,
    float &tScatterCoeff, Point &tCenter1, Point &tCenter2, Point &pPosition, Vector &pLookAt);

int main( int argc, char *argv[] )
{
    if (parseUserInput(argc, argv ,NUMBER_OF_PHOTONS, THREADS_PER_BLOCK, DETECTOR_RADIUS, DETECTOR_POSITION, DETECTOR_LOOKAT, TISSUE_RADIUS, 
        TISSUE_ABSORBTION_COEFFICIENT, TISSUE_SCATTERING_COEFFICIENT, TISSUE_CENTER_1, TISSUE_CENTER_2, SOURCE_POSITION, SOURCE_LOOKAT)) {
        int nBlocks = NUMBER_OF_PHOTONS / THREADS_PER_BLOCK + 1;//NUMBER_OF_PHOTONS + THREADS_PER_BLOCK - 1 / THREADS_PER_BLOCK;   
        hipMalloc((void **)&NUMBER_OF_PHOTONS, sizeof(int));
        hiprandState_t *states;
        hipMalloc((void **)&states, NUMBER_OF_PHOTONS * sizeof(hiprandState_t));
        // Allocate host memory for final positions
        Photon *_cpuPhotons = (Photon *)malloc(sizeof(Photon) * NUMBER_OF_PHOTONS);
        // Allocate device  memory for final positions
        Photon *_gpuPhotons = nullptr;
        hipMalloc((void **)&_gpuPhotons, NUMBER_OF_PHOTONS * sizeof(Photon));
        // Initialize the Boundary and the RandomNumberGenerator
        RNG rng;
        hipMalloc((void **)&rng, sizeof(RNG));
        //Boundary boundary = Boundary(BOUNDARY_RADIUS, Point());
        Detector detector = Detector(DETECTOR_RADIUS, DETECTOR_POSITION, DETECTOR_LOOKAT);
        hipMalloc((void **)&detector, sizeof(Detector));
        Tissue tissue = Tissue(TISSUE_RADIUS, TISSUE_CENTER_1, TISSUE_CENTER_2, TISSUE_ABSORBTION_COEFFICIENT, TISSUE_SCATTERING_COEFFICIENT);
        hipMalloc((void **)&tissue, sizeof(Tissue));

        unsigned int seed = time(0);
        hipMalloc((void **)&seed, sizeof(unsigned int));
        // Register cudaEvents for performance metrics purposes
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        // Start recording before the kernel call
        hipEventRecord(start); 
        // Kernel Call
        //finalPosition<<<nBlocks,THREADS_PER_BLOCK>>>(time(0), states , _gpuPoints, boundary, rng, NUMBER_OF_PHOTONS);
        finalState<<<nBlocks, THREADS_PER_BLOCK>>>(seed, states, _gpuPhotons, detector, rng, tissue, NUMBER_OF_PHOTONS);
        // Stop recording after kernel finishes execution
        hipEventRecord(stop);
        // Copy device data to host memory to stream them out
        hipMemcpy(_cpuPhotons, _gpuPhotons, NUMBER_OF_PHOTONS * sizeof(Photon), hipMemcpyDeviceToHost);
        // Synchronize before using in calculations
        hipEventSynchronize(stop);
        // Print Bandwidth
        printMetrics(start,stop, nBlocks, totalTime);
        hipEventDestroy( start );
        hipEventDestroy( stop );
        streamOut(&_cpuPhotons[0]);
        free(_cpuPhotons);
        hipFree(_gpuPhotons);
        hipFree(states);
        std::cout<< "RandomWalk.o Executed Successfully." << std::endl;
    } else {
        //std::cout<<"Invalid input: Arguments number expected = " <<  25 << ", Recieved = " << argc << std::endl;
        // Running DEFAULT RUN for debugging purposes
        // This part of the code should be erased by the end of developing phase
        // Starts Here
        int NUMBER_OF_TEST_RUNS = 10;
        for (int i= 0; i<NUMBER_OF_TEST_RUNS; i++){
            NUMBER_OF_PHOTONS = 100;
            THREADS_PER_BLOCK = 1024;
            DETECTOR_RADIUS = 10.f;
            DETECTOR_POSITION = Point(0.f, 0.f, 50.f);
            DETECTOR_LOOKAT = Vector(0.f, 0.f, -1.f);
            TISSUE_RADIUS = 100.f;
            TISSUE_ABSORBTION_COEFFICIENT = 1.f;
            TISSUE_SCATTERING_COEFFICIENT = 100.f;
            TISSUE_CENTER_1 = Point(0.f, 0.f, 50.f);
            TISSUE_CENTER_2 = Point(0.f, 0.f, -50.f);
            int nBlocks = NUMBER_OF_PHOTONS / THREADS_PER_BLOCK + 1;//NUMBER_OF_PHOTONS + THREADS_PER_BLOCK - 1 / THREADS_PER_BLOCK;   

            // Initialize the Boundary and the RandomNumberGenerator
            RNG rng;
            //Boundary boundary = Boundary(BOUNDARY_RADIUS, Point());
            Detector detector = Detector(DETECTOR_RADIUS, DETECTOR_POSITION, DETECTOR_LOOKAT);
            Tissue tissue = Tissue(TISSUE_RADIUS, TISSUE_CENTER_1, TISSUE_CENTER_2, TISSUE_ABSORBTION_COEFFICIENT, TISSUE_SCATTERING_COEFFICIENT);
            
            hiprandState_t *states;
            hipMalloc((void **)&states, NUMBER_OF_PHOTONS * sizeof(hiprandState_t));
            // Allocate host memory for final positions
            Photon *_cpuPhotons = (Photon *)malloc(sizeof(Photon) * NUMBER_OF_PHOTONS);
            // Allocate device  memory for final positions
            Photon *_gpuPhotons = nullptr;
            hipMalloc((void **)&_gpuPhotons, NUMBER_OF_PHOTONS * sizeof(Photon));
            unsigned int seed = time(0);
            // Register cudaEvents for performance metrics purposes
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            // Start recording before the kernel call
            hipEventRecord(start); 
            // Kernel Call
            //finalPosition<<<nBlocks,THREADS_PER_BLOCK>>>(time(0), states , _gpuPoints, boundary, rng, NUMBER_OF_PHOTONS);
            finalState<<<nBlocks, THREADS_PER_BLOCK>>>(seed, states, _gpuPhotons, detector, rng, tissue, NUMBER_OF_PHOTONS);
            // Stop recording after kernel finishes execution
            hipEventRecord(stop);
            // Copy device data to host memory to stream them out
            hipMemcpy(_cpuPhotons, _gpuPhotons, NUMBER_OF_PHOTONS * sizeof(Photon), hipMemcpyDeviceToHost);
            // Synchronize before using in calculations
            hipEventSynchronize(stop);
            // Print Bandwidth
            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);
            totalTime += milliseconds;
            //printMetrics(start,stop, nBlocks, totalTime);
            hipEventDestroy( start );
            hipEventDestroy( stop );
            streamOut(&_cpuPhotons[0]);
            free(_cpuPhotons);
            hipFree(_gpuPhotons);
            hipFree(states);
            //std::cout<< "Default Run Executed Successfully." << std::endl;
        }
        //Ends Here
        std::cout << "Average = " << totalTime/NUMBER_OF_TEST_RUNS << " ms" << std::endl;
    }
    return 0;
}

void streamOut(Photon *_cpuPhotons)
{
    FILE *output;
    output = fopen("output.csv", "w");
    std::string state;
    // Adding Meta-data to the output file
    /*
    *   This Particular order should be maintained if the output was to be read using the Plotter 
    */
    fprintf(output, "X, Y, Z, WEIGHT, STATE,photon_num,%i,threads_per_block,%i,detector_radius,%f,detector_pos,%f,%f,%f,detector_lookAt,%f,%f,%f,tissue_radius,%f,absorp_coeff,%f,scatter_coeff,%f,tissue_center_1,%f,%f,%f,tissue_center_2,%f,%f,%f\n"
    ,NUMBER_OF_PHOTONS, THREADS_PER_BLOCK, DETECTOR_RADIUS, DETECTOR_POSITION.x(), DETECTOR_POSITION.y(), DETECTOR_POSITION.z()
    , DETECTOR_LOOKAT.x(), DETECTOR_LOOKAT.y(), DETECTOR_LOOKAT.z(), TISSUE_RADIUS, TISSUE_ABSORBTION_COEFFICIENT
    , TISSUE_SCATTERING_COEFFICIENT, TISSUE_CENTER_1.x(), TISSUE_CENTER_1.y(), TISSUE_CENTER_1.z(), TISSUE_CENTER_2.x(), TISSUE_CENTER_2.y()
    , TISSUE_CENTER_2.z());

    for (int i = 0; i < NUMBER_OF_PHOTONS; i++)
    {
        switch (_cpuPhotons[i].getState())
        {
        case (-1):
            state = "TERMINATED";
            break;
        case (0):
            state = "ROAMING";
            break;
        case (1):
            state = "DETECTED";
            break;
        case (2):
            state = "ESCAPED";
            break;
        }
        // Streaming out my output in a log file
        fprintf(output, "%f,%f,%f,%f,%s\n", _cpuPhotons[i].getPosition().x(), _cpuPhotons[i].getPosition().y(), _cpuPhotons[i].getPosition().z(), _cpuPhotons[i].getWeight(), state.c_str());
    }
}


bool parseUserInput(int argc, char *argv[], int &nPhotons, int &nThreads, float &dRadius, Point &dPosition, Vector &dLookAt, float &tRadius, float &tAbsorpCoeff,
    float &tScatterCoeff, Point &tCenter1, Point &tCenter2, Point &pPosition, Vector &pLookAt){
        if (argc == 25) {
            nPhotons = std::atoi(argv[1]);
            nThreads = std::atoi(argv[2]);
            dRadius = std::atof(argv[3]);
            dPosition = Point(std::atof(argv[4]), std::atof(argv[5]), std::atof(argv[6]));
            dLookAt = Mathematics::calculateNormalizedVector(Vector(std::atof(argv[7]), std::atof(argv[8]), std::atof(argv[9])));
            tRadius = std::atof(argv[10]);
            tAbsorpCoeff = std::atof(argv[11]);
            tScatterCoeff = std::atof(argv[12]);
            tCenter1 = Point(std::atof(argv[13]), std::atof(argv[14]), std::atof(argv[15]));
            tCenter2 = Point(std::atof(argv[16]), std::atof(argv[17]), std::atof(argv[18]));
            pPosition = Point(std::atof(argv[19]), std::atof(argv[20]), std::atof(argv[21]));
            pLookAt = Mathematics::calculateNormalizedVector(Vector(std::atof(argv[22]), std::atof(argv[23]), std::atof(argv[24])));
            return true;
        } else {
            return false;
        }
    }

void printMetrics(hipEvent_t e1, hipEvent_t e2, int NUMBER_OF_BLOCKS, float &time){
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, e1, e2);
    std::cout << milliseconds << std::endl;
    time += milliseconds;
    /* printf("    ---------------------------------------------------------------     \n");
    printf("    Number of Photons   |   Blocks Used  |   Threads per Block used     \n");
    printf("    ---------------------------------------------------------------     \n");
    printf("            %i          |       %i       |              %i              \n", NUMBER_OF_PHOTONS, NUMBER_OF_BLOCKS, THREADS_PER_BLOCK);
    printf("    ---------------------------------------------------------------     \n");
    printf("Elapsed time (ms): %f\n", milliseconds);
    printf("Theoretical Bandwidth (GB/s): %f\n", 2500*1e6*(128/8)*2/1e9);
    printf("Effective Bandwidth (GB/s): %f\n", NUMBER_OF_PHOTONS*4*3/milliseconds/1e6); */
}