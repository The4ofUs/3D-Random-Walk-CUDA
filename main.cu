#include "hip/hip_runtime.h"
#include "code/headers/randomwalk.h"
#include <sstream>


/*
TO-DO:
    - Loop to execute multiple runs on the same parameters and extract average for every metric produces
*/


unsigned int NUMBER_OF_TEST_RUNS = 100;
int NUMBER_OF_PHOTONS = 0;
int THREADS_PER_BLOCK = 32;
float DETECTOR_RADIUS = 0.f;
Point DETECTOR_POSITION = Point();
Vector DETECTOR_LOOKAT = Vector();
float TISSUE_RADIUS = 0.f;
float TISSUE_ABSORBTION_COEFFICIENT = 0.f;
float TISSUE_SCATTERING_COEFFICIENT = 0.f;
Point TISSUE_CENTER_1 = Point();
Point TISSUE_CENTER_2 = Point();
Point SOURCE_POSITION = Point();
Vector SOURCE_LOOKAT = Vector();



void streamOut(Photon *_cpuPhotons);
char *stateToString(int state);
void printMetrics(hipEvent_t e1, hipEvent_t e2, int NUMBER_OF_BLOCKS);

__global__ void finalState(unsigned int seed, hiprandState_t *states, Photon *_gpuPhotons, Detector dectector, RNG rng, Tissue tissue, int n)
{  
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        hiprand_init(seed, idx, 0, &states[idx]);
        Photon finalState = randomWalk(states, idx, dectector, rng,  tissue);
        _gpuPhotons[idx] = finalState;        
    }
}

bool parseUserInput(int argc, char *argv[], int &nPhotons, int &nThreads, float &dRadius, Point &dPosition, Vector &dLookAt, float &tRadius, float &tAbsorpCoeff,
    float &tScatterCoeff, Point &tCenter1, Point &tCenter2, Point &pPosition, Vector &pLookAt);

int main( int argc, char *argv[] )
{
    if (parseUserInput(argc, argv ,NUMBER_OF_PHOTONS, THREADS_PER_BLOCK, DETECTOR_RADIUS, DETECTOR_POSITION, DETECTOR_LOOKAT, TISSUE_RADIUS, 
        TISSUE_ABSORBTION_COEFFICIENT, TISSUE_SCATTERING_COEFFICIENT, TISSUE_CENTER_1, TISSUE_CENTER_2, SOURCE_POSITION, SOURCE_LOOKAT)) {
        int nBlocks = NUMBER_OF_PHOTONS + THREADS_PER_BLOCK - 1 / THREADS_PER_BLOCK;//NUMBER_OF_PHOTONS + THREADS_PER_BLOCK - 1 / THREADS_PER_BLOCK;   
        //hipMalloc((void **)&NUMBER_OF_PHOTONS, sizeof(int));
        hiprandState_t *states;
        hipMalloc((void **)&states, NUMBER_OF_PHOTONS * sizeof(hiprandState_t));
        // Allocate host memory for final positions
        Photon *_cpuPhotons = (Photon *)malloc(sizeof(Photon) * NUMBER_OF_PHOTONS);
        // Allocate device  memory for final positions
        Photon *_gpuPhotons = nullptr;
        hipMalloc((void **)&_gpuPhotons, NUMBER_OF_PHOTONS * sizeof(Photon));
        // Initialize the Boundary and the RandomNumberGenerator
        RNG rng;
        Detector detector = Detector(DETECTOR_RADIUS, DETECTOR_POSITION, DETECTOR_LOOKAT);
        Tissue tissue = Tissue(TISSUE_RADIUS, TISSUE_CENTER_1, TISSUE_CENTER_2, TISSUE_ABSORBTION_COEFFICIENT, TISSUE_SCATTERING_COEFFICIENT);
        unsigned int seed = time(0);
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        // Start recording before the kernel call
        hipEventRecord(start); 
        // Kernel Call
        //finalPosition<<<nBlocks,THREADS_PER_BLOCK>>>(time(0), states , _gpuPoints, boundary, rng, NUMBER_OF_PHOTONS);
        finalState<<<nBlocks, THREADS_PER_BLOCK>>>(seed, states, _gpuPhotons, detector, rng, tissue, NUMBER_OF_PHOTONS);
        // Stop recording after kernel finishes execution
        hipEventRecord(stop);
        // Copy device data to host memory to stream them out
        hipMemcpy(_cpuPhotons, _gpuPhotons, NUMBER_OF_PHOTONS * sizeof(Photon), hipMemcpyDeviceToHost);
        // Synchronize before using in calculations
        hipEventSynchronize(stop);
        // Calculate the total number of operations done by all photons
        for (int i =0; i < NUMBER_OF_PHOTONS; i++){
            std::cout << _cpuPhotons[i].getLifetime() << '\n';
            NUMBER_OF_OPERATIONS += _cpuPhotons[i].getLifetime();
        }
        // Print Bandwidth
        printMetrics(start,stop, nBlocks);
        hipEventDestroy( start );
        hipEventDestroy( stop );
        streamOut(&_cpuPhotons[0]);
        free(_cpuPhotons);
        hipFree(_gpuPhotons);
        hipFree(states);
        std::cout<< "RandomWalk.o Executed Successfully." << std::endl;
    } else { 
        std::cout<<"Invalid input: Arguments number expected = " <<  25 << ", Recieved = " << argc << std::endl;
        std::cout<<"Initiating Default Walk.."<< std::endl;
        // Running DEFAULT RUN for debugging purposes
        // This part of the code should be erased by the end of developing phase
        // Starts Here
        NUMBER_OF_PHOTONS = 10;
        THREADS_PER_BLOCK = 1024;
        DETECTOR_RADIUS = 10.f;
        DETECTOR_POSITION = Point(0.f, 0.f, 50.f);
        DETECTOR_LOOKAT = Vector(0.f, 0.f, -1.f);
        TISSUE_RADIUS = 100.f;
        TISSUE_ABSORBTION_COEFFICIENT = 1.f;
        TISSUE_SCATTERING_COEFFICIENT = 100.f;
        TISSUE_CENTER_1 = Point(0.f, 0.f, 50.f);
        TISSUE_CENTER_2 = Point(0.f, 0.f, -50.f);
        RNG rng;
        Detector detector = Detector(DETECTOR_RADIUS, DETECTOR_POSITION, DETECTOR_LOOKAT);
        Tissue tissue = Tissue(TISSUE_RADIUS, TISSUE_CENTER_1, TISSUE_CENTER_2, TISSUE_ABSORBTION_COEFFICIENT, TISSUE_SCATTERING_COEFFICIENT);
        int nBlocks = NUMBER_OF_PHOTONS / THREADS_PER_BLOCK + 1;//NUMBER_OF_PHOTONS + THREADS_PER_BLOCK - 1 / THREADS_PER_BLOCK;   
        hiprandState_t *states;
        hipMalloc((void **)&states, NUMBER_OF_PHOTONS * sizeof(hiprandState_t));
        // Allocate host memory for final positions
        Photon *_cpuPhotons = (Photon *)malloc(sizeof(Photon) * NUMBER_OF_PHOTONS);
        // Allocate device  memory for final positions
        Photon *_gpuPhotons = nullptr;
        hipMallocManaged((void **)&_gpuPhotons, NUMBER_OF_PHOTONS * sizeof(Photon));
        // Register cudaEvents for performance metrics purposes
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        // Start recording before the kernel call
        hipEventRecord(start); 
        // Kernel Call
        //finalPosition<<<nBlocks,THREADS_PER_BLOCK>>>(time(0), states , _gpuPoints, boundary, rng, NUMBER_OF_PHOTONS);
        finalState<<<nBlocks, THREADS_PER_BLOCK>>>(time(0), states, _gpuPhotons, detector, rng, tissue, NUMBER_OF_PHOTONS);
        // Stop recording after kernel finishes execution
        hipEventRecord(stop);
        // Copy device data to host memory to stream them out
        hipMemcpy(_cpuPhotons, _gpuPhotons, NUMBER_OF_PHOTONS * sizeof(Photon), hipMemcpyDeviceToHost);
        // Synchronize before using in calculations
        hipEventSynchronize(stop);
        // Calculate the total number of operations done by all photons
        for (int i =0; i < NUMBER_OF_PHOTONS; i++){
            NUMBER_OF_OPERATIONS += _cpuPhotons[i].getLifetime();
        }
        // Print Bandwidth
        printMetrics(start,stop, nBlocks);
        hipEventDestroy( start );
        hipEventDestroy( stop );
        streamOut(&_gpuPhotons[0]);
        free(_cpuPhotons);
        hipFree(_cpuPhotons);
        hipFree(states);
        std::cout<< "Default Run Executed Successfully." << std::endl;

        //Ends Here
    }
    return 0;
}

void streamOut(Photon *_cpuPhotons)
{
    FILE *output;
    output = fopen("output.csv", "w");
    std::string state;
    // Adding Meta-data to the output file
    /*
    *   This Particular order should be maintained if the output was to be read using the Plotter 
    */
     fprintf(output, "X, Y, Z, WEIGHT, STATE,photon_num,%i,threads_per_block,%i,detector_radius,%f,detector_pos,%f,%f,%f,detector_lookAt,%f,%f,%f,tissue_radius,%f,absorp_coeff,%f,scatter_coeff,%f,tissue_center_1,%f,%f,%f,tissue_center_2,%f,%f,%f\n"
    ,NUMBER_OF_PHOTONS, THREADS_PER_BLOCK, DETECTOR_RADIUS, DETECTOR_POSITION.x(), DETECTOR_POSITION.y(), DETECTOR_POSITION.z()
    , DETECTOR_LOOKAT.x(), DETECTOR_LOOKAT.y(), DETECTOR_LOOKAT.z(), TISSUE_RADIUS, TISSUE_ABSORBTION_COEFFICIENT
    , TISSUE_SCATTERING_COEFFICIENT, TISSUE_CENTER_1.x(), TISSUE_CENTER_1.y(), TISSUE_CENTER_1.z(), TISSUE_CENTER_2.x(), TISSUE_CENTER_2.y()
    , TISSUE_CENTER_2.z()); 

    for (int i = 0; i < NUMBER_OF_PHOTONS; i++)
    {
        switch (_cpuPhotons[i].getState())
        {
        case (-1):
            state = "TERMINATED";
            break;
        case (0):
            state = "ROAMING";
            break;
        case (1):
            state = "DETECTED";
            break;
        case (2):
            state = "ESCAPED";
            break;
        }
        // Streaming out my output in a log file
        fprintf(output, "%f,%f,%f,%f,%s\n", _cpuPhotons[i].getPosition().x(), _cpuPhotons[i].getPosition().y(), _cpuPhotons[i].getPosition().z(), _cpuPhotons[i].getWeight(), state.c_str());
    }
} 


bool parseUserInput(int argc, char *argv[], int &nPhotons, int &nThreads, float &dRadius, Point &dPosition, Vector &dLookAt, float &tRadius, float &tAbsorpCoeff,
    float &tScatterCoeff, Point &tCenter1, Point &tCenter2, Point &pPosition, Vector &pLookAt){
        if (argc == 25) {
            nPhotons = std::atoi(argv[1]);
            nThreads = std::atoi(argv[2]);
            dRadius = std::atof(argv[3]);
            dPosition = Point(std::atof(argv[4]), std::atof(argv[5]), std::atof(argv[6]));
            dLookAt = Mathematics::calculateNormalizedVector(Vector(std::atof(argv[7]), std::atof(argv[8]), std::atof(argv[9])));
            tRadius = std::atof(argv[10]);
            tAbsorpCoeff = std::atof(argv[11]);
            tScatterCoeff = std::atof(argv[12]);
            tCenter1 = Point(std::atof(argv[13]), std::atof(argv[14]), std::atof(argv[15]));
            tCenter2 = Point(std::atof(argv[16]), std::atof(argv[17]), std::atof(argv[18]));
            pPosition = Point(std::atof(argv[19]), std::atof(argv[20]), std::atof(argv[21]));
            pLookAt = Mathematics::calculateNormalizedVector(Vector(std::atof(argv[22]), std::atof(argv[23]), std::atof(argv[24])));
            return true;
        } else {
            return false;
        }
    } 

void printMetrics(hipEvent_t e1, hipEvent_t e2, int NUMBER_OF_BLOCKS){
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, e1, e2);
    printf("--------------------------------------------------------------------------------------------\n");
    printf("Number of Photons: %i\n", NUMBER_OF_PHOTONS);
    printf("Blocks Used: %i\n", NUMBER_OF_BLOCKS);
    printf("Threads/Block used: %i\n", THREADS_PER_BLOCK);
    printf("Elapsed time (ms): %f\n", milliseconds);
    printf("Total # of operations: %i\n", NUMBER_OF_OPERATIONS);
    printf("Theoretical Bandwidth (GB/s): %f\n", 1122*1e6*(64/8)*2/1e9); // 10e6 becuse core speed is already in MHz
    printf("Effective Bandwidth (GB/s): %f\n", (NUMBER_OF_PHOTONS*sizeof(Photon)*1/1e6)/milliseconds); // 10e6 because time is in milliseconds
    printf("Computational Throughput (GB/s): %f\n", NUMBER_OF_OPERATIONS/ milliseconds/1e6); //Giga-FLoating-point Operations per second, 10e6 because time is in milliseconds
    printf("--------------------------------------------------------------------------------------------\n");
}
