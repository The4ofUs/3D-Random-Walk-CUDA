#include "hip/hip_runtime.h"
#include "code/headers/randomwalk.h"
#include "Network/Client/Headers/socket.h"
#include <QDebug>
#include <QVector>
#include <unistd.h>
#define THREADS_PER_BLOCK 1024
#define DETECTOR_LOOK_DOWNWARDS Vector(0.f, 0.f, -1.f)

/*#define NUMBER_OF_PHOTONS 10
#define DETECTOR_RADIUS 10.f
#define DETECTOR_POSITION Point(0.f, 0.f, 50.f)
#define TISSUE_RADIUS 100.f
#define TISSUE_ABSORBTION_COEFFICIENT 1.f
#define TISSUE_SCATTERING_COEFFICIENT 100.f
#define TISSUE_CENTER_1 Point(0.f, 0.f, 50.f)
#define TISSUE_CENTER_2 Point(0.f, 0.f, -50.f)*/
int numberOfPhotons;
float detectorRadius;
float tissueRadius;
float tissueAbsCoeff;
float tissueScatCoeff;
Point detectorPosition;
Point tissueFirstCenter;
Point tissueSecondCenter;
QVector<Photon> photons;
QVector<Photon> totalSentResults;
QVector<float> X;
QVector<float> Y;
QVector<float> Z;
QVector<float> W;
QVector<int> ST;

bool newBatchAvailable;
char *stateToString(int state);
void sendResults(Photon *_cpuPhotons);
void requestParameters();
void populateParameters(QVector<float> parameters);
void applyMC(Detector detector, Tissue tissue);
void askForNewBatch();
void appendToVectors(QVector<Photon> Photons);
void streamOut(QVector<Photon> results);

__global__ void finalState(unsigned int seed, hiprandState_t *states, Photon *_gpuPhotons, Detector detector, RNG rng, Tissue tissue, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        hiprand_init(seed, idx, 0, &states[idx]);
        Photon finalState = randomWalk(states, idx, detector, rng, tissue);
        _gpuPhotons[idx] = finalState;
    }
}


int main()
{
    requestParameters();
    //Boundary boundary = Boundary(BOUNDARY_RADIUS, Point());
    Detector detector = Detector(detectorRadius, detectorPosition, DETECTOR_LOOK_DOWNWARDS);
    Tissue tissue = Tissue(tissueRadius, tissueFirstCenter, tissueSecondCenter, tissueAbsCoeff, tissueScatCoeff);
    while(newBatchAvailable){
        applyMC(detector, tissue);
        usleep(1000000);
    }

   // applyMC();
    streamOut(totalSentResults);
    return 0;
}

void applyMC(Detector detector, Tissue tissue){
    int nBlocks = numberOfPhotons / THREADS_PER_BLOCK + 1;
    hiprandState_t *states;
    hipMalloc((void **)&states, numberOfPhotons * sizeof(hiprandState_t));
    // Allocate host memory for final positions
    Photon *_cpuPhotons = (Photon *)malloc(sizeof(Photon) * numberOfPhotons);
    // Allocate device  memory for final positions
    Photon *_gpuPhotons = nullptr;
    hipMalloc((void **)&_gpuPhotons, numberOfPhotons * sizeof(Photon));
    // Initialize the Boundary and the RandomNumberGenerator
    RNG rng;

    // Kernel Call
    //finalPosition<<<nBlocks,THREADS_PER_BLOCK>>>(time(0), states , _gpuPoints, boundary, rng, NUMBER_OF_PHOTONS);
    finalState<<<nBlocks, THREADS_PER_BLOCK>>>(time(0), states, _gpuPhotons, detector, rng, tissue, numberOfPhotons);
    // Copy device data to host memory to stream them out
    hipMemcpy(_cpuPhotons, _gpuPhotons, numberOfPhotons * sizeof(Photon), hipMemcpyDeviceToHost);
    //streamOut(&_cpuPhotons[0]);
    sendResults(&_cpuPhotons[0]);
    askForNewBatch();
    free(_cpuPhotons);
    hipFree(_gpuPhotons);
}





void sendResults(Photon *_cpuPhotons){

    QVector<Photon> vectorOfPhotons;
    for (int i = 0; i < numberOfPhotons; i++)
    {
        vectorOfPhotons.push_back(_cpuPhotons[i]);
    }
    socket *newSocket =new socket();
    newSocket->queryType="prepareForReceiving";
    newSocket->socket::getVectorOfPhotons(vectorOfPhotons);
    newSocket->createSocket();
    appendToVectors(vectorOfPhotons);

}


void requestParameters(){
    socket *newSocket =new socket();
    newSocket->queryType="requestParameters";
    newSocket->createSocket();
    QVector<float> parameters = newSocket->getParameters();
    //qDebug()<<parameters<<parameters.size();
    if(parameters.size()>0){
        populateParameters(parameters);
        newBatchAvailable = true;
    }
}

void askForNewBatch(){
    socket *newSocket =new socket();
    newSocket->queryType="requestBatch";
    newSocket->createSocket();
    numberOfPhotons = newSocket->numberOfPhotons;
    if (numberOfPhotons==0){
        newBatchAvailable = false;
    }
}

void populateParameters(QVector<float> parameters){
    numberOfPhotons = (int) parameters[0];
    detectorRadius  = parameters[1];
    detectorPosition = Point(parameters[2],parameters[3],parameters[4]);
    tissueRadius = parameters[5];
    tissueAbsCoeff = parameters[6];
    tissueScatCoeff = parameters[7];
    tissueFirstCenter =  Point(parameters[8], parameters[9], parameters[10]);
    tissueSecondCenter = Point(parameters[11],parameters[12],parameters[13]);
    qDebug()<<"Parameters are received";
}

// Append photons of each patch to a vector to stream the whole photons at the client side in 1 file
// This is used in testing phase only
void appendToVectors(QVector<Photon> Photons){
  totalSentResults.append(Photons);
  qDebug()<<"total sent"<<totalSentResults.size();
}



void streamOut(QVector<Photon> results){
    FILE *output;
    output = fopen("clientSentPhotons.csv", "w");
    std::string state;
    fprintf(output, "X,Y,Z,WEIGHT,STATE\n");

    for (int i = 0; i < results.size(); i++)
    {
        switch (results[i].getState())
        {
        case (-1):
            state = "TERMINATED";
            break;
        case (0):
            state = "ROAMING";
            break;
        case (1):
            state = "DETECTED";
            break;
        case (2):
            state = "ESCAPED";
            break;
        }

        // Streaming out my output in a log file
        fprintf(output, "%f,%f,%f,%f,%s\n", results[i].getPosition().x(), results[i].getPosition().y(), results[i].getPosition().z(), results[i].getWeight(), state.c_str());
       // qDebug()<<results[i].getPosition().x()<< results[i].getPosition().y()<< results[i].getPosition().z()<< results[i].getWeight()<< state.c_str();

    }
    fclose(output);
}
