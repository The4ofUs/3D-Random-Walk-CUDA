#include "hip/hip_runtime.h"
#include "code/headers/randomwalk.h"

#define NUMBER_OF_PHOTONS 1000
#define THREADS_PER_BLOCK 1024
#define DETECTOR_RADIUS 10.f
#define DETECTOR_POSITION Point(0.f, 0.f, 50.f)
#define DETECTOR_LOOK_DOWNWARDS Vector(0.f, 0.f, -1.f)
#define TISSUE_RADIUS 100.f
#define TISSUE_ABSORBTION_COEFFICIENT 1.f
#define TISSUE_SCATTERING_COEFFICIENT 100.f
#define TISSUE_CENTER_1 Point(0.f, 0.f, 50.f)
#define TISSUE_CENTER_2 Point(0.f, 0.f, -50.f)

void streamOut(Photon *_cpuPhotons);
char *stateToString(int state);

__global__ void finalState(unsigned int seed, hiprandState_t *states, Photon *_gpuPhotons, Detector detector, RNG rng, Tissue tissue, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        hiprand_init(seed, idx, 0, &states[idx]);
        Photon finalState = randomWalk(states, idx, detector, rng, tissue);
        _gpuPhotons[idx] = finalState;
    }
}

int main()
{
    int nBlocks = NUMBER_OF_PHOTONS / THREADS_PER_BLOCK + 1;
    hiprandState_t *states;
    hipMalloc((void **)&states, NUMBER_OF_PHOTONS * sizeof(hiprandState_t));
    // Allocate host memory for final positions
    Photon *_cpuPhotons = (Photon *)malloc(sizeof(Photon) * NUMBER_OF_PHOTONS);
    // Allocate device  memory for final positions
    Photon *_gpuPhotons = nullptr;
    hipMalloc((void **)&_gpuPhotons, NUMBER_OF_PHOTONS * sizeof(Photon));
    // Initialize the Boundary and the RandomNumberGenerator
    RNG rng;
    //Boundary boundary = Boundary(BOUNDARY_RADIUS, Point());
    Detector detector = Detector(DETECTOR_RADIUS, DETECTOR_POSITION, DETECTOR_LOOK_DOWNWARDS);
    Tissue tissue = Tissue(TISSUE_RADIUS, TISSUE_CENTER_1, TISSUE_CENTER_2, TISSUE_ABSORBTION_COEFFICIENT, TISSUE_SCATTERING_COEFFICIENT);
    // Kernel Call
    //finalPosition<<<nBlocks,THREADS_PER_BLOCK>>>(time(0), states , _gpuPoints, boundary, rng, NUMBER_OF_PHOTONS);
    finalState<<<nBlocks, THREADS_PER_BLOCK>>>(time(0), states, _gpuPhotons, detector, rng, tissue, NUMBER_OF_PHOTONS);
    // Copy device data to host memory to stream them out
    hipMemcpy(_cpuPhotons, _gpuPhotons, NUMBER_OF_PHOTONS * sizeof(Photon), hipMemcpyDeviceToHost);
    streamOut(&_cpuPhotons[0]);
    free(_cpuPhotons);
    hipFree(_gpuPhotons);
    return 0;
}

void streamOut(Photon *_cpuPhotons)
{
    FILE *output;
    output = fopen("output.csv", "w");
    std::string state;
    // Adding Meta-data to the output file
    /*
    *   This Particular order should be maintained if the output was to be read using the Plotter 
    */
    fprintf(output, "X, Y, Z, WEIGHT, STATE,photon_num,%i,threads_per_block,%i,detector_radius,%f,detector_pos,%f,%f,%f,detector_lookAt,%f,%f,%f,tissue_radius,%f,absorp_coeff,%f,scatter_coeff,%f,tissue_center_1,%f,%f,%f,tissue_center_2,%f,%f,%f\n"
    ,NUMBER_OF_PHOTONS, THREADS_PER_BLOCK, DETECTOR_RADIUS, DETECTOR_POSITION.x(), DETECTOR_POSITION.y(), DETECTOR_POSITION.z()
    , DETECTOR_LOOK_DOWNWARDS.x(), DETECTOR_LOOK_DOWNWARDS.y(), DETECTOR_LOOK_DOWNWARDS.z(), TISSUE_RADIUS, TISSUE_ABSORBTION_COEFFICIENT
    , TISSUE_SCATTERING_COEFFICIENT, TISSUE_CENTER_1.x(), TISSUE_CENTER_1.y(), TISSUE_CENTER_1.z(), TISSUE_CENTER_2.x(), TISSUE_CENTER_2.y()
    , TISSUE_CENTER_2.z());

    for (int i = 0; i < NUMBER_OF_PHOTONS; i++)
    {
        switch (_cpuPhotons[i].getState())
        {
        case (-1):
            state = "TERMINATED";
            break;
        case (0):
            state = "ROAMING";
            break;
        case (1):
            state = "DETECTED";
            break;
        case (2):
            state = "ESCAPED";
            break;
        }
        // Streaming out my output in a log file
        fprintf(output, "%f,%f,%f,%f,%s\n", _cpuPhotons[i].getPosition().x(), _cpuPhotons[i].getPosition().y(), _cpuPhotons[i].getPosition().z(), _cpuPhotons[i].getWeight(), state.c_str());
    }
}
