#include "hip/hip_runtime.h"
#include "code/headers/randomwalk.h"
#include "Network/Client/Headers/socket.h"
#include <QDebug>
#include <QVector>
#define NUMBER_OF_PHOTONS 10000
#define THREADS_PER_BLOCK 1024
#define DETECTOR_RADIUS 10.f
#define DETECTOR_POSITION Point(0.f, 0.f, 50.f)
#define DETECTOR_LOOK_DOWNWARDS Vector(0.f, 0.f, -1.f)
#define TISSUE_RADIUS 100.f
#define TISSUE_ABSORBTION_COEFFICIENT 1.f
#define TISSUE_SCATTERING_COEFFICIENT 100.f
#define TISSUE_CENTER_1 Point(0.f, 0.f, 50.f)
#define TISSUE_CENTER_2 Point(0.f, 0.f, -50.f)
QVector<Photon> photons;
void streamOut(Photon *_cpuPhotons);
char *stateToString(int state);
void writeToSocket(Photon *_cpuPhotons);
__global__ void finalState(unsigned int seed, hiprandState_t *states, Photon *_gpuPhotons, Detector detector, RNG rng, Tissue tissue, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        hiprand_init(seed, idx, 0, &states[idx]);
        Photon finalState = randomWalk(states, idx, detector, rng, tissue);
        _gpuPhotons[idx] = finalState;
    }
}

int main()
{
    int nBlocks = NUMBER_OF_PHOTONS / THREADS_PER_BLOCK + 1;
    hiprandState_t *states;
    hipMalloc((void **)&states, NUMBER_OF_PHOTONS * sizeof(hiprandState_t));
    // Allocate host memory for final positions
    Photon *_cpuPhotons = (Photon *)malloc(sizeof(Photon) * NUMBER_OF_PHOTONS);
    // Allocate device  memory for final positions
    Photon *_gpuPhotons = nullptr;
    hipMalloc((void **)&_gpuPhotons, NUMBER_OF_PHOTONS * sizeof(Photon));
    // Initialize the Boundary and the RandomNumberGenerator
    RNG rng;
    //Boundary boundary = Boundary(BOUNDARY_RADIUS, Point());
    Detector detector = Detector(DETECTOR_RADIUS, DETECTOR_POSITION, DETECTOR_LOOK_DOWNWARDS);
    Tissue tissue = Tissue(TISSUE_RADIUS, TISSUE_CENTER_1, TISSUE_CENTER_2, TISSUE_ABSORBTION_COEFFICIENT, TISSUE_SCATTERING_COEFFICIENT);
    // Kernel Call
    //finalPosition<<<nBlocks,THREADS_PER_BLOCK>>>(time(0), states , _gpuPoints, boundary, rng, NUMBER_OF_PHOTONS);
    finalState<<<nBlocks, THREADS_PER_BLOCK>>>(time(0), states, _gpuPhotons, detector, rng, tissue, NUMBER_OF_PHOTONS);
    // Copy device data to host memory to stream them out
    hipMemcpy(_cpuPhotons, _gpuPhotons, NUMBER_OF_PHOTONS * sizeof(Photon), hipMemcpyDeviceToHost);
    streamOut(&_cpuPhotons[0]);
    writeToSocket(&_cpuPhotons[0]);
    free(_cpuPhotons);
    hipFree(_gpuPhotons);
    return 0;
}


/*
 *
 *
 * Write To socket
 *
 */
void writeToSocket(Photon *_cpuPhotons){

QVector<Photon> vectorOfPhotons;
    for (int i = 0; i < NUMBER_OF_PHOTONS; i++)
    {
     //newSocket->getPhotonFinalState(_cpuPhotons[i].getPosition().x(), _cpuPhotons[i].getPosition().y(), _cpuPhotons[i].getPosition().z(), _cpuPhotons[i].getWeight(),_cpuPhotons[i].getState());
    vectorOfPhotons.push_back(_cpuPhotons[i]);
    qDebug()<<vectorOfPhotons.size()<<vectorOfPhotons[i].getPosition().x();

    }
    socket *newSocket =new socket();
    newSocket->socket::getVectorOfPhotons(vectorOfPhotons);
    newSocket->createSocket();

}



void streamOut(Photon *_cpuPhotons)
{
    FILE *output;
    output = fopen("output.csv", "w");
    std::string state;
    fprintf(output, "X,Y,Z,WEIGHT,STATE\n");
    for (int i = 0; i < NUMBER_OF_PHOTONS; i++)
    {
        switch (_cpuPhotons[i].getState())
        {
        case (-1):
            state = "TERMINATED";
            break;
        case (0):
            state = "ROAMING";
            break;
        case (1):
            state = "DETECTED";
            break;
        case (2):
            state = "ESCAPED";
            break;
        }
        // Streaming out my output in a log file
        fprintf(output, "%f,%f,%f,%f,%s\n", _cpuPhotons[i].getPosition().x(), _cpuPhotons[i].getPosition().y(), _cpuPhotons[i].getPosition().z(), _cpuPhotons[i].getWeight(), state.c_str());


    }
}
