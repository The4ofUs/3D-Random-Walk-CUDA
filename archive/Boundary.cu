#include "hip/hip_runtime.h"
#include "../headers/Boundary.h"
using namespace std;

__device__ __host__ Boundary::Boundary(float r, Point c)
{
    _radius = r;
    _center = c;
}

__device__ void Boundary::setRadius(float r) { _radius = r; }
__device__ float Boundary::getRadius() const { return _radius; }
__device__ void Boundary::setCenter(Point c) { _center = c; }
__device__ Point Boundary::getCenter() const { return _center; }
__device__ bool Boundary::isHit(Ray ray)
{
    float absDistance = (float)sqrtf((float)powf(ray.getTip().x(), 2) + (float)powf(ray.getTip().y(), 2) + (float)powf(ray.getTip().z(), 2));
    if (absDistance >= _radius)
    {
        return true;
    }
    else
    {
        return false;
    }
}

__device__ Point Boundary::getIntersectionPoint(Ray ray)
{
    /**
            P(t) = A + tB
            P(t) is a point on the ray 
            A is the ray origin
            B is the ray direction
            t is a parameter used to move away from ray origin
            S = P - Center
            ||S||^2 = r^2
            Sphere: dot(S,S) = r^2
            Ray: P(t) = A + tB
            Combined: dot((A + tB - Center),(A + tB - Center)) = r^2
            in Quadratic form: t^2.dot(B,B) + 2t.dot(B, A - C) + dot(A - C, A - C) - r^2 = 0
            let a = dot(B,B)
                b = 2.dot(B, A - C)
                c = dot(A - C, A - C) - r^2
            t1, t2 = (-b (+/-) sqrt(b^2 - 4ac) / 2a)
        */
    Point A = ray.getOrigin();
    Vector B = ray.getDirection();
    Vector S = A + _center;
    Vector A_C = A - _center;
    float a = dotProduct(B, B);
    float b = 2.0 * dotProduct(B, A_C);
    float c = A_C.dot(A_C) - _radius * _radius;
    float discriminant = b * b - 4 * a * c;
    float t1 = (-b + sqrtf(discriminant)) / (2.0 * a);
    float t2 = (-b - sqrtf(discriminant)) / (2.0 * a);
    float t;
    if (t1 < 0)
    {
        t = t2;
    }
    else
    {
        t = t1;
    }
    return (A + B * t);
}
