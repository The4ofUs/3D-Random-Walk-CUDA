#include "hip/hip_runtime.h"
#include "code/headers/randomwalk.h"
#include <sstream>

//#define NUMBER_OF_PHOTONS 1000
#define THREADS_PER_BLOCK 1024
#define DETECTOR_RADIUS 10.f
#define DETECTOR_POSITION Point(0.f, 0.f, 50.f)
#define DETECTOR_LOOKAT Vector(0.f, 0.f, -1.f)
#define TISSUE_RADIUS 1000.f
//#define TISSUE_ABSORBTION_COEFFICIENT 1.f
#define TISSUE_SCATTERING_COEFFICIENT 100.f
#define TISSUE_CENTER_1 Point(0.f, 0.f, 50.f)
#define TISSUE_CENTER_2 Point(0.f, 0.f, -50.f)

__global__ void finalState(unsigned int seed, hiprandState_t *states, Photon *_gpuPhotons, Detector detector, RNG rng, Tissue tissue, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        hiprand_init(seed, idx, 0, &states[idx]);
        Photon finalState = randomWalk(states, idx, detector, rng, tissue);
        _gpuPhotons[idx] = finalState;
    }
}

void streamOut(int a, float c, float d, unsigned long long e, unsigned long long f, float g, float h, float i, int j, int k)
{
    FILE *output;
    output = fopen("metrics.csv", "a");
    if (output != NULL){
        fprintf(output, "%i, %f, %f, %llu, %llu, %f, %f, %f, %i, %i\n", a, c, d, e, f, g, h, i ,j ,k);
        fclose(output);
    } else {
        std::cout<<"Failed to open file, retrying!" << std::endl;
        streamOut(a,c,d,e,f,g,h,i,j,k);
    }
}

void run(int n, float Ma){
    int NUMBER_OF_PHOTONS = n;
    float TISSUE_ABSORBTION_COEFFICIENT  = Ma;
    unsigned int NUMBER_OF_TEST_RUNS = 100;
    float totalTime = 0.f;
    unsigned long long totalLifetime = 0;
    float detected = 0.f;
    float terminated = 0.f;
    float escaped = 0.f;
    int nBlocks = NUMBER_OF_PHOTONS + THREADS_PER_BLOCK - 1 / THREADS_PER_BLOCK;
    for (int i= 0; i<NUMBER_OF_TEST_RUNS; i++){
        RNG rng;
        Detector detector = Detector(DETECTOR_RADIUS, DETECTOR_POSITION, DETECTOR_LOOKAT);
        Tissue tissue = Tissue(TISSUE_RADIUS, TISSUE_CENTER_1, TISSUE_CENTER_2, TISSUE_ABSORBTION_COEFFICIENT, TISSUE_SCATTERING_COEFFICIENT);
        hiprandState_t *states;
        hipMalloc((void **)&states, NUMBER_OF_PHOTONS * sizeof(hiprandState_t));
        Photon *_cpuPhotons = (Photon *)malloc(sizeof(Photon) * NUMBER_OF_PHOTONS);
        Photon *_gpuPhotons = nullptr;
        hipMalloc((void **)&_gpuPhotons, NUMBER_OF_PHOTONS * sizeof(Photon));
        unsigned int seed = time(0);
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start); 
        finalState<<<nBlocks, THREADS_PER_BLOCK>>>(seed, states, _gpuPhotons, detector, rng, tissue, NUMBER_OF_PHOTONS);
        hipEventRecord(stop);
        hipMemcpy(_cpuPhotons, _gpuPhotons, NUMBER_OF_PHOTONS * sizeof(Photon), hipMemcpyDeviceToHost);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        totalTime += milliseconds;
        hipEventDestroy( start );
        hipEventDestroy( stop );
        for (int j=0; j<NUMBER_OF_PHOTONS; j++){
            totalLifetime += _cpuPhotons[j].getLifetime();
            if ( _cpuPhotons[j].getState() == Photon::DETECTED){
                detected += 1;
            } else if ( _cpuPhotons[j].getState() == Photon::TERMINATED){
                terminated += 1;
            } else {
                escaped += 1 ;
            }
        }
        free(_cpuPhotons);
        hipFree(_gpuPhotons);
        hipFree(states);
        }
    /*std::cout << "# Photons = " << NUMBER_OF_PHOTONS << std::endl;
    std::cout << "Attenuation Coefficient = " << TISSUE_ABSORBTION_COEFFICIENT + TISSUE_SCATTERING_COEFFICIENT << std::endl;
    std::cout << "Average Time = " << totalTime/NUMBER_OF_TEST_RUNS << " ms" << std::endl;
    std::cout << "Average Total # of Walks = " << totalLifetime/NUMBER_OF_TEST_RUNS << " walks" << std::endl;
    std::cout << "Average Detected/Total = " << (detectedRatio/NUMBER_OF_TEST_RUNS)/NUMBER_OF_PHOTONS << std::endl;
    std::cout << "Average Terminated/Total = " << (terminatedRatio/NUMBER_OF_TEST_RUNS)/NUMBER_OF_PHOTONS << std::endl;
    std::cout << "Average Escaped/Total = " << (escapedRatio/NUMBER_OF_TEST_RUNS)/NUMBER_OF_PHOTONS << std::endl;
    std::cout << "#Threads/Block = " << THREADS_PER_BLOCK << std::endl;
    std::cout << "# Blocks = " << NUMBER_OF_PHOTONS / THREADS_PER_BLOCK + 1 << std::endl; */
    float detectedRatio = ((detected/NUMBER_OF_TEST_RUNS)/NUMBER_OF_PHOTONS);
    float terminatedRatio = ((terminated/NUMBER_OF_TEST_RUNS)/NUMBER_OF_PHOTONS);
    float escapedRatio = ((escaped/NUMBER_OF_TEST_RUNS)/NUMBER_OF_PHOTONS);
    streamOut(NUMBER_OF_PHOTONS, (TISSUE_ABSORBTION_COEFFICIENT + TISSUE_SCATTERING_COEFFICIENT), (totalTime/NUMBER_OF_TEST_RUNS), (totalLifetime/NUMBER_OF_TEST_RUNS), (totalLifetime/NUMBER_OF_TEST_RUNS)/NUMBER_OF_PHOTONS, detectedRatio, terminatedRatio, escapedRatio, THREADS_PER_BLOCK, (NUMBER_OF_PHOTONS / THREADS_PER_BLOCK + 1));
}

int main(){
    int number[] = {1,10,100,1000,10000};
    float coefficients[] = {100,1000,10000};
    for(int i = 0; i<(sizeof(number)/sizeof(int)); i++){
        for(int j= 0; j<(sizeof(coefficients)/sizeof(float)); j++){
            run(number[i],coefficients[j]);
            std::cout<<"( "<< number[i] << ", " << coefficients[j] << ")    Done!" << std::endl;
        }
    }
    printf("--------------------------------------\n--------------------------------------\n--------------------------------------\n--------------------------------------\n");
    return 0;
}




