#include "hip/hip_runtime.h"
#include "code/headers/randomwalk.h"

//#define NUMBER_OF_PHOTONS 1000
#define THREADS_PER_BLOCK 1024
#define DETECTOR_RADIUS 10.f
#define DETECTOR_POSITION Point(0.f, 0.f, 50.f)
#define DETECTOR_LOOKAT Vector(0.f, 0.f, -1.f)
#define TISSUE_RADIUS 100.f
#define TISSUE_ABSORBTION_COEFFICIENT 1.f
//#define TISSUE_SCATTERING_COEFFICIENT 100.f
#define TISSUE_CENTER_1 Point(0.f, 0.f, 50.f)
#define TISSUE_CENTER_2 Point(0.f, 0.f, -50.f)

__global__ void finalState(unsigned int seed, hiprandState_t *states, Photon *_gpuPhotons, Detector detector, RNG rng, Tissue tissue, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        hiprand_init(seed, idx, 0, &states[idx]);
        Photon finalState = randomWalk(states, idx, detector, rng, tissue);
        _gpuPhotons[idx] = finalState;
    }
}

int main( int argc, char *argv[] ){
    int NUMBER_OF_PHOTONS = std::atoi(argv[1]);
    unsigned int NUMBER_OF_TEST_RUNS = 10000;
    float TISSUE_SCATTERING_COEFFICIENT  = std::atoi(argv[2]);
    float totalTime = 0;
    unsigned int totalLifetime = 0;
    float detectedRatio, terminatedRatio, escapedRatio = 0.f;
        for (int i= 0; i<NUMBER_OF_TEST_RUNS; i++){
            int nBlocks = NUMBER_OF_PHOTONS / THREADS_PER_BLOCK + 1;
            RNG rng;
            Detector detector = Detector(DETECTOR_RADIUS, DETECTOR_POSITION, DETECTOR_LOOKAT);
            Tissue tissue = Tissue(TISSUE_RADIUS, TISSUE_CENTER_1, TISSUE_CENTER_2, TISSUE_ABSORBTION_COEFFICIENT, TISSUE_SCATTERING_COEFFICIENT);
            hiprandState_t *states;
            hipMalloc((void **)&states, NUMBER_OF_PHOTONS * sizeof(hiprandState_t));
            Photon *_cpuPhotons = (Photon *)malloc(sizeof(Photon) * NUMBER_OF_PHOTONS);
            Photon *_gpuPhotons = nullptr;
            hipMalloc((void **)&_gpuPhotons, NUMBER_OF_PHOTONS * sizeof(Photon));
            unsigned int seed = time(0);
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start); 
            finalState<<<nBlocks, THREADS_PER_BLOCK>>>(seed, states, _gpuPhotons, detector, rng, tissue, NUMBER_OF_PHOTONS);
            hipEventRecord(stop);
            hipMemcpy(_cpuPhotons, _gpuPhotons, NUMBER_OF_PHOTONS * sizeof(Photon), hipMemcpyDeviceToHost);
            hipEventSynchronize(stop);
            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);
            totalTime += milliseconds;
            hipEventDestroy( start );
            hipEventDestroy( stop );
            for (int j=0; j<NUMBER_OF_PHOTONS; j++){
                totalLifetime += _cpuPhotons[j].getLifetime();
                if ( _cpuPhotons[j].getState() == Photon::DETECTED){
                    detectedRatio += 1;
                } else if ( _cpuPhotons[j].getState() == Photon::TERMINATED){
                    terminatedRatio += 1;
                } else {
                    escapedRatio += 1 ;
                }
            }
            free(_cpuPhotons);
            hipFree(_gpuPhotons);
            hipFree(states);
        }
        std::cout << "# Photons = " << NUMBER_OF_PHOTONS << std::endl;
        std::cout << "Attenuation Coefficient = " << TISSUE_ABSORBTION_COEFFICIENT + TISSUE_SCATTERING_COEFFICIENT << std::endl;
        std::cout << "Average Time = " << totalTime/NUMBER_OF_TEST_RUNS << " ms" << std::endl;
        std::cout << "Average Total # of Walks = " << totalLifetime/NUMBER_OF_TEST_RUNS << " walks" << std::endl;
        std::cout << "Average Detected/Total = " << (detectedRatio/NUMBER_OF_TEST_RUNS)/NUMBER_OF_PHOTONS << std::endl;
        std::cout << "Average Terminated/Total = " << (terminatedRatio/NUMBER_OF_TEST_RUNS)/NUMBER_OF_PHOTONS << std::endl;
        std::cout << "Average Escaped/Total = " << (escapedRatio/NUMBER_OF_TEST_RUNS)/NUMBER_OF_PHOTONS << std::endl;
        std::cout << "#Threads/Block = " << THREADS_PER_BLOCK << std::endl;
        std::cout << "# Blocks = " << NUMBER_OF_PHOTONS / THREADS_PER_BLOCK + 1 << std::endl;
        return 0;
}