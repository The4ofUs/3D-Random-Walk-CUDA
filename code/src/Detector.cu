#include "hip/hip_runtime.h"
#include "../headers/Detector.h"

__host__ Detector::Detector(float radius, Point center, Vector normal)
{
    if (radius > 0)
    {
        this->_radius = radius;
        this->_center = center;
        this->_normal = getNormalizedVector(normal);
    }
    else if (radius < 0)
    {
        radius = fabs(radius);
        this->_radius = radius;
        this->_center = center;
        this->_normal = getNormalizedVector(normal);
    }
}

__device__ Point Detector::getCenter() { return this->_center; }

__device__ Vector Detector::getNormal() { return this->_normal; }

__device__ bool Detector::isHit(Photon &photon, Ray path)
{
    float relative_distance = getAbsDistance(path.getTip(), this->_center);
    float rayTipAbsDistance = getAbsDistance(path.getTip());
    float rayOriginAbsDistance = getAbsDistance(path.getOrigin());
    float detectorAbsDistance = getAbsDistance(this->_center);
    if (rayTipAbsDistance >= detectorAbsDistance && rayOriginAbsDistance <= detectorAbsDistance && relative_distance < detectorAbsDistance)
    {
        Point point = getIntersectionPoint(path);
        float dfromc = getAbsDistance(point, this->_center);
        if (dfromc <= this->_radius)
        {
            photon.setPosition(point);
            return true;
        }
        else
            return false;
    }
    else
        return false;
}

__device__ Point Detector::getIntersectionPoint(Ray path)
{
    /**
        P is a point on the ray
        A is the ray's origin
        B is the direction of the ray
        let's assume that P is on the detector's plane
        We know that P = A + B*t    (1)
        where t is a parameter that determines how far the ray will move in direction B
        (P - Center) will yield a vector on the plane
        so (P - Center).normal = 0  (2)
        now substitute from (1) in (2)
        we will yield an equation with t as the unknown
        we calculate t and substitute in the ray's equation to get the intersection point
        and Voila
    */
    Point A = path.getOrigin();
    Vector B = path.getDirection();
    Vector V = Vector(A, this->_center);
    float t = getDotProduct(V, this->_normal) / getDotProduct(B, this->_normal);
    return A + B * t;
}
