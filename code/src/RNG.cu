#include "hip/hip_runtime.h"
#include "../headers/RNG.h"

__device__ float RNG::generate(hiprandState *globalState, int i)
{
    hiprandState localState = globalState[i];
    float random = hiprand_uniform(&localState);
    globalState[i] = localState;
    return random;
}

__device__ float RNG::getRandomStep(hiprandState *globalState, int i)
{
    float step = 0.f; // Intialize for step value
    step = generate(globalState, i);
    return step;
}

__device__ Vector RNG::getRandomDirection(hiprandState *globalState, int i)
{
    float u = generate(globalState, i);
    float v = generate(globalState, i);

    float theta = 2 * M_PI * u;
    float phi = acos(1 - 2 * v);

    // Transforming into the cartesian space
    float x = sin(phi) * cos(theta);
    float y = sin(phi) * sin(theta);
    float z = cos(phi);

    return Mathematics::calculateNormalizedVector(Vector(x, y, z));
}

__device__ Point RNG::getRandomPoint(hiprandState *globalState, int i)
{
    float u = generate(globalState, i);
    float v = generate(globalState, i);

    float theta = 2 * M_PI * u;
    float phi = acos(1 - 2 * v);

    // Transforming into the cartesian space
    float x = sin(phi) * cos(theta);
    float y = sin(phi) * sin(theta);
    float z = cos(phi);

    return Point(x, y, z);
}

__device__ void RNG::roulette(Photon &photon, float chance, hiprandState *globalState, int i)
{
    if (generate(globalState, i) >= chance)
    {
        photon.terminate();
    }
    else
    {
        photon.boost(chance);
    }
}
