#include "hip/hip_runtime.h"
#include "../headers/Ray.h"

/** TODO: Make sure step is normalized by getting the maximum generated step from RNG
 * */
__device__ __host__ Ray::Ray(Point origin, Vector direction, float step)
{
    this->_origin = origin;
    this->_direction = Mathematics::calculateNormalizedVector(direction);
    this->_step = step;
    this->_tip = Mathematics::calculateRayTip(origin, direction, step);
}

__device__ __host__ Ray::Ray()
{
    this->_origin = Point();
    this->_direction = Vector();
}

__device__ __host__ Vector Ray::getDirection() const { return this->_direction; }

__device__ __host__ Point Ray::getOrigin() const { return this->_origin; }

__device__ __host__ float Ray::getStep() const { return this->_step; }

__device__ __host__ Point Ray::getTip() const { return this->_tip; }

__device__ __host__ void Ray::setDirection(Vector v) { this->_direction = Mathematics::calculateNormalizedVector(v); }

__device__ __host__ void Ray::setOrigin(Point p) { this->_origin = p; }

__device__ __host__ void Ray::setStep(float step)
{
    this->_step = fabs(step);
    this->_tip = this->_direction * this->_step;
}
