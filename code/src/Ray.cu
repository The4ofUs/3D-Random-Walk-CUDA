#include "hip/hip_runtime.h"
#include "../headers/Ray.h"

/** TODO: Make sure step is normalized by getting the maximum generated step from RNG
 * */
__device__ __host__ Ray::Ray(Point origin, Vector direction, float step)
{
    this->_origin = origin;
    this->_direction = getNormalizedVector(direction);
    this->_step = step;
}

__device__ __host__ Ray::Ray()
{
    this->_origin = Point();
    this->_direction = Vector();
}

__device__ __host__ Vector Ray::getDirection() const { return this->_direction; }

__device__ __host__ Point Ray::getOrigin() const { return this->_origin; }

__device__ __host__ float Ray::getStep() const { return this->_step; }

__device__ __host__ Point Ray::getTip() const { return this->_tip; }

__device__ __host__ void Ray::setDirection(Vector v) { this->_direction = getNormalizedVector(v); }

__device__ __host__ void Ray::setOrigin(Point p) { this->_origin = p; }

__device__ __host__ void Ray::setStep(float step)
{
    this->_step = fabs(step);
    this->_tip = this->_direction * this->_step;
}
